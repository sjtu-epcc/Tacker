
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hipblas.h>
#include <mma.h>
#include <malloc.h>
#include <sys/time.h>
#include <vector>
#include <algorithm>
using namespace nvcuda; 

#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
   }
}

#define checkKernelErrors(expr)                             \
  do {                                                      \
    expr;                                                   \
                                                            \
    hipError_t __err = hipGetLastError();                 \
    if (__err != hipSuccess) {                             \
      printf("Line %d: '%s' failed: %s\n", __LINE__, #expr, \
             hipGetErrorString(__err));                    \
      abort();                                              \
    }                                                       \
  } while (0)

#include "header/mriq_header.h"
#include "kernel/mriq_kernel.cu"

int main(int argc, char* argv[]) {
    int errors = 0;

	  // variables
    // ---------------------------------------------------------------------------------------
		float kernel_time;
		hipEvent_t startKERNEL;
		hipEvent_t stopKERNEL;
		cudaErrCheck(hipEventCreate(&startKERNEL));
		cudaErrCheck(hipEventCreate(&stopKERNEL));
    // ---------------------------------------------------------------------------------------

    // mriq variables
    // ---------------------------------------------------------------------------------------
        int mriq_blks = 4;
        int mriq_iter = 1;
        int numK = 2097152;
        int numX = 2097152;
        float *base_kx, *base_ky, *base_kz;		/* K trajectory (3D vectors) */
        float *base_x, *base_y, *base_z;		/* X coordinates (3D vectors) */
        float *base_phiR, *base_phiI;		    /* Phi values (complex) */
        // float *base_phiMag;		                /* Magnitude of Phi */
        // float *base_Qr, *base_Qi;		        /* Q signal (complex) */
        struct mriq_kValues* mriq_kVals;

        // kernel 1
        float *mriq_ori_phiR, *mriq_ori_phiI;
        float *mriq_ori_phiMag, *host_mriq_ori_phiMag;
        // kernel 2
        float *mriq_ori_x, *mriq_ori_y, *mriq_ori_z;
        float *mriq_ori_Qr, *mriq_ori_Qi, *host_mriq_ori_Qi;

        // // kernel 1
        // float *ptb_phiR, *ptb_phiI;
        // float *ptb_phiMag, *host_ptb_phiMag;
        // kernel 2
        float *mriq_ptb_x, *mriq_ptb_y, *mriq_ptb_z;
        float *mriq_ptb_Qr, *mriq_ptb_Qi, *host_mriq_ptb_Qi;

        // gptb kernel 2
        float *mriq_gptb_x, *mriq_gptb_y, *mriq_gptb_z;
        float *mriq_gptb_Qr, *mriq_gptb_Qi, *host_mriq_gptb_Qi;

        inputData(&numK, &numX,
            &base_kx, &base_ky, &base_kz,
            &base_x, &base_y, &base_z,
            &base_phiR, &base_phiI);
        numK = 2097152;

        // Memory allocation
        // base_phiMag = (float* ) memalign(16, numK * sizeof(float));
        // base_Qr = (float*) memalign(16, numX * sizeof (float));
        // base_Qi = (float*) memalign(16, numX * sizeof (float));
        cudaErrCheck(hipMalloc((void **)&mriq_ori_phiR, numK * sizeof(float)));   
        cudaErrCheck(hipMalloc((void **)&mriq_ori_phiI, numK * sizeof(float)));
        cudaErrCheck(hipMalloc((void **)&mriq_ori_phiMag, numK * sizeof(float)));
        host_mriq_ori_phiMag = (float* ) memalign(16, numK * sizeof(float));
        cudaErrCheck(hipMemcpy(mriq_ori_phiR, base_phiR, numK * sizeof(float), hipMemcpyHostToDevice));
        cudaErrCheck(hipMemcpy(mriq_ori_phiI, base_phiI, numK * sizeof(float), hipMemcpyHostToDevice));

        cudaErrCheck(hipMalloc((void **)&mriq_ori_x, numX * sizeof(float)));
        cudaErrCheck(hipMalloc((void **)&mriq_ori_y, numX * sizeof(float)));
        cudaErrCheck(hipMalloc((void **)&mriq_ori_z, numX * sizeof(float)));
        cudaErrCheck(hipMemcpy(mriq_ori_x, base_x, numX * sizeof(float), hipMemcpyHostToDevice));
        cudaErrCheck(hipMemcpy(mriq_ori_y, base_y, numX * sizeof(float), hipMemcpyHostToDevice));
        cudaErrCheck(hipMemcpy(mriq_ori_z, base_z, numX * sizeof(float), hipMemcpyHostToDevice));
        cudaErrCheck(hipMalloc((void **)&mriq_ori_Qr, numX * sizeof(float)));
        cudaErrCheck(hipMalloc((void **)&mriq_ori_Qi, numX * sizeof(float)));
        hipMemset((void *)mriq_ori_Qr, 0, numX * sizeof(float));
        hipMemset((void *)mriq_ori_Qi, 0, numX * sizeof(float));
        host_mriq_ori_Qi = (float*) memalign(16, numX * sizeof (float));

        // cudaErrCheck(hipMalloc((void **)&ptb_phiR, numK * sizeof(float)));   
        // cudaErrCheck(hipMalloc((void **)&ptb_phiI, numK * sizeof(float)));
        // cudaErrCheck(hipMalloc((void **)&ptb_phiMag, numK * sizeof(float)));
        // host_ptb_phiMag = (float* ) memalign(16, numK * sizeof(float));
        // cudaErrCheck(hipMemcpy(ptb_phiR, base_phiR, numK * sizeof(float), hipMemcpyHostToDevice));
        // cudaErrCheck(hipMemcpy(ptb_phiI, base_phiI, numK * sizeof(float), hipMemcpyHostToDevice));

        cudaErrCheck(hipMalloc((void **)&mriq_ptb_x, numX * sizeof(float)));
        cudaErrCheck(hipMalloc((void **)&mriq_ptb_y, numX * sizeof(float)));
        cudaErrCheck(hipMalloc((void **)&mriq_ptb_z, numX * sizeof(float)));
        cudaErrCheck(hipMemcpy(mriq_ptb_x, base_x, numX * sizeof(float), hipMemcpyHostToDevice));
        cudaErrCheck(hipMemcpy(mriq_ptb_y, base_y, numX * sizeof(float), hipMemcpyHostToDevice));
        cudaErrCheck(hipMemcpy(mriq_ptb_z, base_z, numX * sizeof(float), hipMemcpyHostToDevice));
        cudaErrCheck(hipMalloc((void **)&mriq_ptb_Qr, numX * sizeof(float)));
        cudaErrCheck(hipMalloc((void **)&mriq_ptb_Qi, numX * sizeof(float)));
        hipMemset((void *)mriq_ptb_Qr, 0, numX * sizeof(float));
        hipMemset((void *)mriq_ptb_Qi, 0, numX * sizeof(float));
        host_mriq_ptb_Qi = (float*) memalign(16, numX * sizeof (float));

        // gptb
        cudaErrCheck(hipMalloc((void **)&mriq_gptb_x, numX * sizeof(float)));
        cudaErrCheck(hipMalloc((void **)&mriq_gptb_y, numX * sizeof(float)));
        cudaErrCheck(hipMalloc((void **)&mriq_gptb_z, numX * sizeof(float)));
        cudaErrCheck(hipMemcpy(mriq_gptb_x, base_x, numX * sizeof(float), hipMemcpyHostToDevice));
        cudaErrCheck(hipMemcpy(mriq_gptb_y, base_y, numX * sizeof(float), hipMemcpyHostToDevice));
        cudaErrCheck(hipMemcpy(mriq_gptb_z, base_z, numX * sizeof(float), hipMemcpyHostToDevice));
        cudaErrCheck(hipMalloc((void **)&mriq_gptb_Qr, numX * sizeof(float)));
        cudaErrCheck(hipMalloc((void **)&mriq_gptb_Qi, numX * sizeof(float)));
        hipMemset((void *)mriq_gptb_Qr, 0, numX * sizeof(float));
        hipMemset((void *)mriq_gptb_Qi, 0, numX * sizeof(float));
        host_mriq_gptb_Qi = (float*) memalign(16, numX * sizeof (float));
    // ---------------------------------------------------------------------------------------

    // PRE running
    // ---------------------------------------------------------------------------------------
        dim3 mriq_grid1;
        dim3 mriq_block1;
        mriq_grid1.x = numK / KERNEL_PHI_MAG_THREADS_PER_BLOCK;
        mriq_grid1.y = 1;
        mriq_block1.x = KERNEL_PHI_MAG_THREADS_PER_BLOCK;
        mriq_block1.y = 1;
        printf("[ORI] Running with mriq...\n");
        printf("[ORI] mriq_grid1 -- %d * %d * %d mriq_block1 -- %d * %d * %d \n", 
            mriq_grid1.x, mriq_grid1.y, mriq_grid1.z, mriq_block1.x, mriq_block1.y, mriq_block1.z);

        checkKernelErrors((ori_ComputePhiMag <<< mriq_grid1, mriq_block1 >>> (mriq_ori_phiR, mriq_ori_phiI, mriq_ori_phiMag, numK)));
        hipMemcpy(host_mriq_ori_phiMag, mriq_ori_phiMag, numK * sizeof(float), hipMemcpyDeviceToHost);

        mriq_kVals = (struct mriq_kValues*)calloc(numK, sizeof (struct mriq_kValues));
        for (int k = 0; k < numK; k++) {
            mriq_kVals[k].Kx = base_kx[k];
            mriq_kVals[k].Ky = base_ky[k];
            mriq_kVals[k].Kz = base_kz[k];
            mriq_kVals[k].PhiMag = host_mriq_ori_phiMag[k];
        }
    // ---------------------------------------------------------------------------------------

    // SOLO running
    // ---------------------------------------------------------------------------------------
        numX = (numX / 10) * mriq_iter;

        dim3 mriq_grid2, ori_mriq_grid2;
        dim3 mriq_block2, ori_mriq_block2;
        mriq_grid2.x = numX / KERNEL_Q_THREADS_PER_BLOCK;
        mriq_grid2.y = 1;
        mriq_block2.x = KERNEL_Q_THREADS_PER_BLOCK;
        mriq_block2.y = 1;
        ori_mriq_grid2 = mriq_grid2;
        ori_mriq_block2 = mriq_block2;
        printf("[ORI] mriq_grid2 -- %d * %d * %d mriq_block2 -- %d * %d * %d \n", 
            mriq_grid2.x, mriq_grid2.y, mriq_grid2.z, mriq_block2.x, mriq_block2.y, mriq_block2.z);

        int QGridBase = 0 * KERNEL_Q_K_ELEMS_PER_GRID;
        mriq_kValues* kValsTile = mriq_kVals + QGridBase;
        hipMemcpyToSymbol(HIP_SYMBOL(ck), kValsTile, KERNEL_Q_K_ELEMS_PER_GRID * sizeof(mriq_kValues), 0);

        cudaErrCheck(hipEventRecord(startKERNEL));
        checkKernelErrors((ori_mriq <<< mriq_grid2, mriq_block2 >>>(numK, QGridBase, 
                                mriq_ori_x, mriq_ori_y, mriq_ori_z, mriq_ori_Qr, mriq_ori_Qi, 
                                1)));
        cudaErrCheck(hipEventRecord(stopKERNEL));
        cudaErrCheck(hipEventSynchronize(stopKERNEL));
        cudaErrCheck(hipEventElapsedTime(&kernel_time, startKERNEL, stopKERNEL));
        printf("[ORI] mriq took %f ms\n\n", kernel_time);
    // ---------------------------------------------------------------------------------------


    // PTB running
    // ---------------------------------------------------------------------------------------
        int mriq_grid2_dim_x = mriq_grid2.x;
        // int mriq_block2_dim_x = mriq_block2.x;
        mriq_grid2.x = SM_NUM * 2;
        mriq_grid2.x = mriq_blks == 0 ? mriq_grid2_dim_x : SM_NUM * mriq_blks;
        printf("[PTB] Running with mriq...\n");
        printf("[PTB] mriq_grid2 -- %d * %d * %d mriq_block2 -- %d * %d * %d \n", 
            mriq_grid2.x, mriq_grid2.y, mriq_grid2.z, mriq_block2.x, mriq_block2.y, mriq_block2.z);

        cudaErrCheck(hipEventRecord(startKERNEL));
        checkKernelErrors((ptb2_mriq <<< mriq_grid2, mriq_block2 >>>(numK, QGridBase, 
                                mriq_ptb_x, mriq_ptb_y, mriq_ptb_z, mriq_ptb_Qr, mriq_ptb_Qi, 
                                mriq_grid2_dim_x, 
                                1)));
        cudaErrCheck(hipEventRecord(stopKERNEL));
        cudaErrCheck(hipEventSynchronize(stopKERNEL));
        cudaErrCheck(hipEventElapsedTime(&kernel_time, startKERNEL, stopKERNEL));
        printf("[PTB] mriq took %f ms\n\n", kernel_time);
    // ---------------------------------------------------------------------------------------


    std::vector<float> time_vec;
    // GPTB
    // ---------------------------------------------------------------------------------------
        dim3 gptb_kernel_grid = dim3(272, 1, 1);
        dim3 gptb_kernel_block = dim3(256, 1, 1);
        for(int i = 0; i < 30; ++i) {
            cudaErrCheck(hipEventRecord(startKERNEL));
            checkKernelErrors((g_general_ptb_mriq <<<gptb_kernel_grid, gptb_kernel_block>>>(numK, QGridBase, mriq_gptb_x, mriq_gptb_y, mriq_gptb_z, mriq_gptb_Qr, mriq_gptb_Qi, 
    ori_mriq_grid2.x, ori_mriq_grid2.y, ori_mriq_grid2.z, ori_mriq_block2.x, ori_mriq_block2.y, ori_mriq_block2.z,
    0, gptb_kernel_grid.x * gptb_kernel_grid.y * gptb_kernel_grid.z, 816, 0)));
            cudaErrCheck(hipEventRecord(stopKERNEL));
            cudaErrCheck(hipEventSynchronize(stopKERNEL));
            cudaErrCheck(hipEventElapsedTime(&kernel_time, startKERNEL, stopKERNEL));
            time_vec.push_back(kernel_time);
        }

        // sort & get average
        std::sort(time_vec.begin(), time_vec.end());
        float gptb_mriq_time = 0.0f;
        for(int i = 10; i < 20; ++i) {
            gptb_mriq_time += time_vec[i];
        }
        gptb_mriq_time /= 10.0f;
        time_vec.clear();
        printf("[GPTB] mriq took %f ms\n", gptb_mriq_time);
        printf("[GPTB] mriq blks: %d\n\n", 816 - 0);

        printf("---------------------------\n");

}
