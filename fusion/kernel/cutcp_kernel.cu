#include "hip/hip_runtime.h"

__global__ void ori_cutcp(
    int binDim_x,
    int binDim_y,
    float4 *binZeroAddr,    /* address of atom bins starting at origin */
    float h,                /* lattice spacing */
    float cutoff2,          /* square of cutoff distance */
    float inv_cutoff2,
    float *regionZeroAddr, /* address of lattice regions starting at origin */
    int zRegionIndex_t,
    int iteration
    )
{
	__shared__ float AtomBinCache[BIN_CACHE_MAXLEN * BIN_DEPTH * 4];
    // __shared__ float *myRegionAddr;
	// __shared__ int3 myBinIndex;

    float *myRegionAddr;
	int3 myBinIndex;

	const int xRegionIndex = blockIdx.x;
	const int yRegionIndex = blockIdx.y;
    const int zRegionIndex = blockIdx.z;
	/* thread id */
	const int tid = (threadIdx.z*blockDim.y + threadIdx.y)*blockDim.x + threadIdx.x;

    for (int loop = 0; loop < iteration; loop++) {
        /* neighbor index */
        int nbrid;

        /* this is the start of the sub-region indexed by tid */
        myRegionAddr = regionZeroAddr + ((zRegionIndex*gridDim.y + yRegionIndex)*gridDim.x + xRegionIndex)*REGION_SIZE;

        /* spatial coordinate of this lattice point */
        float x = (8 * xRegionIndex + threadIdx.x) * h;
        float y = (8 * yRegionIndex + threadIdx.y) * h;
        float z = (8 * zRegionIndex + threadIdx.z) * h;

        int totalbins = 0;
        int numbins;

        /* bin number determined by center of region */
        myBinIndex.x = (int) floorf((8 * xRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.y = (int) floorf((8 * yRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.z = (int) floorf((8 * zRegionIndex + 4) * h * BIN_INVLEN);

        /* first neighbor in list for me to cache */
        nbrid = (tid >> 4);

        numbins = BIN_CACHE_MAXLEN;

        float energy0 = 0.f;
        float energy1 = 0.f;
        float energy2 = 0.f;
        float energy3 = 0.f;


        for (totalbins = 0;  totalbins < NbrListLen;  totalbins += numbins) {
            int bincnt;

            /* start of where to write in shared memory */
            int startoff = BIN_SIZE * (tid >> 4);

            /* each half-warp to cache up to 4 atom bins */
            for (bincnt = 0;  bincnt < 4 && nbrid < NbrListLen;  bincnt++, nbrid += 8) {
                int i = myBinIndex.x + NbrList[nbrid].x;
                int j = myBinIndex.y + NbrList[nbrid].y;
                int k = myBinIndex.z + NbrList[nbrid].z;

                /* determine global memory location of atom bin */
                float *p_global = ((float *) binZeroAddr) + (((__mul24(k, binDim_y) + j)*binDim_x + i) << BIN_SHIFT);

                /* coalesced read from global memory -
                * retain same ordering in shared memory for now */
                int binIndex = startoff + (bincnt << (3 + BIN_SHIFT));
                int tidmask = tid & 15;

                AtomBinCache[binIndex + tidmask   ] = p_global[tidmask   ];
                AtomBinCache[binIndex + tidmask+16] = p_global[tidmask+16];
            }
            __syncthreads();

            /* no warp divergence */
            if (totalbins + BIN_CACHE_MAXLEN > NbrListLen) {
                numbins = NbrListLen - totalbins;
            }

            int stopbin = (numbins << BIN_SHIFT);
            for (bincnt = 0; bincnt < stopbin; bincnt+=BIN_SIZE) {
                for (int i = 0;  i < BIN_DEPTH;  i++) {
                    int off = bincnt + (i<<2);

                    float aq = AtomBinCache[off + 3];
                    if (0.f == aq) 
                        break;  /* no more atoms in bin */

                    float dx = AtomBinCache[off    ] - x;
                    float dz = AtomBinCache[off + 2] - z;
                    float dxdz2 = dx*dx + dz*dz;
                    float dy = AtomBinCache[off + 1] - y;
                    float r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy0 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy1 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy2 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy3 += aq * rsqrtf(r2) * s * s;
                    }
                } /* end loop over atoms in bin */
            } /* end loop over cached atom bins */
            __syncthreads();
        } /* end loop over neighbor list */

        /* store into global memory */
        myRegionAddr[(tid>>4)*64 + (tid&15)     ] = energy0;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 16] = energy1;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 32] = energy2;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 48] = energy3;
    }
}


__global__ void ptb_cutcp(
    int binDim_x,
    int binDim_y,
    float4 *binZeroAddr,    /* address of atom bins starting at origin */
    float h,                /* lattice spacing */
    float cutoff2,          /* square of cutoff distance */
    float inv_cutoff2,
    float *regionZeroAddr, /* address of lattice regions starting at origin */
    int zRegionIndex_t,
    int grid_dimension_x,
    int grid_dimension_y,
    int grid_dimension_z,
    int block_dimension_x,
    int block_dimension_y,
    int block_dimension_z,
    int iteration
    ) {
    unsigned int block_pos = blockIdx.x;
    int thread_id_x = threadIdx.x / (block_dimension_y * block_dimension_z);
    int thread_id_y = (threadIdx.x % (block_dimension_y * block_dimension_z)) / block_dimension_z;
    int thread_id_z = (threadIdx.x % (block_dimension_y * block_dimension_z)) % block_dimension_z;
    // 这里等于 threadIdx.x % block_dimension_z

    /* thread id */
	const int tid = (thread_id_z * block_dimension_y + thread_id_y) * block_dimension_x + thread_id_x;

    __shared__ float AtomBinCache[BIN_CACHE_MAXLEN * BIN_DEPTH * 4];
	// __shared__ float *myRegionAddr;
	// __shared__ int3 myBinIndex;

    float *myRegionAddr;
	int3 myBinIndex;

    for (;; block_pos += gridDim.x) {
        if (block_pos >= grid_dimension_x * grid_dimension_y * grid_dimension_z) {
            return;
        }

        int block_id_x = block_pos / (grid_dimension_y * grid_dimension_z);
        int block_id_y = (block_pos % (grid_dimension_y * grid_dimension_z)) / grid_dimension_z;
        int block_id_z = (block_pos % (grid_dimension_y * grid_dimension_z)) % grid_dimension_z;

        for (int loop = 0; loop < iteration; loop++) {

            int xRegionIndex = block_id_x;
            int yRegionIndex = block_id_y;
            int zRegionIndex = block_id_z;
        
            /* neighbor index */
            int nbrid;

            /* this is the start of the sub-region indexed by tid */
            myRegionAddr = regionZeroAddr + ((zRegionIndex*grid_dimension_y + yRegionIndex)*grid_dimension_x + xRegionIndex)*REGION_SIZE;

            /* spatial coordinate of this lattice point */
            float x = (8 * xRegionIndex + thread_id_x) * h;
            float y = (8 * yRegionIndex + thread_id_y) * h;
            float z = (8 * zRegionIndex + thread_id_z) * h;

            int totalbins = 0;
            int numbins;

            /* bin number determined by center of region */
            myBinIndex.x = (int) floorf((8 * xRegionIndex + 4) * h * BIN_INVLEN);
            myBinIndex.y = (int) floorf((8 * yRegionIndex + 4) * h * BIN_INVLEN);
            myBinIndex.z = (int) floorf((8 * zRegionIndex + 4) * h * BIN_INVLEN);

            /* first neighbor in list for me to cache */
            nbrid = (tid >> 4);

            numbins = BIN_CACHE_MAXLEN;

            float energy0 = 0.f;
            float energy1 = 0.f;
            float energy2 = 0.f;
            float energy3 = 0.f;


            for (totalbins = 0;  totalbins < NbrListLen;  totalbins += numbins) {
                int bincnt;

                /* start of where to write in shared memory */
                int startoff = BIN_SIZE * (tid >> 4);

                /* each half-warp to cache up to 4 atom bins */
                for (bincnt = 0;  bincnt < 4 && nbrid < NbrListLen;  bincnt++, nbrid += 8) {
                    int i = myBinIndex.x + NbrList[nbrid].x;
                    int j = myBinIndex.y + NbrList[nbrid].y;
                    int k = myBinIndex.z + NbrList[nbrid].z;

                    /* determine global memory location of atom bin */
                    float *p_global = ((float *) binZeroAddr) + (((__mul24(k, binDim_y) + j)*binDim_x + i) << BIN_SHIFT);

                    /* coalesced read from global memory -
                    * retain same ordering in shared memory for now */
                    int binIndex = startoff + (bincnt << (3 + BIN_SHIFT));
                    int tidmask = tid & 15;

                    AtomBinCache[binIndex + tidmask   ] = p_global[tidmask   ];
                    AtomBinCache[binIndex + tidmask+16] = p_global[tidmask+16];
                }
               __syncthreads();

                /* no warp divergence */
                if (totalbins + BIN_CACHE_MAXLEN > NbrListLen) {
                    numbins = NbrListLen - totalbins;
                }

                int stopbin = (numbins << BIN_SHIFT);
                for (bincnt = 0; bincnt < stopbin; bincnt+=BIN_SIZE) {
                    for (int i = 0;  i < BIN_DEPTH;  i++) {
                        int off = bincnt + (i<<2);

                        float aq = AtomBinCache[off + 3];
                        if (0.f == aq) 
                            break;  /* no more atoms in bin */

                        float dx = AtomBinCache[off    ] - x;
                        float dz = AtomBinCache[off + 2] - z;
                        float dxdz2 = dx*dx + dz*dz;
                        float dy = AtomBinCache[off + 1] - y;
                        float r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy0 += aq * rsqrtf(r2) * s * s;
                        }
                        dy -= 2.0f*h;
                        r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy1 += aq * rsqrtf(r2) * s * s;
                        }
                        dy -= 2.0f*h;
                        r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy2 += aq * rsqrtf(r2) * s * s;
                        }
                        dy -= 2.0f*h;
                        r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy3 += aq * rsqrtf(r2) * s * s;
                        }
                    } /* end loop over atoms in bin */
                } /* end loop over cached atom bins */
               __syncthreads();
            } /* end loop over neighbor list */

            /* store into global memory */
            myRegionAddr[(tid>>4)*64 + (tid&15)     ] = energy0;
            myRegionAddr[(tid>>4)*64 + (tid&15) + 16] = energy1;
            myRegionAddr[(tid>>4)*64 + (tid&15) + 32] = energy2;
            myRegionAddr[(tid>>4)*64 + (tid&15) + 48] = energy3;
        }
    }
}


__global__ void ptb2_cutcp(
    int binDim_x,
    int binDim_y,
    float4 *binZeroAddr,    /* address of atom bins starting at origin */
    float h,                /* lattice spacing */
    float cutoff2,          /* square of cutoff distance */
    float inv_cutoff2,
    float *regionZeroAddr, /* address of lattice regions starting at origin */
    int zRegionIndex_t,
    int grid_dimension_x,
    int grid_dimension_y,
    int grid_dimension_z,
    int iteration
    ) {
    unsigned int block_pos = blockIdx.x;
    /* thread id */
	const int tid = (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;

    __shared__ float AtomBinCache[BIN_CACHE_MAXLEN * BIN_DEPTH * 4];
	// __shared__ float *myRegionAddr;
	// __shared__ int3 myBinIndex;

    float *myRegionAddr;
	int3 myBinIndex;

    for (;; block_pos += gridDim.x) {
        if (block_pos >= grid_dimension_x * grid_dimension_y * grid_dimension_z) {
            return;
        }

        int block_id_x = block_pos / (grid_dimension_y * grid_dimension_z);
        int block_id_y = (block_pos % (grid_dimension_y * grid_dimension_z)) / grid_dimension_z;
        int block_id_z = (block_pos % (grid_dimension_y * grid_dimension_z)) % grid_dimension_z;

        for (int loop = 0; loop < iteration; loop++) {

            int xRegionIndex = block_id_x;
            int yRegionIndex = block_id_y;
            int zRegionIndex = block_id_z;
        
            /* neighbor index */
            int nbrid;

            /* this is the start of the sub-region indexed by tid */
            myRegionAddr = regionZeroAddr + ((zRegionIndex*grid_dimension_y + yRegionIndex)*grid_dimension_x + xRegionIndex)*REGION_SIZE;

            /* spatial coordinate of this lattice point */
            float x = (8 * xRegionIndex + threadIdx.x) * h;
            float y = (8 * yRegionIndex + threadIdx.y) * h;
            float z = (8 * zRegionIndex + threadIdx.z) * h;

            int totalbins = 0;
            int numbins;

            /* bin number determined by center of region */
            myBinIndex.x = (int) floorf((8 * xRegionIndex + 4) * h * BIN_INVLEN);
            myBinIndex.y = (int) floorf((8 * yRegionIndex + 4) * h * BIN_INVLEN);
            myBinIndex.z = (int) floorf((8 * zRegionIndex + 4) * h * BIN_INVLEN);

            /* first neighbor in list for me to cache */
            nbrid = (tid >> 4);

            numbins = BIN_CACHE_MAXLEN;

            float energy0 = 0.f;
            float energy1 = 0.f;
            float energy2 = 0.f;
            float energy3 = 0.f;


            for (totalbins = 0;  totalbins < NbrListLen;  totalbins += numbins) {
                int bincnt;

                /* start of where to write in shared memory */
                int startoff = BIN_SIZE * (tid >> 4);

                /* each half-warp to cache up to 4 atom bins */
                for (bincnt = 0;  bincnt < 4 && nbrid < NbrListLen;  bincnt++, nbrid += 8) {
                    int i = myBinIndex.x + NbrList[nbrid].x;
                    int j = myBinIndex.y + NbrList[nbrid].y;
                    int k = myBinIndex.z + NbrList[nbrid].z;

                    /* determine global memory location of atom bin */
                    float *p_global = ((float *) binZeroAddr) + (((__mul24(k, binDim_y) + j)*binDim_x + i) << BIN_SHIFT);

                    /* coalesced read from global memory -
                    * retain same ordering in shared memory for now */
                    int binIndex = startoff + (bincnt << (3 + BIN_SHIFT));
                    int tidmask = tid & 15;

                    AtomBinCache[binIndex + tidmask   ] = p_global[tidmask   ];
                    AtomBinCache[binIndex + tidmask+16] = p_global[tidmask+16];
                }
               __syncthreads();

                /* no warp divergence */
                if (totalbins + BIN_CACHE_MAXLEN > NbrListLen) {
                    numbins = NbrListLen - totalbins;
                }

                int stopbin = (numbins << BIN_SHIFT);
                for (bincnt = 0; bincnt < stopbin; bincnt+=BIN_SIZE) {
                    for (int i = 0;  i < BIN_DEPTH;  i++) {
                        int off = bincnt + (i<<2);

                        float aq = AtomBinCache[off + 3];
                        if (0.f == aq) 
                            break;  /* no more atoms in bin */

                        float dx = AtomBinCache[off    ] - x;
                        float dz = AtomBinCache[off + 2] - z;
                        float dxdz2 = dx*dx + dz*dz;
                        float dy = AtomBinCache[off + 1] - y;
                        float r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy0 += aq * rsqrtf(r2) * s * s;
                        }
                        dy -= 2.0f*h;
                        r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy1 += aq * rsqrtf(r2) * s * s;
                        }
                        dy -= 2.0f*h;
                        r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy2 += aq * rsqrtf(r2) * s * s;
                        }
                        dy -= 2.0f*h;
                        r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy3 += aq * rsqrtf(r2) * s * s;
                        }
                    } /* end loop over atoms in bin */
                } /* end loop over cached atom bins */
               __syncthreads();
            } /* end loop over neighbor list */

            /* store into global memory */
            myRegionAddr[(tid>>4)*64 + (tid&15)     ] = energy0;
            myRegionAddr[(tid>>4)*64 + (tid&15) + 16] = energy1;
            myRegionAddr[(tid>>4)*64 + (tid&15) + 32] = energy2;
            myRegionAddr[(tid>>4)*64 + (tid&15) + 48] = energy3;
        }
    }
}


__device__ void mix_cutcp0(
    int binDim_x,
    int binDim_y,
    float4 *binZeroAddr,    /* address of atom bins starting at origin */
    float h,                /* lattice spacing */
    float cutoff2,          /* square of cutoff distance */
    float inv_cutoff2,
    float *regionZeroAddr, /* address of lattice regions starting at origin */
    int zRegionIndex_t,
    int grid_dimension_x,
    int grid_dimension_y,
    int grid_dimension_z,
    int block_dimension_x,
    int block_dimension_y,
    int block_dimension_z,
    int thread_step,
    int iteration
    ) {
    unsigned int block_pos = blockIdx.x;
    int thread_id_x = (threadIdx.x - thread_step) / (block_dimension_y * block_dimension_z);
    int thread_id_y = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) / block_dimension_z;
    int thread_id_z = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) % block_dimension_z;

    /* thread id */
	const int tid = (thread_id_z * block_dimension_y + thread_id_y) * block_dimension_x + thread_id_x;

    __shared__ float AtomBinCache[BIN_CACHE_MAXLEN * BIN_DEPTH * 4];
	// __shared__ float *myRegionAddr;
	// __shared__ int3 myBinIndex;

    float *myRegionAddr;
	int3 myBinIndex;

    for (;; block_pos += CUTCP_GRID_DIM) {
        if (block_pos >= grid_dimension_x * grid_dimension_y * grid_dimension_z) {
            return;
        }

        int block_id_x = block_pos / (grid_dimension_y * grid_dimension_z);
        int block_id_y = (block_pos % (grid_dimension_y * grid_dimension_z)) / grid_dimension_z;
        int block_id_z = (block_pos % (grid_dimension_y * grid_dimension_z)) % grid_dimension_z;

        for (int loop = 0; loop < iteration; loop++) {

            int xRegionIndex = block_id_x;
            int yRegionIndex = block_id_y;
            int zRegionIndex = block_id_z;
        
            /* neighbor index */
            int nbrid;

            /* this is the start of the sub-region indexed by tid */
            myRegionAddr = regionZeroAddr + ((zRegionIndex*grid_dimension_y + yRegionIndex)*grid_dimension_x + xRegionIndex)*REGION_SIZE;

            /* spatial coordinate of this lattice point */
            float x = (8 * xRegionIndex + thread_id_x) * h;
            float y = (8 * yRegionIndex + thread_id_y) * h;
            float z = (8 * zRegionIndex + thread_id_z) * h;

            int totalbins = 0;
            int numbins;

            /* bin number determined by center of region */
            myBinIndex.x = (int) floorf((8 * xRegionIndex + 4) * h * BIN_INVLEN);
            myBinIndex.y = (int) floorf((8 * yRegionIndex + 4) * h * BIN_INVLEN);
            myBinIndex.z = (int) floorf((8 * zRegionIndex + 4) * h * BIN_INVLEN);

            /* first neighbor in list for me to cache */
            nbrid = (tid >> 4);

            numbins = BIN_CACHE_MAXLEN;

            float energy0 = 0.f;
            float energy1 = 0.f;
            float energy2 = 0.f;
            float energy3 = 0.f;


            for (totalbins = 0;  totalbins < NbrListLen;  totalbins += numbins) {
                int bincnt;

                /* start of where to write in shared memory */
                int startoff = BIN_SIZE * (tid >> 4);

                /* each half-warp to cache up to 4 atom bins */
                for (bincnt = 0;  bincnt < 4 && nbrid < NbrListLen;  bincnt++, nbrid += 8) {
                    int i = myBinIndex.x + NbrList[nbrid].x;
                    int j = myBinIndex.y + NbrList[nbrid].y;
                    int k = myBinIndex.z + NbrList[nbrid].z;

                    /* determine global memory location of atom bin */
                    float *p_global = ((float *) binZeroAddr) + (((__mul24(k, binDim_y) + j)*binDim_x + i) << BIN_SHIFT);

                    /* coalesced read from global memory -
                    * retain same ordering in shared memory for now */
                    int binIndex = startoff + (bincnt << (3 + BIN_SHIFT));
                    int tidmask = tid & 15;

                    AtomBinCache[binIndex + tidmask   ] = p_global[tidmask   ];
                    AtomBinCache[binIndex + tidmask+16] = p_global[tidmask+16];
                }
                // __syncthreads();
                asm volatile("bar.sync %0, %1;" : : "r"(2), "r"(128) : "memory");

                /* no warp divergence */
                if (totalbins + BIN_CACHE_MAXLEN > NbrListLen) {
                    numbins = NbrListLen - totalbins;
                }

                int stopbin = (numbins << BIN_SHIFT);
                for (bincnt = 0; bincnt < stopbin; bincnt+=BIN_SIZE) {
                    for (int i = 0;  i < BIN_DEPTH;  i++) {
                        int off = bincnt + (i<<2);

                        float aq = AtomBinCache[off + 3];
                        if (0.f == aq) 
                            break;  /* no more atoms in bin */

                        float dx = AtomBinCache[off    ] - x;
                        float dz = AtomBinCache[off + 2] - z;
                        float dxdz2 = dx*dx + dz*dz;
                        float dy = AtomBinCache[off + 1] - y;
                        float r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy0 += aq * rsqrtf(r2) * s * s;
                        }
                        dy -= 2.0f*h;
                        r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy1 += aq * rsqrtf(r2) * s * s;
                        }
                        dy -= 2.0f*h;
                        r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy2 += aq * rsqrtf(r2) * s * s;
                        }
                        dy -= 2.0f*h;
                        r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy3 += aq * rsqrtf(r2) * s * s;
                        }
                    } /* end loop over atoms in bin */
                } /* end loop over cached atom bins */
                // __syncthreads();
                asm volatile("bar.sync %0, %1;" : : "r"(2), "r"(128) : "memory");
            } /* end loop over neighbor list */

            /* store into global memory */
            myRegionAddr[(tid>>4)*64 + (tid&15)     ] = energy0;
            myRegionAddr[(tid>>4)*64 + (tid&15) + 16] = energy1;
            myRegionAddr[(tid>>4)*64 + (tid&15) + 32] = energy2;
            myRegionAddr[(tid>>4)*64 + (tid&15) + 48] = energy3;
        }
    }
}


__device__ void mix_cutcp1(
    int binDim_x,
    int binDim_y,
    float4 *binZeroAddr,    /* address of atom bins starting at origin */
    float h,                /* lattice spacing */
    float cutoff2,          /* square of cutoff distance */
    float inv_cutoff2,
    float *regionZeroAddr, /* address of lattice regions starting at origin */
    int zRegionIndex_t,
    int grid_dimension_x,
    int grid_dimension_y,
    int grid_dimension_z,
    int block_dimension_x,
    int block_dimension_y,
    int block_dimension_z,
    int thread_step,
    int iteration
    ) {
    unsigned int block_pos = blockIdx.x + 68 * 2;

    int thread_id_x = (threadIdx.x - thread_step) / (block_dimension_y * block_dimension_z);
    int thread_id_y = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) / block_dimension_z;
    int thread_id_z = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) % block_dimension_z;

    /* thread id */
	const int tid = (thread_id_z * block_dimension_y + thread_id_y) * block_dimension_x + thread_id_x;

    __shared__ float AtomBinCache[BIN_CACHE_MAXLEN * BIN_DEPTH * 4];
	// __shared__ float *myRegionAddr;
	// __shared__ int3 myBinIndex;

    float *myRegionAddr;
	int3 myBinIndex;

    for (;; block_pos += SM_NUM * 4) {
        if (block_pos >= grid_dimension_x * grid_dimension_y * grid_dimension_z) {
            return;
        }

        int block_id_x = block_pos / (grid_dimension_y * grid_dimension_z);
        int block_id_y = (block_pos % (grid_dimension_y * grid_dimension_z)) / grid_dimension_z;
        int block_id_z = (block_pos % (grid_dimension_y * grid_dimension_z)) % grid_dimension_z;

        for (int loop = 0; loop < iteration; loop++) {

            int xRegionIndex = block_id_x;
            int yRegionIndex = block_id_y;
            int zRegionIndex = block_id_z;
        
            /* neighbor index */
            int nbrid;

            /* this is the start of the sub-region indexed by tid */
            myRegionAddr = regionZeroAddr + ((zRegionIndex*grid_dimension_y + yRegionIndex)*grid_dimension_x + xRegionIndex)*REGION_SIZE;

            /* spatial coordinate of this lattice point */
            float x = (8 * xRegionIndex + thread_id_x) * h;
            float y = (8 * yRegionIndex + thread_id_y) * h;
            float z = (8 * zRegionIndex + thread_id_z) * h;

            int totalbins = 0;
            int numbins;

            /* bin number determined by center of region */
            myBinIndex.x = (int) floorf((8 * xRegionIndex + 4) * h * BIN_INVLEN);
            myBinIndex.y = (int) floorf((8 * yRegionIndex + 4) * h * BIN_INVLEN);
            myBinIndex.z = (int) floorf((8 * zRegionIndex + 4) * h * BIN_INVLEN);

            /* first neighbor in list for me to cache */
            nbrid = (tid >> 4);

            numbins = BIN_CACHE_MAXLEN;

            float energy0 = 0.f;
            float energy1 = 0.f;
            float energy2 = 0.f;
            float energy3 = 0.f;


            for (totalbins = 0;  totalbins < NbrListLen;  totalbins += numbins) {
                int bincnt;

                /* start of where to write in shared memory */
                int startoff = BIN_SIZE * (tid >> 4);

                /* each half-warp to cache up to 4 atom bins */
                for (bincnt = 0;  bincnt < 4 && nbrid < NbrListLen;  bincnt++, nbrid += 8) {
                    int i = myBinIndex.x + NbrList[nbrid].x;
                    int j = myBinIndex.y + NbrList[nbrid].y;
                    int k = myBinIndex.z + NbrList[nbrid].z;

                    /* determine global memory location of atom bin */
                    float *p_global = ((float *) binZeroAddr) + (((__mul24(k, binDim_y) + j)*binDim_x + i) << BIN_SHIFT);

                    /* coalesced read from global memory -
                    * retain same ordering in shared memory for now */
                    int binIndex = startoff + (bincnt << (3 + BIN_SHIFT));
                    int tidmask = tid & 15;

                    AtomBinCache[binIndex + tidmask   ] = p_global[tidmask   ];
                    AtomBinCache[binIndex + tidmask+16] = p_global[tidmask+16];
                }
            //    __syncthreads();
                asm volatile("bar.sync %0, %1;" : : "r"(3), "r"(128) : "memory");

                /* no warp divergence */
                if (totalbins + BIN_CACHE_MAXLEN > NbrListLen) {
                    numbins = NbrListLen - totalbins;
                }

                int stopbin = (numbins << BIN_SHIFT);
                for (bincnt = 0; bincnt < stopbin; bincnt+=BIN_SIZE) {
                    for (int i = 0;  i < BIN_DEPTH;  i++) {
                        int off = bincnt + (i<<2);

                        float aq = AtomBinCache[off + 3];
                        if (0.f == aq) 
                            break;  /* no more atoms in bin */

                        float dx = AtomBinCache[off    ] - x;
                        float dz = AtomBinCache[off + 2] - z;
                        float dxdz2 = dx*dx + dz*dz;
                        float dy = AtomBinCache[off + 1] - y;
                        float r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy0 += aq * rsqrtf(r2) * s * s;
                        }
                        dy -= 2.0f*h;
                        r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy1 += aq * rsqrtf(r2) * s * s;
                        }
                        dy -= 2.0f*h;
                        r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy2 += aq * rsqrtf(r2) * s * s;
                        }
                        dy -= 2.0f*h;
                        r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy3 += aq * rsqrtf(r2) * s * s;
                        }
                    } /* end loop over atoms in bin */
                } /* end loop over cached atom bins */
            //    __syncthreads();
                asm volatile("bar.sync %0, %1;" : : "r"(3), "r"(128) : "memory");
            } /* end loop over neighbor list */

            /* store into global memory */
            myRegionAddr[(tid>>4)*64 + (tid&15)     ] = energy0;
            myRegionAddr[(tid>>4)*64 + (tid&15) + 16] = energy1;
            myRegionAddr[(tid>>4)*64 + (tid&15) + 32] = energy2;
            myRegionAddr[(tid>>4)*64 + (tid&15) + 48] = energy3;
        }
    }
}


__device__ void mix_cutcp2(
    int binDim_x,
    int binDim_y,
    float4 *binZeroAddr,    /* address of atom bins starting at origin */
    float h,                /* lattice spacing */
    float cutoff2,          /* square of cutoff distance */
    float inv_cutoff2,
    float *regionZeroAddr, /* address of lattice regions starting at origin */
    int zRegionIndex_t,
    int grid_dimension_x,
    int grid_dimension_y,
    int grid_dimension_z,
    int block_dimension_x,
    int block_dimension_y,
    int block_dimension_z,
    int thread_step,
    int iteration
    ) {
    unsigned int block_pos = blockIdx.x + SM_NUM * 2;
    int thread_id_x = (threadIdx.x - thread_step) / (block_dimension_y * block_dimension_z);
    int thread_id_y = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) / block_dimension_z;
    int thread_id_z = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) % block_dimension_z;

    /* thread id */
	const int tid = (thread_id_z * block_dimension_y + thread_id_y) * block_dimension_x + thread_id_x;

    __shared__ float AtomBinCache[BIN_CACHE_MAXLEN * BIN_DEPTH * 4];
	// __shared__ float *myRegionAddr;
	// __shared__ int3 myBinIndex;

    float *myRegionAddr;
	int3 myBinIndex;

    for (;; block_pos += SM_NUM * 3) {
        if (block_pos >= grid_dimension_x * grid_dimension_y * grid_dimension_z) {
            return;
        }

        int block_id_x = block_pos / (grid_dimension_y * grid_dimension_z);
        int block_id_y = (block_pos % (grid_dimension_y * grid_dimension_z)) / grid_dimension_z;
        int block_id_z = (block_pos % (grid_dimension_y * grid_dimension_z)) % grid_dimension_z;

        for (int loop = 0; loop < iteration; loop++) {

            int xRegionIndex = block_id_x;
            int yRegionIndex = block_id_y;
            int zRegionIndex = block_id_z;
        
            /* neighbor index */
            int nbrid;

            /* this is the start of the sub-region indexed by tid */
            myRegionAddr = regionZeroAddr + ((zRegionIndex*grid_dimension_y + yRegionIndex)*grid_dimension_x + xRegionIndex)*REGION_SIZE;

            /* spatial coordinate of this lattice point */
            float x = (8 * xRegionIndex + thread_id_x) * h;
            float y = (8 * yRegionIndex + thread_id_y) * h;
            float z = (8 * zRegionIndex + thread_id_z) * h;

            int totalbins = 0;
            int numbins;

            /* bin number determined by center of region */
            myBinIndex.x = (int) floorf((8 * xRegionIndex + 4) * h * BIN_INVLEN);
            myBinIndex.y = (int) floorf((8 * yRegionIndex + 4) * h * BIN_INVLEN);
            myBinIndex.z = (int) floorf((8 * zRegionIndex + 4) * h * BIN_INVLEN);

            /* first neighbor in list for me to cache */
            nbrid = (tid >> 4);

            numbins = BIN_CACHE_MAXLEN;

            float energy0 = 0.f;
            float energy1 = 0.f;
            float energy2 = 0.f;
            float energy3 = 0.f;


            for (totalbins = 0;  totalbins < NbrListLen;  totalbins += numbins) {
                int bincnt;

                /* start of where to write in shared memory */
                int startoff = BIN_SIZE * (tid >> 4);

                /* each half-warp to cache up to 4 atom bins */
                for (bincnt = 0;  bincnt < 4 && nbrid < NbrListLen;  bincnt++, nbrid += 8) {
                    int i = myBinIndex.x + NbrList[nbrid].x;
                    int j = myBinIndex.y + NbrList[nbrid].y;
                    int k = myBinIndex.z + NbrList[nbrid].z;

                    /* determine global memory location of atom bin */
                    float *p_global = ((float *) binZeroAddr) + (((__mul24(k, binDim_y) + j)*binDim_x + i) << BIN_SHIFT);

                    /* coalesced read from global memory -
                    * retain same ordering in shared memory for now */
                    int binIndex = startoff + (bincnt << (3 + BIN_SHIFT));
                    int tidmask = tid & 15;

                    AtomBinCache[binIndex + tidmask   ] = p_global[tidmask   ];
                    AtomBinCache[binIndex + tidmask+16] = p_global[tidmask+16];
                }
            //    __syncthreads();
                asm volatile("bar.sync %0, %1;" : : "r"(4), "r"(128) : "memory");

                /* no warp divergence */
                if (totalbins + BIN_CACHE_MAXLEN > NbrListLen) {
                    numbins = NbrListLen - totalbins;
                }

                int stopbin = (numbins << BIN_SHIFT);
                for (bincnt = 0; bincnt < stopbin; bincnt+=BIN_SIZE) {
                    for (int i = 0;  i < BIN_DEPTH;  i++) {
                        int off = bincnt + (i<<2);

                        float aq = AtomBinCache[off + 3];
                        if (0.f == aq) 
                            break;  /* no more atoms in bin */

                        float dx = AtomBinCache[off    ] - x;
                        float dz = AtomBinCache[off + 2] - z;
                        float dxdz2 = dx*dx + dz*dz;
                        float dy = AtomBinCache[off + 1] - y;
                        float r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy0 += aq * rsqrtf(r2) * s * s;
                        }
                        dy -= 2.0f*h;
                        r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy1 += aq * rsqrtf(r2) * s * s;
                        }
                        dy -= 2.0f*h;
                        r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy2 += aq * rsqrtf(r2) * s * s;
                        }
                        dy -= 2.0f*h;
                        r2 = dy*dy + dxdz2;

                        if (r2 < cutoff2) {
                            float s = (1.f - r2 * inv_cutoff2);
                            energy3 += aq * rsqrtf(r2) * s * s;
                        }
                    } /* end loop over atoms in bin */
                } /* end loop over cached atom bins */
            //    __syncthreads();
                asm volatile("bar.sync %0, %1;" : : "r"(4), "r"(128) : "memory");
            } /* end loop over neighbor list */

            /* store into global memory */
            myRegionAddr[(tid>>4)*64 + (tid&15)     ] = energy0;
            myRegionAddr[(tid>>4)*64 + (tid&15) + 16] = energy1;
            myRegionAddr[(tid>>4)*64 + (tid&15) + 32] = energy2;
            myRegionAddr[(tid>>4)*64 + (tid&15) + 48] = energy3;
        }
    }
}


__device__ void general_ptb_cutcp0(
    int binDim_x,
    int binDim_y,
    float4 *binZeroAddr,    /* address of atom bins starting at origin */
    float h,                /* lattice spacing */
    float cutoff2,          /* square of cutoff distance */
    float inv_cutoff2,
    float *regionZeroAddr, /* address of lattice regions starting at origin */
    int zRegionIndex_t,
    int grid_dimension_x,
    int grid_dimension_y,
    int grid_dimension_z,
    int block_dimension_x,
    int block_dimension_y,
    int block_dimension_z,
    int ptb_start_block_pos,
    int ptb_iter_block_step,
    int ptb_end_block_pos,
    int thread_base
    ) {
    // unsigned int block_pos = blockIdx.x + SM_NUM * 2; // TODO: why SM_NUM * 2?
    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) / (block_dimension_y * block_dimension_z);
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_z) % block_dimension_y;
    int thread_id_z = (threadIdx.x - thread_base) % block_dimension_z;

    // // ori
    // int thread_id_x = (threadIdx.x - thread_step) / (block_dimension_y * block_dimension_z);
    // int thread_id_y = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) / block_dimension_z;
    // int thread_id_z = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) % block_dimension_z;

    /* thread id */
	const int tid = (thread_id_z * block_dimension_y + thread_id_y) * block_dimension_x + thread_id_x;

    __shared__ float AtomBinCache[BIN_CACHE_MAXLEN * BIN_DEPTH * 4];
	// __shared__ float *myRegionAddr;
	// __shared__ int3 myBinIndex;

    float *myRegionAddr;
	int3 myBinIndex;

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos / (grid_dimension_y * grid_dimension_z);
        int block_id_y = (block_pos / grid_dimension_z) % grid_dimension_y;
        int block_id_z = block_pos % grid_dimension_z;


        int xRegionIndex = block_id_x;
        int yRegionIndex = block_id_y;
        int zRegionIndex = block_id_z;
    
        /* neighbor index */
        int nbrid;

        /* this is the start of the sub-region indexed by tid */
        myRegionAddr = regionZeroAddr + ((zRegionIndex*grid_dimension_y + yRegionIndex)*grid_dimension_x + xRegionIndex)*REGION_SIZE;

        /* spatial coordinate of this lattice point */
        float x = (8 * xRegionIndex + thread_id_x) * h;
        float y = (8 * yRegionIndex + thread_id_y) * h;
        float z = (8 * zRegionIndex + thread_id_z) * h;

        int totalbins = 0;
        int numbins;

        /* bin number determined by center of region */
        myBinIndex.x = (int) floorf((8 * xRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.y = (int) floorf((8 * yRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.z = (int) floorf((8 * zRegionIndex + 4) * h * BIN_INVLEN);

        /* first neighbor in list for me to cache */
        nbrid = (tid >> 4);

        numbins = BIN_CACHE_MAXLEN;

        float energy0 = 0.f;
        float energy1 = 0.f;
        float energy2 = 0.f;
        float energy3 = 0.f;


        for (totalbins = 0;  totalbins < NbrListLen;  totalbins += numbins) {
            int bincnt;

            /* start of where to write in shared memory */
            int startoff = BIN_SIZE * (tid >> 4);

            /* each half-warp to cache up to 4 atom bins */
            for (bincnt = 0;  bincnt < 4 && nbrid < NbrListLen;  bincnt++, nbrid += 8) {
                int i = myBinIndex.x + NbrList[nbrid].x;
                int j = myBinIndex.y + NbrList[nbrid].y;
                int k = myBinIndex.z + NbrList[nbrid].z;

                /* determine global memory location of atom bin */
                float *p_global = ((float *) binZeroAddr) + (((__mul24(k, binDim_y) + j)*binDim_x + i) << BIN_SHIFT);

                /* coalesced read from global memory -
                * retain same ordering in shared memory for now */
                int binIndex = startoff + (bincnt << (3 + BIN_SHIFT));
                int tidmask = tid & 15;

                AtomBinCache[binIndex + tidmask   ] = p_global[tidmask   ];
                AtomBinCache[binIndex + tidmask+16] = p_global[tidmask+16];
            }
        //    __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(1), "r"(128) : "memory");

            /* no warp divergence */
            if (totalbins + BIN_CACHE_MAXLEN > NbrListLen) {
                numbins = NbrListLen - totalbins;
            }

            int stopbin = (numbins << BIN_SHIFT);
            for (bincnt = 0; bincnt < stopbin; bincnt+=BIN_SIZE) {
                for (int i = 0;  i < BIN_DEPTH;  i++) {
                    int off = bincnt + (i<<2);

                    float aq = AtomBinCache[off + 3];
                    if (0.f == aq) 
                        break;  /* no more atoms in bin */

                    float dx = AtomBinCache[off    ] - x;
                    float dz = AtomBinCache[off + 2] - z;
                    float dxdz2 = dx*dx + dz*dz;
                    float dy = AtomBinCache[off + 1] - y;
                    float r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy0 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy1 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy2 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy3 += aq * rsqrtf(r2) * s * s;
                    }
                } /* end loop over atoms in bin */
            } /* end loop over cached atom bins */
        //    __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(1), "r"(128) : "memory");
        } /* end loop over neighbor list */

        /* store into global memory */
        myRegionAddr[(tid>>4)*64 + (tid&15)     ] = energy0;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 16] = energy1;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 32] = energy2;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 48] = energy3;
    }
}

__device__ void general_ptb_cutcp1(
    int binDim_x,
    int binDim_y,
    float4 *binZeroAddr,    /* address of atom bins starting at origin */
    float h,                /* lattice spacing */
    float cutoff2,          /* square of cutoff distance */
    float inv_cutoff2,
    float *regionZeroAddr, /* address of lattice regions starting at origin */
    int zRegionIndex_t,
    int grid_dimension_x,
    int grid_dimension_y,
    int grid_dimension_z,
    int block_dimension_x,
    int block_dimension_y,
    int block_dimension_z,
    int ptb_start_block_pos,
    int ptb_iter_block_step,
    int ptb_end_block_pos,
    int thread_base
    ) {
    // unsigned int block_pos = blockIdx.x + SM_NUM * 2; // TODO: why SM_NUM * 2?
    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) / (block_dimension_y * block_dimension_z);
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_z) % block_dimension_y;
    int thread_id_z = (threadIdx.x - thread_base) % block_dimension_z;

    // // ori
    // int thread_id_x = (threadIdx.x - thread_step) / (block_dimension_y * block_dimension_z);
    // int thread_id_y = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) / block_dimension_z;
    // int thread_id_z = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) % block_dimension_z;

    /* thread id */
	const int tid = (thread_id_z * block_dimension_y + thread_id_y) * block_dimension_x + thread_id_x;

    __shared__ float AtomBinCache[BIN_CACHE_MAXLEN * BIN_DEPTH * 4];
	// __shared__ float *myRegionAddr;
	// __shared__ int3 myBinIndex;

    float *myRegionAddr;
	int3 myBinIndex;

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos / (grid_dimension_y * grid_dimension_z);
        int block_id_y = (block_pos / grid_dimension_z) % grid_dimension_y;
        int block_id_z = block_pos % grid_dimension_z;


        int xRegionIndex = block_id_x;
        int yRegionIndex = block_id_y;
        int zRegionIndex = block_id_z;
    
        /* neighbor index */
        int nbrid;

        /* this is the start of the sub-region indexed by tid */
        myRegionAddr = regionZeroAddr + ((zRegionIndex*grid_dimension_y + yRegionIndex)*grid_dimension_x + xRegionIndex)*REGION_SIZE;

        /* spatial coordinate of this lattice point */
        float x = (8 * xRegionIndex + thread_id_x) * h;
        float y = (8 * yRegionIndex + thread_id_y) * h;
        float z = (8 * zRegionIndex + thread_id_z) * h;

        int totalbins = 0;
        int numbins;

        /* bin number determined by center of region */
        myBinIndex.x = (int) floorf((8 * xRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.y = (int) floorf((8 * yRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.z = (int) floorf((8 * zRegionIndex + 4) * h * BIN_INVLEN);

        /* first neighbor in list for me to cache */
        nbrid = (tid >> 4);

        numbins = BIN_CACHE_MAXLEN;

        float energy0 = 0.f;
        float energy1 = 0.f;
        float energy2 = 0.f;
        float energy3 = 0.f;


        for (totalbins = 0;  totalbins < NbrListLen;  totalbins += numbins) {
            int bincnt;

            /* start of where to write in shared memory */
            int startoff = BIN_SIZE * (tid >> 4);

            /* each half-warp to cache up to 4 atom bins */
            for (bincnt = 0;  bincnt < 4 && nbrid < NbrListLen;  bincnt++, nbrid += 8) {
                int i = myBinIndex.x + NbrList[nbrid].x;
                int j = myBinIndex.y + NbrList[nbrid].y;
                int k = myBinIndex.z + NbrList[nbrid].z;

                /* determine global memory location of atom bin */
                float *p_global = ((float *) binZeroAddr) + (((__mul24(k, binDim_y) + j)*binDim_x + i) << BIN_SHIFT);

                /* coalesced read from global memory -
                * retain same ordering in shared memory for now */
                int binIndex = startoff + (bincnt << (3 + BIN_SHIFT));
                int tidmask = tid & 15;

                AtomBinCache[binIndex + tidmask   ] = p_global[tidmask   ];
                AtomBinCache[binIndex + tidmask+16] = p_global[tidmask+16];
            }
        //    __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(2), "r"(128) : "memory");

            /* no warp divergence */
            if (totalbins + BIN_CACHE_MAXLEN > NbrListLen) {
                numbins = NbrListLen - totalbins;
            }

            int stopbin = (numbins << BIN_SHIFT);
            for (bincnt = 0; bincnt < stopbin; bincnt+=BIN_SIZE) {
                for (int i = 0;  i < BIN_DEPTH;  i++) {
                    int off = bincnt + (i<<2);

                    float aq = AtomBinCache[off + 3];
                    if (0.f == aq) 
                        break;  /* no more atoms in bin */

                    float dx = AtomBinCache[off    ] - x;
                    float dz = AtomBinCache[off + 2] - z;
                    float dxdz2 = dx*dx + dz*dz;
                    float dy = AtomBinCache[off + 1] - y;
                    float r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy0 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy1 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy2 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy3 += aq * rsqrtf(r2) * s * s;
                    }
                } /* end loop over atoms in bin */
            } /* end loop over cached atom bins */
        //    __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(2), "r"(128) : "memory");
        } /* end loop over neighbor list */

        /* store into global memory */
        myRegionAddr[(tid>>4)*64 + (tid&15)     ] = energy0;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 16] = energy1;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 32] = energy2;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 48] = energy3;
    }
}

__device__ void general_ptb_cutcp2(
    int binDim_x,
    int binDim_y,
    float4 *binZeroAddr,    /* address of atom bins starting at origin */
    float h,                /* lattice spacing */
    float cutoff2,          /* square of cutoff distance */
    float inv_cutoff2,
    float *regionZeroAddr, /* address of lattice regions starting at origin */
    int zRegionIndex_t,
    int grid_dimension_x,
    int grid_dimension_y,
    int grid_dimension_z,
    int block_dimension_x,
    int block_dimension_y,
    int block_dimension_z,
    int ptb_start_block_pos,
    int ptb_iter_block_step,
    int ptb_end_block_pos,
    int thread_base
    ) {
    // unsigned int block_pos = blockIdx.x + SM_NUM * 2; // TODO: why SM_NUM * 2?
    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) / (block_dimension_y * block_dimension_z);
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_z) % block_dimension_y;
    int thread_id_z = (threadIdx.x - thread_base) % block_dimension_z;

    // // ori
    // int thread_id_x = (threadIdx.x - thread_step) / (block_dimension_y * block_dimension_z);
    // int thread_id_y = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) / block_dimension_z;
    // int thread_id_z = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) % block_dimension_z;

    /* thread id */
	const int tid = (thread_id_z * block_dimension_y + thread_id_y) * block_dimension_x + thread_id_x;

    __shared__ float AtomBinCache[BIN_CACHE_MAXLEN * BIN_DEPTH * 4];
	// __shared__ float *myRegionAddr;
	// __shared__ int3 myBinIndex;

    float *myRegionAddr;
	int3 myBinIndex;

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos / (grid_dimension_y * grid_dimension_z);
        int block_id_y = (block_pos / grid_dimension_z) % grid_dimension_y;
        int block_id_z = block_pos % grid_dimension_z;


        int xRegionIndex = block_id_x;
        int yRegionIndex = block_id_y;
        int zRegionIndex = block_id_z;
    
        /* neighbor index */
        int nbrid;

        /* this is the start of the sub-region indexed by tid */
        myRegionAddr = regionZeroAddr + ((zRegionIndex*grid_dimension_y + yRegionIndex)*grid_dimension_x + xRegionIndex)*REGION_SIZE;

        /* spatial coordinate of this lattice point */
        float x = (8 * xRegionIndex + thread_id_x) * h;
        float y = (8 * yRegionIndex + thread_id_y) * h;
        float z = (8 * zRegionIndex + thread_id_z) * h;

        int totalbins = 0;
        int numbins;

        /* bin number determined by center of region */
        myBinIndex.x = (int) floorf((8 * xRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.y = (int) floorf((8 * yRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.z = (int) floorf((8 * zRegionIndex + 4) * h * BIN_INVLEN);

        /* first neighbor in list for me to cache */
        nbrid = (tid >> 4);

        numbins = BIN_CACHE_MAXLEN;

        float energy0 = 0.f;
        float energy1 = 0.f;
        float energy2 = 0.f;
        float energy3 = 0.f;


        for (totalbins = 0;  totalbins < NbrListLen;  totalbins += numbins) {
            int bincnt;

            /* start of where to write in shared memory */
            int startoff = BIN_SIZE * (tid >> 4);

            /* each half-warp to cache up to 4 atom bins */
            for (bincnt = 0;  bincnt < 4 && nbrid < NbrListLen;  bincnt++, nbrid += 8) {
                int i = myBinIndex.x + NbrList[nbrid].x;
                int j = myBinIndex.y + NbrList[nbrid].y;
                int k = myBinIndex.z + NbrList[nbrid].z;

                /* determine global memory location of atom bin */
                float *p_global = ((float *) binZeroAddr) + (((__mul24(k, binDim_y) + j)*binDim_x + i) << BIN_SHIFT);

                /* coalesced read from global memory -
                * retain same ordering in shared memory for now */
                int binIndex = startoff + (bincnt << (3 + BIN_SHIFT));
                int tidmask = tid & 15;

                AtomBinCache[binIndex + tidmask   ] = p_global[tidmask   ];
                AtomBinCache[binIndex + tidmask+16] = p_global[tidmask+16];
            }
        //    __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(3), "r"(128) : "memory");

            /* no warp divergence */
            if (totalbins + BIN_CACHE_MAXLEN > NbrListLen) {
                numbins = NbrListLen - totalbins;
            }

            int stopbin = (numbins << BIN_SHIFT);
            for (bincnt = 0; bincnt < stopbin; bincnt+=BIN_SIZE) {
                for (int i = 0;  i < BIN_DEPTH;  i++) {
                    int off = bincnt + (i<<2);

                    float aq = AtomBinCache[off + 3];
                    if (0.f == aq) 
                        break;  /* no more atoms in bin */

                    float dx = AtomBinCache[off    ] - x;
                    float dz = AtomBinCache[off + 2] - z;
                    float dxdz2 = dx*dx + dz*dz;
                    float dy = AtomBinCache[off + 1] - y;
                    float r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy0 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy1 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy2 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy3 += aq * rsqrtf(r2) * s * s;
                    }
                } /* end loop over atoms in bin */
            } /* end loop over cached atom bins */
        //    __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(3), "r"(128) : "memory");
        } /* end loop over neighbor list */

        /* store into global memory */
        myRegionAddr[(tid>>4)*64 + (tid&15)     ] = energy0;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 16] = energy1;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 32] = energy2;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 48] = energy3;
    }
}

__device__ void general_ptb_cutcp3(
    int binDim_x,
    int binDim_y,
    float4 *binZeroAddr,    /* address of atom bins starting at origin */
    float h,                /* lattice spacing */
    float cutoff2,          /* square of cutoff distance */
    float inv_cutoff2,
    float *regionZeroAddr, /* address of lattice regions starting at origin */
    int zRegionIndex_t,
    int grid_dimension_x,
    int grid_dimension_y,
    int grid_dimension_z,
    int block_dimension_x,
    int block_dimension_y,
    int block_dimension_z,
    int ptb_start_block_pos,
    int ptb_iter_block_step,
    int ptb_end_block_pos,
    int thread_base
    ) {
    // unsigned int block_pos = blockIdx.x + SM_NUM * 2; // TODO: why SM_NUM * 2?
    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) / (block_dimension_y * block_dimension_z);
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_z) % block_dimension_y;
    int thread_id_z = (threadIdx.x - thread_base) % block_dimension_z;

    // // ori
    // int thread_id_x = (threadIdx.x - thread_step) / (block_dimension_y * block_dimension_z);
    // int thread_id_y = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) / block_dimension_z;
    // int thread_id_z = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) % block_dimension_z;

    /* thread id */
	const int tid = (thread_id_z * block_dimension_y + thread_id_y) * block_dimension_x + thread_id_x;

    __shared__ float AtomBinCache[BIN_CACHE_MAXLEN * BIN_DEPTH * 4];
	// __shared__ float *myRegionAddr;
	// __shared__ int3 myBinIndex;

    float *myRegionAddr;
	int3 myBinIndex;

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos / (grid_dimension_y * grid_dimension_z);
        int block_id_y = (block_pos / grid_dimension_z) % grid_dimension_y;
        int block_id_z = block_pos % grid_dimension_z;


        int xRegionIndex = block_id_x;
        int yRegionIndex = block_id_y;
        int zRegionIndex = block_id_z;
    
        /* neighbor index */
        int nbrid;

        /* this is the start of the sub-region indexed by tid */
        myRegionAddr = regionZeroAddr + ((zRegionIndex*grid_dimension_y + yRegionIndex)*grid_dimension_x + xRegionIndex)*REGION_SIZE;

        /* spatial coordinate of this lattice point */
        float x = (8 * xRegionIndex + thread_id_x) * h;
        float y = (8 * yRegionIndex + thread_id_y) * h;
        float z = (8 * zRegionIndex + thread_id_z) * h;

        int totalbins = 0;
        int numbins;

        /* bin number determined by center of region */
        myBinIndex.x = (int) floorf((8 * xRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.y = (int) floorf((8 * yRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.z = (int) floorf((8 * zRegionIndex + 4) * h * BIN_INVLEN);

        /* first neighbor in list for me to cache */
        nbrid = (tid >> 4);

        numbins = BIN_CACHE_MAXLEN;

        float energy0 = 0.f;
        float energy1 = 0.f;
        float energy2 = 0.f;
        float energy3 = 0.f;


        for (totalbins = 0;  totalbins < NbrListLen;  totalbins += numbins) {
            int bincnt;

            /* start of where to write in shared memory */
            int startoff = BIN_SIZE * (tid >> 4);

            /* each half-warp to cache up to 4 atom bins */
            for (bincnt = 0;  bincnt < 4 && nbrid < NbrListLen;  bincnt++, nbrid += 8) {
                int i = myBinIndex.x + NbrList[nbrid].x;
                int j = myBinIndex.y + NbrList[nbrid].y;
                int k = myBinIndex.z + NbrList[nbrid].z;

                /* determine global memory location of atom bin */
                float *p_global = ((float *) binZeroAddr) + (((__mul24(k, binDim_y) + j)*binDim_x + i) << BIN_SHIFT);

                /* coalesced read from global memory -
                * retain same ordering in shared memory for now */
                int binIndex = startoff + (bincnt << (3 + BIN_SHIFT));
                int tidmask = tid & 15;

                AtomBinCache[binIndex + tidmask   ] = p_global[tidmask   ];
                AtomBinCache[binIndex + tidmask+16] = p_global[tidmask+16];
            }
        //    __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(4), "r"(128) : "memory");

            /* no warp divergence */
            if (totalbins + BIN_CACHE_MAXLEN > NbrListLen) {
                numbins = NbrListLen - totalbins;
            }

            int stopbin = (numbins << BIN_SHIFT);
            for (bincnt = 0; bincnt < stopbin; bincnt+=BIN_SIZE) {
                for (int i = 0;  i < BIN_DEPTH;  i++) {
                    int off = bincnt + (i<<2);

                    float aq = AtomBinCache[off + 3];
                    if (0.f == aq) 
                        break;  /* no more atoms in bin */

                    float dx = AtomBinCache[off    ] - x;
                    float dz = AtomBinCache[off + 2] - z;
                    float dxdz2 = dx*dx + dz*dz;
                    float dy = AtomBinCache[off + 1] - y;
                    float r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy0 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy1 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy2 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy3 += aq * rsqrtf(r2) * s * s;
                    }
                } /* end loop over atoms in bin */
            } /* end loop over cached atom bins */
        //    __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(4), "r"(128) : "memory");
        } /* end loop over neighbor list */

        /* store into global memory */
        myRegionAddr[(tid>>4)*64 + (tid&15)     ] = energy0;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 16] = energy1;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 32] = energy2;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 48] = energy3;
    }
}

__device__ void general_ptb_cutcp4(
    int binDim_x,
    int binDim_y,
    float4 *binZeroAddr,    /* address of atom bins starting at origin */
    float h,                /* lattice spacing */
    float cutoff2,          /* square of cutoff distance */
    float inv_cutoff2,
    float *regionZeroAddr, /* address of lattice regions starting at origin */
    int zRegionIndex_t,
    int grid_dimension_x,
    int grid_dimension_y,
    int grid_dimension_z,
    int block_dimension_x,
    int block_dimension_y,
    int block_dimension_z,
    int ptb_start_block_pos,
    int ptb_iter_block_step,
    int ptb_end_block_pos,
    int thread_base
    ) {
    // unsigned int block_pos = blockIdx.x + SM_NUM * 2; // TODO: why SM_NUM * 2?
    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) / (block_dimension_y * block_dimension_z);
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_z) % block_dimension_y;
    int thread_id_z = (threadIdx.x - thread_base) % block_dimension_z;

    // // ori
    // int thread_id_x = (threadIdx.x - thread_step) / (block_dimension_y * block_dimension_z);
    // int thread_id_y = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) / block_dimension_z;
    // int thread_id_z = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) % block_dimension_z;

    /* thread id */
	const int tid = (thread_id_z * block_dimension_y + thread_id_y) * block_dimension_x + thread_id_x;

    __shared__ float AtomBinCache[BIN_CACHE_MAXLEN * BIN_DEPTH * 4];
	// __shared__ float *myRegionAddr;
	// __shared__ int3 myBinIndex;

    float *myRegionAddr;
	int3 myBinIndex;

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos / (grid_dimension_y * grid_dimension_z);
        int block_id_y = (block_pos / grid_dimension_z) % grid_dimension_y;
        int block_id_z = block_pos % grid_dimension_z;


        int xRegionIndex = block_id_x;
        int yRegionIndex = block_id_y;
        int zRegionIndex = block_id_z;
    
        /* neighbor index */
        int nbrid;

        /* this is the start of the sub-region indexed by tid */
        myRegionAddr = regionZeroAddr + ((zRegionIndex*grid_dimension_y + yRegionIndex)*grid_dimension_x + xRegionIndex)*REGION_SIZE;

        /* spatial coordinate of this lattice point */
        float x = (8 * xRegionIndex + thread_id_x) * h;
        float y = (8 * yRegionIndex + thread_id_y) * h;
        float z = (8 * zRegionIndex + thread_id_z) * h;

        int totalbins = 0;
        int numbins;

        /* bin number determined by center of region */
        myBinIndex.x = (int) floorf((8 * xRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.y = (int) floorf((8 * yRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.z = (int) floorf((8 * zRegionIndex + 4) * h * BIN_INVLEN);

        /* first neighbor in list for me to cache */
        nbrid = (tid >> 4);

        numbins = BIN_CACHE_MAXLEN;

        float energy0 = 0.f;
        float energy1 = 0.f;
        float energy2 = 0.f;
        float energy3 = 0.f;


        for (totalbins = 0;  totalbins < NbrListLen;  totalbins += numbins) {
            int bincnt;

            /* start of where to write in shared memory */
            int startoff = BIN_SIZE * (tid >> 4);

            /* each half-warp to cache up to 4 atom bins */
            for (bincnt = 0;  bincnt < 4 && nbrid < NbrListLen;  bincnt++, nbrid += 8) {
                int i = myBinIndex.x + NbrList[nbrid].x;
                int j = myBinIndex.y + NbrList[nbrid].y;
                int k = myBinIndex.z + NbrList[nbrid].z;

                /* determine global memory location of atom bin */
                float *p_global = ((float *) binZeroAddr) + (((__mul24(k, binDim_y) + j)*binDim_x + i) << BIN_SHIFT);

                /* coalesced read from global memory -
                * retain same ordering in shared memory for now */
                int binIndex = startoff + (bincnt << (3 + BIN_SHIFT));
                int tidmask = tid & 15;

                AtomBinCache[binIndex + tidmask   ] = p_global[tidmask   ];
                AtomBinCache[binIndex + tidmask+16] = p_global[tidmask+16];
            }
        //    __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(5), "r"(128) : "memory");

            /* no warp divergence */
            if (totalbins + BIN_CACHE_MAXLEN > NbrListLen) {
                numbins = NbrListLen - totalbins;
            }

            int stopbin = (numbins << BIN_SHIFT);
            for (bincnt = 0; bincnt < stopbin; bincnt+=BIN_SIZE) {
                for (int i = 0;  i < BIN_DEPTH;  i++) {
                    int off = bincnt + (i<<2);

                    float aq = AtomBinCache[off + 3];
                    if (0.f == aq) 
                        break;  /* no more atoms in bin */

                    float dx = AtomBinCache[off    ] - x;
                    float dz = AtomBinCache[off + 2] - z;
                    float dxdz2 = dx*dx + dz*dz;
                    float dy = AtomBinCache[off + 1] - y;
                    float r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy0 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy1 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy2 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy3 += aq * rsqrtf(r2) * s * s;
                    }
                } /* end loop over atoms in bin */
            } /* end loop over cached atom bins */
        //    __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(5), "r"(128) : "memory");
        } /* end loop over neighbor list */

        /* store into global memory */
        myRegionAddr[(tid>>4)*64 + (tid&15)     ] = energy0;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 16] = energy1;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 32] = energy2;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 48] = energy3;
    }
}

__device__ void general_ptb_cutcp5(
    int binDim_x,
    int binDim_y,
    float4 *binZeroAddr,    /* address of atom bins starting at origin */
    float h,                /* lattice spacing */
    float cutoff2,          /* square of cutoff distance */
    float inv_cutoff2,
    float *regionZeroAddr, /* address of lattice regions starting at origin */
    int zRegionIndex_t,
    int grid_dimension_x,
    int grid_dimension_y,
    int grid_dimension_z,
    int block_dimension_x,
    int block_dimension_y,
    int block_dimension_z,
    int ptb_start_block_pos,
    int ptb_iter_block_step,
    int ptb_end_block_pos,
    int thread_base
    ) {
    // unsigned int block_pos = blockIdx.x + SM_NUM * 2; // TODO: why SM_NUM * 2?
    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) / (block_dimension_y * block_dimension_z);
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_z) % block_dimension_y;
    int thread_id_z = (threadIdx.x - thread_base) % block_dimension_z;

    // // ori
    // int thread_id_x = (threadIdx.x - thread_step) / (block_dimension_y * block_dimension_z);
    // int thread_id_y = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) / block_dimension_z;
    // int thread_id_z = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) % block_dimension_z;

    /* thread id */
	const int tid = (thread_id_z * block_dimension_y + thread_id_y) * block_dimension_x + thread_id_x;

    __shared__ float AtomBinCache[BIN_CACHE_MAXLEN * BIN_DEPTH * 4];
	// __shared__ float *myRegionAddr;
	// __shared__ int3 myBinIndex;

    float *myRegionAddr;
	int3 myBinIndex;

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos / (grid_dimension_y * grid_dimension_z);
        int block_id_y = (block_pos / grid_dimension_z) % grid_dimension_y;
        int block_id_z = block_pos % grid_dimension_z;


        int xRegionIndex = block_id_x;
        int yRegionIndex = block_id_y;
        int zRegionIndex = block_id_z;
    
        /* neighbor index */
        int nbrid;

        /* this is the start of the sub-region indexed by tid */
        myRegionAddr = regionZeroAddr + ((zRegionIndex*grid_dimension_y + yRegionIndex)*grid_dimension_x + xRegionIndex)*REGION_SIZE;

        /* spatial coordinate of this lattice point */
        float x = (8 * xRegionIndex + thread_id_x) * h;
        float y = (8 * yRegionIndex + thread_id_y) * h;
        float z = (8 * zRegionIndex + thread_id_z) * h;

        int totalbins = 0;
        int numbins;

        /* bin number determined by center of region */
        myBinIndex.x = (int) floorf((8 * xRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.y = (int) floorf((8 * yRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.z = (int) floorf((8 * zRegionIndex + 4) * h * BIN_INVLEN);

        /* first neighbor in list for me to cache */
        nbrid = (tid >> 4);

        numbins = BIN_CACHE_MAXLEN;

        float energy0 = 0.f;
        float energy1 = 0.f;
        float energy2 = 0.f;
        float energy3 = 0.f;


        for (totalbins = 0;  totalbins < NbrListLen;  totalbins += numbins) {
            int bincnt;

            /* start of where to write in shared memory */
            int startoff = BIN_SIZE * (tid >> 4);

            /* each half-warp to cache up to 4 atom bins */
            for (bincnt = 0;  bincnt < 4 && nbrid < NbrListLen;  bincnt++, nbrid += 8) {
                int i = myBinIndex.x + NbrList[nbrid].x;
                int j = myBinIndex.y + NbrList[nbrid].y;
                int k = myBinIndex.z + NbrList[nbrid].z;

                /* determine global memory location of atom bin */
                float *p_global = ((float *) binZeroAddr) + (((__mul24(k, binDim_y) + j)*binDim_x + i) << BIN_SHIFT);

                /* coalesced read from global memory -
                * retain same ordering in shared memory for now */
                int binIndex = startoff + (bincnt << (3 + BIN_SHIFT));
                int tidmask = tid & 15;

                AtomBinCache[binIndex + tidmask   ] = p_global[tidmask   ];
                AtomBinCache[binIndex + tidmask+16] = p_global[tidmask+16];
            }
        //    __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(6), "r"(128) : "memory");

            /* no warp divergence */
            if (totalbins + BIN_CACHE_MAXLEN > NbrListLen) {
                numbins = NbrListLen - totalbins;
            }

            int stopbin = (numbins << BIN_SHIFT);
            for (bincnt = 0; bincnt < stopbin; bincnt+=BIN_SIZE) {
                for (int i = 0;  i < BIN_DEPTH;  i++) {
                    int off = bincnt + (i<<2);

                    float aq = AtomBinCache[off + 3];
                    if (0.f == aq) 
                        break;  /* no more atoms in bin */

                    float dx = AtomBinCache[off    ] - x;
                    float dz = AtomBinCache[off + 2] - z;
                    float dxdz2 = dx*dx + dz*dz;
                    float dy = AtomBinCache[off + 1] - y;
                    float r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy0 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy1 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy2 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy3 += aq * rsqrtf(r2) * s * s;
                    }
                } /* end loop over atoms in bin */
            } /* end loop over cached atom bins */
        //    __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(6), "r"(128) : "memory");
        } /* end loop over neighbor list */

        /* store into global memory */
        myRegionAddr[(tid>>4)*64 + (tid&15)     ] = energy0;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 16] = energy1;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 32] = energy2;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 48] = energy3;
    }
}

__device__ void general_ptb_cutcp6(
    int binDim_x,
    int binDim_y,
    float4 *binZeroAddr,    /* address of atom bins starting at origin */
    float h,                /* lattice spacing */
    float cutoff2,          /* square of cutoff distance */
    float inv_cutoff2,
    float *regionZeroAddr, /* address of lattice regions starting at origin */
    int zRegionIndex_t,
    int grid_dimension_x,
    int grid_dimension_y,
    int grid_dimension_z,
    int block_dimension_x,
    int block_dimension_y,
    int block_dimension_z,
    int ptb_start_block_pos,
    int ptb_iter_block_step,
    int ptb_end_block_pos,
    int thread_base
    ) {
    // unsigned int block_pos = blockIdx.x + SM_NUM * 2; // TODO: why SM_NUM * 2?
    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) / (block_dimension_y * block_dimension_z);
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_z) % block_dimension_y;
    int thread_id_z = (threadIdx.x - thread_base) % block_dimension_z;

    // // ori
    // int thread_id_x = (threadIdx.x - thread_step) / (block_dimension_y * block_dimension_z);
    // int thread_id_y = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) / block_dimension_z;
    // int thread_id_z = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) % block_dimension_z;

    /* thread id */
	const int tid = (thread_id_z * block_dimension_y + thread_id_y) * block_dimension_x + thread_id_x;

    __shared__ float AtomBinCache[BIN_CACHE_MAXLEN * BIN_DEPTH * 4];
	// __shared__ float *myRegionAddr;
	// __shared__ int3 myBinIndex;

    float *myRegionAddr;
	int3 myBinIndex;

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos / (grid_dimension_y * grid_dimension_z);
        int block_id_y = (block_pos / grid_dimension_z) % grid_dimension_y;
        int block_id_z = block_pos % grid_dimension_z;


        int xRegionIndex = block_id_x;
        int yRegionIndex = block_id_y;
        int zRegionIndex = block_id_z;
    
        /* neighbor index */
        int nbrid;

        /* this is the start of the sub-region indexed by tid */
        myRegionAddr = regionZeroAddr + ((zRegionIndex*grid_dimension_y + yRegionIndex)*grid_dimension_x + xRegionIndex)*REGION_SIZE;

        /* spatial coordinate of this lattice point */
        float x = (8 * xRegionIndex + thread_id_x) * h;
        float y = (8 * yRegionIndex + thread_id_y) * h;
        float z = (8 * zRegionIndex + thread_id_z) * h;

        int totalbins = 0;
        int numbins;

        /* bin number determined by center of region */
        myBinIndex.x = (int) floorf((8 * xRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.y = (int) floorf((8 * yRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.z = (int) floorf((8 * zRegionIndex + 4) * h * BIN_INVLEN);

        /* first neighbor in list for me to cache */
        nbrid = (tid >> 4);

        numbins = BIN_CACHE_MAXLEN;

        float energy0 = 0.f;
        float energy1 = 0.f;
        float energy2 = 0.f;
        float energy3 = 0.f;


        for (totalbins = 0;  totalbins < NbrListLen;  totalbins += numbins) {
            int bincnt;

            /* start of where to write in shared memory */
            int startoff = BIN_SIZE * (tid >> 4);

            /* each half-warp to cache up to 4 atom bins */
            for (bincnt = 0;  bincnt < 4 && nbrid < NbrListLen;  bincnt++, nbrid += 8) {
                int i = myBinIndex.x + NbrList[nbrid].x;
                int j = myBinIndex.y + NbrList[nbrid].y;
                int k = myBinIndex.z + NbrList[nbrid].z;

                /* determine global memory location of atom bin */
                float *p_global = ((float *) binZeroAddr) + (((__mul24(k, binDim_y) + j)*binDim_x + i) << BIN_SHIFT);

                /* coalesced read from global memory -
                * retain same ordering in shared memory for now */
                int binIndex = startoff + (bincnt << (3 + BIN_SHIFT));
                int tidmask = tid & 15;

                AtomBinCache[binIndex + tidmask   ] = p_global[tidmask   ];
                AtomBinCache[binIndex + tidmask+16] = p_global[tidmask+16];
            }
        //    __syncthreads();
            asm volatile("bar.sync %0, %1;" : : "r"(7), "r"(128) : "memory");

            /* no warp divergence */
            if (totalbins + BIN_CACHE_MAXLEN > NbrListLen) {
                numbins = NbrListLen - totalbins;
            }

            int stopbin = (numbins << BIN_SHIFT);
            for (bincnt = 0; bincnt < stopbin; bincnt+=BIN_SIZE) {
                for (int i = 0;  i < BIN_DEPTH;  i++) {
                    int off = bincnt + (i<<2);

                    float aq = AtomBinCache[off + 3];
                    if (0.f == aq) 
                        break;  /* no more atoms in bin */

                    float dx = AtomBinCache[off    ] - x;
                    float dz = AtomBinCache[off + 2] - z;
                    float dxdz2 = dx*dx + dz*dz;
                    float dy = AtomBinCache[off + 1] - y;
                    float r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy0 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy1 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy2 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy3 += aq * rsqrtf(r2) * s * s;
                    }
                } /* end loop over atoms in bin */
            } /* end loop over cached atom bins */
        //    __syncthreads();
            asm volatile("bar.sync %0, %1;" : : "r"(7), "r"(128) : "memory");
        } /* end loop over neighbor list */

        /* store into global memory */
        myRegionAddr[(tid>>4)*64 + (tid&15)     ] = energy0;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 16] = energy1;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 32] = energy2;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 48] = energy3;
    }
}

extern "C" __global__ void g_general_ptb_cutcp(
    int binDim_x,
    int binDim_y,
    float4 *binZeroAddr,    /* address of atom bins starting at origin */
    float h,                /* lattice spacing */
    float cutoff2,          /* square of cutoff distance */
    float inv_cutoff2,
    float *regionZeroAddr, /* address of lattice regions starting at origin */
    int zRegionIndex_t,
    int grid_dimension_x,
    int grid_dimension_y,
    int grid_dimension_z,
    int block_dimension_x,
    int block_dimension_y,
    int block_dimension_z,
    int ptb_start_block_pos,
    int ptb_iter_block_step,
    int ptb_end_block_pos,
    int thread_base
    ) {
    // unsigned int block_pos = blockIdx.x + SM_NUM * 2; // TODO: why SM_NUM * 2?
    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) / (block_dimension_y * block_dimension_z);
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_z) % block_dimension_y;
    int thread_id_z = (threadIdx.x - thread_base) % block_dimension_z;

    // // ori
    // int thread_id_x = (threadIdx.x - thread_step) / (block_dimension_y * block_dimension_z);
    // int thread_id_y = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) / block_dimension_z;
    // int thread_id_z = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) % block_dimension_z;

    /* thread id */
	const int tid = (thread_id_z * block_dimension_y + thread_id_y) * block_dimension_x + thread_id_x;

    __shared__ float AtomBinCache[BIN_CACHE_MAXLEN * BIN_DEPTH * 4];
	// __shared__ float *myRegionAddr;
	// __shared__ int3 myBinIndex;

    float *myRegionAddr;
	int3 myBinIndex;

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos / (grid_dimension_y * grid_dimension_z);
        int block_id_y = (block_pos / grid_dimension_z) % grid_dimension_y;
        int block_id_z = block_pos % grid_dimension_z;


        int xRegionIndex = block_id_x;
        int yRegionIndex = block_id_y;
        int zRegionIndex = block_id_z;
    
        /* neighbor index */
        int nbrid;

        /* this is the start of the sub-region indexed by tid */
        myRegionAddr = regionZeroAddr + ((zRegionIndex*grid_dimension_y + yRegionIndex)*grid_dimension_x + xRegionIndex)*REGION_SIZE;

        /* spatial coordinate of this lattice point */
        float x = (8 * xRegionIndex + thread_id_x) * h;
        float y = (8 * yRegionIndex + thread_id_y) * h;
        float z = (8 * zRegionIndex + thread_id_z) * h;

        int totalbins = 0;
        int numbins;

        /* bin number determined by center of region */
        myBinIndex.x = (int) floorf((8 * xRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.y = (int) floorf((8 * yRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.z = (int) floorf((8 * zRegionIndex + 4) * h * BIN_INVLEN);

        /* first neighbor in list for me to cache */
        nbrid = (tid >> 4);

        numbins = BIN_CACHE_MAXLEN;

        float energy0 = 0.f;
        float energy1 = 0.f;
        float energy2 = 0.f;
        float energy3 = 0.f;


        for (totalbins = 0;  totalbins < NbrListLen;  totalbins += numbins) {
            int bincnt;

            /* start of where to write in shared memory */
            int startoff = BIN_SIZE * (tid >> 4);

            /* each half-warp to cache up to 4 atom bins */
            for (bincnt = 0;  bincnt < 4 && nbrid < NbrListLen;  bincnt++, nbrid += 8) {
                int i = myBinIndex.x + NbrList[nbrid].x;
                int j = myBinIndex.y + NbrList[nbrid].y;
                int k = myBinIndex.z + NbrList[nbrid].z;

                /* determine global memory location of atom bin */
                float *p_global = ((float *) binZeroAddr) + (((__mul24(k, binDim_y) + j)*binDim_x + i) << BIN_SHIFT);

                /* coalesced read from global memory -
                * retain same ordering in shared memory for now */
                int binIndex = startoff + (bincnt << (3 + BIN_SHIFT));
                int tidmask = tid & 15;

                AtomBinCache[binIndex + tidmask   ] = p_global[tidmask   ];
                AtomBinCache[binIndex + tidmask+16] = p_global[tidmask+16];
            }
        //    __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(1), "r"(128) : "memory");

            /* no warp divergence */
            if (totalbins + BIN_CACHE_MAXLEN > NbrListLen) {
                numbins = NbrListLen - totalbins;
            }

            int stopbin = (numbins << BIN_SHIFT);
            for (bincnt = 0; bincnt < stopbin; bincnt+=BIN_SIZE) {
                for (int i = 0;  i < BIN_DEPTH;  i++) {
                    int off = bincnt + (i<<2);

                    float aq = AtomBinCache[off + 3];
                    if (0.f == aq) 
                        break;  /* no more atoms in bin */

                    float dx = AtomBinCache[off    ] - x;
                    float dz = AtomBinCache[off + 2] - z;
                    float dxdz2 = dx*dx + dz*dz;
                    float dy = AtomBinCache[off + 1] - y;
                    float r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy0 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy1 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy2 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy3 += aq * rsqrtf(r2) * s * s;
                    }
                } /* end loop over atoms in bin */
            } /* end loop over cached atom bins */
        //    __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(1), "r"(128) : "memory");
        } /* end loop over neighbor list */

        /* store into global memory */
        myRegionAddr[(tid>>4)*64 + (tid&15)     ] = energy0;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 16] = energy1;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 32] = energy2;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 48] = energy3;
    }
}