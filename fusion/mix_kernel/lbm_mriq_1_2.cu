#include "hip/hip_runtime.h"
// lbm-mriq-1-2
__global__ void mixed_lbm_mriq_kernel_1_2(float* lbm0_srcGrid, float* lbm0_dstGrid, int lbm0_grid_dimension_x, int lbm0_grid_dimension_y, int lbm0_grid_dimension_z, int lbm0_block_dimension_x, int lbm0_block_dimension_y, int lbm0_block_dimension_z, int lbm0_ptb_start_block_pos, int lbm0_ptb_iter_block_step, int lbm0_ptb_end_block_pos, int mriq1_numK, int mriq1_kGlobalIndex, float* mriq1_x, float* mriq1_y, float* mriq1_z, float* mriq1_Qr, float* mriq1_Qi, int mriq1_grid_dimension_x, int mriq1_grid_dimension_y, int mriq1_grid_dimension_z, int mriq1_block_dimension_x, int mriq1_block_dimension_y, int mriq1_block_dimension_z, int mriq1_ptb_start_block_pos, int mriq1_ptb_iter_block_step, int mriq1_ptb_end_block_pos){
    if (threadIdx.x < 128) {
        general_ptb_lbm0(
            lbm0_srcGrid, lbm0_dstGrid, lbm0_grid_dimension_x, lbm0_grid_dimension_y, lbm0_grid_dimension_z, lbm0_block_dimension_x, lbm0_block_dimension_y, lbm0_block_dimension_z, lbm0_ptb_start_block_pos + 0 * lbm0_ptb_iter_block_step, lbm0_ptb_iter_block_step * 1, lbm0_ptb_end_block_pos, 0
        );
    }
    else if (threadIdx.x < 384) {
        general_ptb_mriq0(
            mriq1_numK, mriq1_kGlobalIndex, mriq1_x, mriq1_y, mriq1_z, mriq1_Qr, mriq1_Qi, mriq1_grid_dimension_x, mriq1_grid_dimension_y, mriq1_grid_dimension_z, mriq1_block_dimension_x, mriq1_block_dimension_y, mriq1_block_dimension_z, mriq1_ptb_start_block_pos + 0 * mriq1_ptb_iter_block_step, mriq1_ptb_iter_block_step * 2, mriq1_ptb_end_block_pos, 128
        );
    }
    else if (threadIdx.x < 640) {
        general_ptb_mriq1(
            mriq1_numK, mriq1_kGlobalIndex, mriq1_x, mriq1_y, mriq1_z, mriq1_Qr, mriq1_Qi, mriq1_grid_dimension_x, mriq1_grid_dimension_y, mriq1_grid_dimension_z, mriq1_block_dimension_x, mriq1_block_dimension_y, mriq1_block_dimension_z, mriq1_ptb_start_block_pos + 1 * mriq1_ptb_iter_block_step, mriq1_ptb_iter_block_step * 2, mriq1_ptb_end_block_pos, 384
        );
    }

}
