#include "hip/hip_runtime.h"
// cp-sgemm-3-1
__global__ void mixed_cp_sgemm_kernel_3_1(int cp0_numatoms, float cp0_gridspacing, float* cp0_energygrid, int cp0_grid_dimension_x, int cp0_grid_dimension_y, int cp0_grid_dimension_z, int cp0_block_dimension_x, int cp0_block_dimension_y, int cp0_block_dimension_z, int cp0_ptb_start_block_pos, int cp0_ptb_iter_block_step, int cp0_ptb_end_block_pos, float* sgemm1_A, float* sgemm1_B, float* sgemm1_C, int sgemm1_NORMAL_M, int sgemm1_NORMAL_N, int sgemm1_NORMAL_K, int sgemm1_grid_dimension_x, int sgemm1_grid_dimension_y, int sgemm1_grid_dimension_z, int sgemm1_block_dimension_x, int sgemm1_block_dimension_y, int sgemm1_block_dimension_z, int sgemm1_ptb_start_block_pos, int sgemm1_ptb_iter_block_step, int sgemm1_ptb_end_block_pos){
    if (threadIdx.x < 128) {
        general_ptb_cp0(
            cp0_numatoms, cp0_gridspacing, cp0_energygrid, cp0_grid_dimension_x, cp0_grid_dimension_y, cp0_grid_dimension_z, cp0_block_dimension_x, cp0_block_dimension_y, cp0_block_dimension_z, cp0_ptb_start_block_pos + 0 * cp0_ptb_iter_block_step, cp0_ptb_iter_block_step * 3, cp0_ptb_end_block_pos, 0
        );
    }
    else if (threadIdx.x < 256) {
        general_ptb_cp1(
            cp0_numatoms, cp0_gridspacing, cp0_energygrid, cp0_grid_dimension_x, cp0_grid_dimension_y, cp0_grid_dimension_z, cp0_block_dimension_x, cp0_block_dimension_y, cp0_block_dimension_z, cp0_ptb_start_block_pos + 1 * cp0_ptb_iter_block_step, cp0_ptb_iter_block_step * 3, cp0_ptb_end_block_pos, 128
        );
    }
    else if (threadIdx.x < 384) {
        general_ptb_cp2(
            cp0_numatoms, cp0_gridspacing, cp0_energygrid, cp0_grid_dimension_x, cp0_grid_dimension_y, cp0_grid_dimension_z, cp0_block_dimension_x, cp0_block_dimension_y, cp0_block_dimension_z, cp0_ptb_start_block_pos + 2 * cp0_ptb_iter_block_step, cp0_ptb_iter_block_step * 3, cp0_ptb_end_block_pos, 256
        );
    }
    else if (threadIdx.x < 512) {
        general_ptb_sgemm0(
            sgemm1_A, sgemm1_B, sgemm1_C, sgemm1_NORMAL_M, sgemm1_NORMAL_N, sgemm1_NORMAL_K, sgemm1_grid_dimension_x, sgemm1_grid_dimension_y, sgemm1_grid_dimension_z, sgemm1_block_dimension_x, sgemm1_block_dimension_y, sgemm1_block_dimension_z, sgemm1_ptb_start_block_pos + 0 * sgemm1_ptb_iter_block_step, sgemm1_ptb_iter_block_step * 1, sgemm1_ptb_end_block_pos, 384
        );
    }

}
