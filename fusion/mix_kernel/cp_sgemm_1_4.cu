#include "hip/hip_runtime.h"
// cp-sgemm-1-4
__global__ void mixed_cp_sgemm_kernel_1_4(int cp0_numatoms, float cp0_gridspacing, float* cp0_energygrid, int cp0_grid_dimension_x, int cp0_grid_dimension_y, int cp0_grid_dimension_z, int cp0_block_dimension_x, int cp0_block_dimension_y, int cp0_block_dimension_z, int cp0_ptb_start_block_pos, int cp0_ptb_iter_block_step, int cp0_ptb_end_block_pos, float* sgemm1_A, float* sgemm1_B, float* sgemm1_C, int sgemm1_NORMAL_M, int sgemm1_NORMAL_N, int sgemm1_NORMAL_K, int sgemm1_grid_dimension_x, int sgemm1_grid_dimension_y, int sgemm1_grid_dimension_z, int sgemm1_block_dimension_x, int sgemm1_block_dimension_y, int sgemm1_block_dimension_z, int sgemm1_ptb_start_block_pos, int sgemm1_ptb_iter_block_step, int sgemm1_ptb_end_block_pos){
    if (threadIdx.x < 128) {
        general_ptb_cp0(
            cp0_numatoms, cp0_gridspacing, cp0_energygrid, cp0_grid_dimension_x, cp0_grid_dimension_y, cp0_grid_dimension_z, cp0_block_dimension_x, cp0_block_dimension_y, cp0_block_dimension_z, cp0_ptb_start_block_pos + 0 * cp0_ptb_iter_block_step, cp0_ptb_iter_block_step * 1, cp0_ptb_end_block_pos, 0
        );
    }
    else if (threadIdx.x < 256) {
        general_ptb_sgemm0(
            sgemm1_A, sgemm1_B, sgemm1_C, sgemm1_NORMAL_M, sgemm1_NORMAL_N, sgemm1_NORMAL_K, sgemm1_grid_dimension_x, sgemm1_grid_dimension_y, sgemm1_grid_dimension_z, sgemm1_block_dimension_x, sgemm1_block_dimension_y, sgemm1_block_dimension_z, sgemm1_ptb_start_block_pos + 0 * sgemm1_ptb_iter_block_step, sgemm1_ptb_iter_block_step * 4, sgemm1_ptb_end_block_pos, 128
        );
    }
    else if (threadIdx.x < 384) {
        general_ptb_sgemm1(
            sgemm1_A, sgemm1_B, sgemm1_C, sgemm1_NORMAL_M, sgemm1_NORMAL_N, sgemm1_NORMAL_K, sgemm1_grid_dimension_x, sgemm1_grid_dimension_y, sgemm1_grid_dimension_z, sgemm1_block_dimension_x, sgemm1_block_dimension_y, sgemm1_block_dimension_z, sgemm1_ptb_start_block_pos + 1 * sgemm1_ptb_iter_block_step, sgemm1_ptb_iter_block_step * 4, sgemm1_ptb_end_block_pos, 256
        );
    }
    else if (threadIdx.x < 512) {
        general_ptb_sgemm2(
            sgemm1_A, sgemm1_B, sgemm1_C, sgemm1_NORMAL_M, sgemm1_NORMAL_N, sgemm1_NORMAL_K, sgemm1_grid_dimension_x, sgemm1_grid_dimension_y, sgemm1_grid_dimension_z, sgemm1_block_dimension_x, sgemm1_block_dimension_y, sgemm1_block_dimension_z, sgemm1_ptb_start_block_pos + 2 * sgemm1_ptb_iter_block_step, sgemm1_ptb_iter_block_step * 4, sgemm1_ptb_end_block_pos, 384
        );
    }
    else if (threadIdx.x < 640) {
        general_ptb_sgemm3(
            sgemm1_A, sgemm1_B, sgemm1_C, sgemm1_NORMAL_M, sgemm1_NORMAL_N, sgemm1_NORMAL_K, sgemm1_grid_dimension_x, sgemm1_grid_dimension_y, sgemm1_grid_dimension_z, sgemm1_block_dimension_x, sgemm1_block_dimension_y, sgemm1_block_dimension_z, sgemm1_ptb_start_block_pos + 3 * sgemm1_ptb_iter_block_step, sgemm1_ptb_iter_block_step * 4, sgemm1_ptb_end_block_pos, 512
        );
    }

}
