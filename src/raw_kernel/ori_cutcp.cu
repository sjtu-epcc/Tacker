#include "hip/hip_runtime.h"
#include <mma.h>
#include <iostream>
using namespace nvcuda; 
#include "header/atom.h"
#include "header/cutcp_header.h"

extern "C" __global__ void ori_cutcp(
    int binDim_x,
    int binDim_y,
    float4 *binZeroAddr,    /* address of atom bins starting at origin */
    float h,                /* lattice spacing */
    float cutoff2,          /* square of cutoff distance */
    float inv_cutoff2,
    float *regionZeroAddr, /* address of lattice regions starting at origin */
    int zRegionIndex_t
    )
{
	__shared__ float AtomBinCache[BIN_CACHE_MAXLEN * BIN_DEPTH * 4];
    // if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0 && blockIdx.x == 0)
    //     printf("%d, %d, %d\n", blockIdx.x, blockIdx.y, blockIdx.z);
    // __shared__ float *myRegionAddr;
	// __shared__ int3 myBinIndex;

    float *myRegionAddr;
	int3 myBinIndex;

	const int xRegionIndex = blockIdx.x;
	const int yRegionIndex = blockIdx.y;
    const int zRegionIndex = blockIdx.z;
	/* thread id */
	const int tid = (threadIdx.z*blockDim.y + threadIdx.y)*blockDim.x + threadIdx.x;

        /* neighbor index */
        int nbrid;

        /* this is the start of the sub-region indexed by tid */
        myRegionAddr = regionZeroAddr + ((zRegionIndex*gridDim.y + yRegionIndex)*gridDim.x + xRegionIndex)*REGION_SIZE;

        /* spatial coordinate of this lattice point */
        float x = (8 * xRegionIndex + threadIdx.x) * h;
        float y = (8 * yRegionIndex + threadIdx.y) * h;
        float z = (8 * zRegionIndex + threadIdx.z) * h;

        int totalbins = 0;
        int numbins;

        /* bin number determined by center of region */
        myBinIndex.x = (int) floorf((8 * xRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.y = (int) floorf((8 * yRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.z = (int) floorf((8 * zRegionIndex + 4) * h * BIN_INVLEN);

        /* first neighbor in list for me to cache */
        nbrid = (tid >> 4);

        numbins = BIN_CACHE_MAXLEN;

        float energy0 = 0.f;
        float energy1 = 0.f;
        float energy2 = 0.f;
        float energy3 = 0.f;


        for (totalbins = 0;  totalbins < NbrListLen;  totalbins += numbins) {
            int bincnt;

            /* start of where to write in shared memory */
            int startoff = BIN_SIZE * (tid >> 4);

            /* each half-warp to cache up to 4 atom bins */
            for (bincnt = 0;  bincnt < 4 && nbrid < NbrListLen;  bincnt++, nbrid += 8) {
                int i = myBinIndex.x + NbrList[nbrid].x;
                int j = myBinIndex.y + NbrList[nbrid].y;
                int k = myBinIndex.z + NbrList[nbrid].z;

                /* determine global memory location of atom bin */
                float *p_global = ((float *) binZeroAddr) + (((__mul24(k, binDim_y) + j)*binDim_x + i) << BIN_SHIFT);

                /* coalesced read from global memory -
                * retain same ordering in shared memory for now */
                int binIndex = startoff + (bincnt << (3 + BIN_SHIFT));
                int tidmask = tid & 15;

                AtomBinCache[binIndex + tidmask   ] = p_global[tidmask   ];
                AtomBinCache[binIndex + tidmask+16] = p_global[tidmask+16];
            }
            __syncthreads();

            /* no warp divergence */
            if (totalbins + BIN_CACHE_MAXLEN > NbrListLen) {
                numbins = NbrListLen - totalbins;
            }

            int stopbin = (numbins << BIN_SHIFT);
            for (bincnt = 0; bincnt < stopbin; bincnt+=BIN_SIZE) {
                for (int i = 0;  i < BIN_DEPTH;  i++) {
                    int off = bincnt + (i<<2);

                    float aq = AtomBinCache[off + 3];
                    if (0.f == aq) 
                        break;  /* no more atoms in bin */

                    float dx = AtomBinCache[off    ] - x;
                    float dz = AtomBinCache[off + 2] - z;
                    float dxdz2 = dx*dx + dz*dz;
                    float dy = AtomBinCache[off + 1] - y;
                    float r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy0 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy1 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy2 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy3 += aq * rsqrtf(r2) * s * s;
                    }
                } /* end loop over atoms in bin */
            } /* end loop over cached atom bins */
            __syncthreads();
        //     if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0 && blockIdx.x == 0 && blockIdx.y == 0 && blockIdx.z == 0)
        //         printf("numbins: %d\n", numbins);
        // } /* end loop over neighbor list */

        /* store into global memory */
        myRegionAddr[(tid>>4)*64 + (tid&15)     ] = energy0;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 16] = energy1;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 32] = energy2;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 48] = energy3;
}
}