
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

#define BLOCK_SIZE 256
#define STR_SIZE 256
#define DEVICE 0
#define HALO 1 
// halo width along one direction when advancing to the next iteration

#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

void run(int argc, char** argv);

int rows, cols;
int* data;
int** wall;
int* result;
#define M_SEED 9
int pyramid_height;

void init(int argc, char** argv)
{
	if(argc==4){
		cols = atoi(argv[1]);
		rows = atoi(argv[2]);
        pyramid_height=atoi(argv[3]);
	}else{
        printf("Usage: dynproc row_len col_len pyramid_height\n");
        exit(0);
    }
	data = new int[rows*cols];
	wall = new int*[rows];
	for(int n=0; n<rows; n++)
		wall[n]=data+cols*n;
	result = new int[cols];
	
	int seed = M_SEED;
	srand(seed);

	for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            wall[i][j] = rand() % 10;
        }
    }
}

void fatal(char *s)
{
	fprintf(stderr, "error: %s\n", s);

}

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))

__global__ void dynproc_kernel(
                int iteration, 
                int *gpuWall,
                int *gpuSrc,
                int *gpuResults,
                int cols, 
                int rows,
                int startStep,
                int border)
{
    __shared__ int prev[BLOCK_SIZE];
    __shared__ int result[BLOCK_SIZE];

	int bx = blockIdx.x;
	int tx=threadIdx.x;

    // each block finally computes result for a small block
    // after N iterations. 
    // it is the non-overlapping small blocks that cover 
    // all the input data

    // calculate the small block size
	int small_block_cols = BLOCK_SIZE-iteration*HALO*2;

    // calculate the boundary for the block according to 
    // the boundary of its small block
    int blkX = small_block_cols*bx-border;
    int blkXmax = blkX+BLOCK_SIZE-1;

        // calculate the global thread coordination
	int xidx = blkX+tx;
    
    // effective range within this block that falls within 
    // the valid range of the input data
    // used to rule out computation outside the boundary.
    int validXmin = (blkX < 0) ? -blkX : 0;
    int validXmax = (blkXmax > cols-1) ? BLOCK_SIZE-1-(blkXmax-cols+1) : BLOCK_SIZE-1;

    int W = tx-1;
    int E = tx+1;
    
    W = (W < validXmin) ? validXmin : W;
    E = (E > validXmax) ? validXmax : E;

    bool isValid = IN_RANGE(tx, validXmin, validXmax);

	if(IN_RANGE(xidx, 0, cols-1)){
            prev[tx] = gpuSrc[xidx];
	}
	__syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
    bool computed;
    for (int i=0; i<iteration ; i++){ 
        computed = false;
        if( IN_RANGE(tx, i+1, BLOCK_SIZE-i-2) &&  \
                isValid){
                computed = true;
                int left = prev[W];
                int up = prev[tx];
                int right = prev[E];
                int shortest = MIN(left, up);
                shortest = MIN(shortest, right);
                int index = cols*(startStep+i)+xidx;
                result[tx] = shortest + gpuWall[index];

        }
        __syncthreads();
        if(i==iteration-1)
            break;
        if(computed)	 //Assign the computation range
            prev[tx]= result[tx];
        __syncthreads(); // [Ronny] Added sync to avoid race on prev Aug. 14 2012
    }

    // update the global memory
    // after the last iteration, only threads coordinated within the 
    // small block perform the calculation and switch on ``computed''
    if (computed){
        gpuResults[xidx]=result[tx];		
    }
}

/*
   compute N time steps
*/
int calc_path(int *gpuWall, int *gpuResult[2], int rows, int cols, \
	 int pyramid_height, int blockCols, int borderCols)
{
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(blockCols);  

    int src = 1, dst = 0;
    for (int t = 0; t < rows-1; t+=pyramid_height) {
        int temp = src;
        src = dst;
        dst = temp;
        dynproc_kernel<<<dimGrid, dimBlock>>>(
            MIN(pyramid_height, rows-t-1), 
            gpuWall, gpuResult[src], gpuResult[dst],
            cols,rows, t, borderCols);

        // for the measurement fairness
        hipDeviceSynchronize();
    }
    return dst;
}

int main(int argc, char** argv)
{
    init(argc, argv);

    float kernel_time;
	hipEvent_t startKERNEL;
	hipEvent_t stopKERNEL;
	cudaErrCheck(hipEventCreate(&startKERNEL));
	cudaErrCheck(hipEventCreate(&stopKERNEL));

    /* --------------- pyramid parameters --------------- */
    int borderCols = (pyramid_height)*HALO;
    int smallBlockCol = BLOCK_SIZE-(pyramid_height)*HALO*2;
    int blockCols = cols/smallBlockCol+((cols%smallBlockCol==0)?0:1);

    printf("pyramidHeight: %d\ngridSize: [%d]\nborder:[%d]\nblockSize: %d\nblockGrid:[%d]\ntargetBlock:[%d]\n",\
	pyramid_height, cols, borderCols, BLOCK_SIZE, blockCols, smallBlockCol);
	
    int *gpuWall, *gpuResult[2];
    int size = rows*cols;

    hipMalloc((void**)&gpuResult[0], sizeof(int)*cols);
    hipMalloc((void**)&gpuResult[1], sizeof(int)*cols);
    hipMemcpy(gpuResult[0], data, sizeof(int)*cols, hipMemcpyHostToDevice);
    hipMalloc((void**)&gpuWall, sizeof(int)*(size-cols));
    hipMemcpy(gpuWall, data+cols, sizeof(int)*(size-cols), hipMemcpyHostToDevice);


    cudaErrCheck(hipEventRecord(startKERNEL));
    int final_ret = calc_path(gpuWall, gpuResult, rows, cols, pyramid_height, blockCols, borderCols);
    cudaErrCheck(hipEventRecord(stopKERNEL));
	cudaErrCheck(hipEventSynchronize(stopKERNEL));
	cudaErrCheck(hipEventElapsedTime(&kernel_time, startKERNEL, stopKERNEL));
	printf("[ORI] path took %f ms\n\n", kernel_time);

    hipMemcpy(result, gpuResult[final_ret], sizeof(int)*cols, hipMemcpyDeviceToHost);

    hipFree(gpuWall);
    hipFree(gpuResult[0]);
    hipFree(gpuResult[1]);

    delete [] data;
    delete [] wall;
    delete [] result;

    return 0;
}

