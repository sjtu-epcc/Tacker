#include "hip/hip_runtime.h"
#include <stdio.h>	
#include <stdlib.h>	
#include <stdbool.h>

#define NUMBER_PAR_PER_BOX 100							// keep this low to allow more blocks that share shared memory to run concurrently, code does not work for larger than 110, more speedup can be achieved with larger number and no shared memory used

/* #define NUMBER_THREADS 128								// this should be roughly equal to NUMBER_PAR_PER_BOX for best performance */

// Parameterized work group size
#define NUMBER_THREADS 128
#define DOT(A,B) ((A.x)*(B.x)+(A.y)*(B.y)+(A.z)*(B.z))	// STABLE

typedef struct
{

	float x, y, z;
} THREE_VECTOR;

typedef struct
{
	float v, x, y, z;
} FOUR_VECTOR;

typedef struct nei_str
{
	// neighbor box
	int x, y, z;
	int number;
	long offset;
} nei_str;

typedef struct box_str
{
	// home box
	int x, y, z;
	int number;
	long offset;

	// neighbor boxes
	int nn;
	nei_str nei[26];
} box_str;

typedef struct par_str
{
	float alpha;
} par_str;

typedef struct dim_str
{
	// input arguments
	int cur_arg;
	int arch_arg;
	int cores_arg;
	int boxes1d_arg;

	// system memory
	long number_boxes;
	long box_mem;
	long space_elem;
	long space_mem;
	long space_mem2;
} dim_str;

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		// floatrintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
		printf("Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
		fflush(NULL);
		exit(EXIT_FAILURE);
	}
}


#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}


__global__ void kernel_gpu_cuda(par_str d_par_gpu,
								dim_str d_dim_gpu,
								box_str* d_box_gpu,
								FOUR_VECTOR* d_rv_gpu,
								float* d_qv_gpu,
								FOUR_VECTOR* d_fv_gpu)
{

	int bx = blockIdx.x;
	int tx = threadIdx.x;
	int wtx = tx;

	if(bx<d_dim_gpu.number_boxes){
		// parameters
		float a2 = 2.0*d_par_gpu.alpha*d_par_gpu.alpha;

		// home box
		int first_i;
		FOUR_VECTOR* rA;
		FOUR_VECTOR* fA;
		__shared__ FOUR_VECTOR rA_shared[100];

		// nei box
		int pointer;
		int k = 0;
		int first_j;
		FOUR_VECTOR* rB;
		float* qB;
		int j = 0;
		__shared__ FOUR_VECTOR rB_shared[100];
		__shared__ double qB_shared[100];

		// common
		float r2;
		float u2;
		float vij;
		float fs;
		float fxij;
		float fyij;
		float fzij;
		THREE_VECTOR d;

		// home box - box parameters
		first_i = d_box_gpu[bx].offset;

		// home box - distance, force, charge and type parameters
		rA = &d_rv_gpu[first_i];
		fA = &d_fv_gpu[first_i];

		// home box - shared memory
		while(wtx<NUMBER_PAR_PER_BOX){
			rA_shared[wtx] = rA[wtx];
			wtx = wtx + NUMBER_THREADS;
		}
		wtx = tx;

		// synchronize threads  - not needed, but just to be safe
		__syncthreads();

		// loop over neiing boxes of home box
		for (k=0; k<(1+d_box_gpu[bx].nn); k++){

			if(k==0){
				pointer = bx; // set first box to be processed to home box
			}
			else{
				pointer = d_box_gpu[bx].nei[k-1].number; 
				// remaining boxes are nei boxes
			}

			// nei box - box parameters
			first_j = d_box_gpu[pointer].offset;

			// nei box - distance, (force), charge and (type) parameters
			rB = &d_rv_gpu[first_j];
			qB = &d_qv_gpu[first_j];

			// nei box - shared memory
			while(wtx<NUMBER_PAR_PER_BOX){
				rB_shared[wtx] = rB[wtx];
				qB_shared[wtx] = qB[wtx];
				wtx = wtx + NUMBER_THREADS;
			}
			wtx = tx;

			// synchronize threads because in next section each thread accesses data brought in by different threads here
			__syncthreads();

			// loop for the number of particles in the home box
			while(wtx<NUMBER_PAR_PER_BOX){

				// loop for the number of particles in the current nei box
				for (j=0; j<NUMBER_PAR_PER_BOX; j++){

					r2 = (float)rA_shared[wtx].v + (float)rB_shared[j].v - DOT((float)rA_shared[wtx],(float)rB_shared[j]); 
					u2 = a2*r2;
					vij= exp(-u2);
					fs = 2*vij;

					d.x = (float)rA_shared[wtx].x  - (float)rB_shared[j].x;
					fxij=fs*d.x;
					d.y = (float)rA_shared[wtx].y  - (float)rB_shared[j].y;
					fyij=fs*d.y;
					d.z = (float)rA_shared[wtx].z  - (float)rB_shared[j].z;
					fzij=fs*d.z;

					fA[wtx].v +=  (double)((float)qB_shared[j]*vij);
					fA[wtx].x +=  (double)((float)qB_shared[j]*fxij);
					fA[wtx].y +=  (double)((float)qB_shared[j]*fyij);
					fA[wtx].z +=  (double)((float)qB_shared[j]*fzij);

				}

				// increment work thread index
				wtx = wtx + NUMBER_THREADS;

			}

			// reset work index
			wtx = tx;

			// synchronize after finishing force contributions from current nei box not to cause conflicts when starting next box
			__syncthreads();

		}

	}

}

void kernel_gpu_cuda_wrapper(par_str par_cpu,
						dim_str dim_cpu,
						box_str* box_cpu,
						FOUR_VECTOR* rv_cpu,
						float* qv_cpu,
						FOUR_VECTOR* fv_cpu)
{

	hipDeviceSynchronize();

	float kernel_time;
	hipEvent_t startKERNEL;
	hipEvent_t stopKERNEL;
	cudaErrCheck(hipEventCreate(&startKERNEL));
	cudaErrCheck(hipEventCreate(&stopKERNEL));

	box_str* d_box_gpu;
	FOUR_VECTOR* d_rv_gpu;
	float* d_qv_gpu;
	FOUR_VECTOR* d_fv_gpu;

	dim3 threads;
	dim3 blocks;

	blocks.x = dim_cpu.number_boxes;
	blocks.y = 1;
	threads.x = NUMBER_THREADS;											// define the number of threads in the block
	threads.y = 1;

	printf("[ORI] block.x %d thread.x %d\n", blocks.x, threads.x);

	hipMalloc(	(void **)&d_box_gpu, 
				dim_cpu.box_mem);
	hipMalloc(	(void **)&d_rv_gpu, 
				dim_cpu.space_mem);
	hipMalloc(	(void **)&d_qv_gpu, 
				dim_cpu.space_mem2);
	hipMalloc(	(void **)&d_fv_gpu, 
				dim_cpu.space_mem);
	hipMemcpy(	d_box_gpu, 
				box_cpu,
				dim_cpu.box_mem, 
				hipMemcpyHostToDevice);
	hipMemcpy(	d_rv_gpu,
				rv_cpu,
				dim_cpu.space_mem,
				hipMemcpyHostToDevice);
	hipMemcpy(	d_qv_gpu,
				qv_cpu,
				dim_cpu.space_mem2,
				hipMemcpyHostToDevice);
	hipMemcpy(	d_fv_gpu, 
				fv_cpu, 
				dim_cpu.space_mem, 
				hipMemcpyHostToDevice);

	cudaErrCheck(hipEventRecord(startKERNEL));
	// launch kernel - all boxes
	kernel_gpu_cuda<<<blocks, threads>>>(	par_cpu,
											dim_cpu,
											d_box_gpu,
											d_rv_gpu,
											d_qv_gpu,
											d_fv_gpu);

	cudaErrCheck(hipEventRecord(stopKERNEL));
	cudaErrCheck(hipEventSynchronize(stopKERNEL));
	cudaErrCheck(hipEventElapsedTime(&kernel_time, startKERNEL, stopKERNEL));
	printf("[ORI] lava took %f ms\n\n", kernel_time);

	hipMemcpy(	fv_cpu, 
				d_fv_gpu, 
				dim_cpu.space_mem, 
				hipMemcpyDeviceToHost);

	hipFree(d_rv_gpu);
	hipFree(d_qv_gpu);
	hipFree(d_fv_gpu);
	hipFree(d_box_gpu);
}

int isInteger(char *str){
	if (*str == '\0'){
		return 0;
	}

	for(; *str != '\0'; str++){
		if (*str < 48 || *str > 57){	
			// digit characters (need to include . if checking for float)
			return 0;
		}
	}

	return 1;
}



int main(	int argc, 
		char *argv [])
{
	printf("thread block size of kernel = %d \n", NUMBER_THREADS);

	// counters
	int i, j, k, l, m, n;

	// system memory
	par_str par_cpu;
	dim_str dim_cpu;
	box_str* box_cpu;
	FOUR_VECTOR* rv_cpu;
	float* qv_cpu;
	FOUR_VECTOR* fv_cpu;
	int nh;

	// assing default values
	dim_cpu.boxes1d_arg = 1;

	// go through arguments
	for(dim_cpu.cur_arg=1; dim_cpu.cur_arg<argc; dim_cpu.cur_arg++){
		// check if -boxes1d
		if(strcmp(argv[dim_cpu.cur_arg], "-boxes1d")==0){
			// check if value provided
			if(argc>=dim_cpu.cur_arg+1){
				// check if value is a number
				if(isInteger(argv[dim_cpu.cur_arg+1])==1){
					dim_cpu.boxes1d_arg = atoi(argv[dim_cpu.cur_arg+1]);
					if(dim_cpu.boxes1d_arg<0){
						printf("ERROR: Wrong value to -boxes1d parameter, cannot be <=0\n");
						return 0;
					}
					dim_cpu.cur_arg = dim_cpu.cur_arg+1;
				}
				// value is not a number
				else{
					printf("ERROR: Value to -boxes1d parameter in not a number\n");
					return 0;
				}
			}
			// value not provided
			else{
				printf("ERROR: Missing value to -boxes1d parameter\n");
				return 0;
			}
		}
		// unknown
		else{
			printf("ERROR: Unknown parameter\n");
			return 0;
		}
	}

	// Print configuration
	printf("Configuration used: boxes1d = %d\n", dim_cpu.boxes1d_arg);

	par_cpu.alpha = 0.5;

	// total number of boxes
	dim_cpu.number_boxes = dim_cpu.boxes1d_arg * dim_cpu.boxes1d_arg * dim_cpu.boxes1d_arg;

	// how many particles space has in each direction
	dim_cpu.space_elem = dim_cpu.number_boxes * NUMBER_PAR_PER_BOX;
	dim_cpu.space_mem = dim_cpu.space_elem * sizeof(FOUR_VECTOR);
	dim_cpu.space_mem2 = dim_cpu.space_elem * sizeof(float);

	// box array
	dim_cpu.box_mem = dim_cpu.number_boxes * sizeof(box_str);

	// allocate boxes
	box_cpu = (box_str*)malloc(dim_cpu.box_mem);

	// initialize number of home boxes
	nh = 0;

	// home boxes in z direction
	for(i=0; i<dim_cpu.boxes1d_arg; i++){
		// home boxes in y direction
		for(j=0; j<dim_cpu.boxes1d_arg; j++){
			// home boxes in x direction
			for(k=0; k<dim_cpu.boxes1d_arg; k++){

				// current home box
				box_cpu[nh].x = k;
				box_cpu[nh].y = j;
				box_cpu[nh].z = i;
				box_cpu[nh].number = nh;
				box_cpu[nh].offset = nh * NUMBER_PAR_PER_BOX;

				// initialize number of neighbor boxes
				box_cpu[nh].nn = 0;

				// neighbor boxes in z direction
				for(l=-1; l<2; l++){
					// neighbor boxes in y direction
					for(m=-1; m<2; m++){
						// neighbor boxes in x direction
						for(n=-1; n<2; n++){

							// check if (this neighbor exists) and (it is not the same as home box)
							if(		(((i+l)>=0 && (j+m)>=0 && (k+n)>=0)==true && ((i+l)<dim_cpu.boxes1d_arg && (j+m)<dim_cpu.boxes1d_arg && (k+n)<dim_cpu.boxes1d_arg)==true)	&&
									(l==0 && m==0 && n==0)==false	){

								// current neighbor box
								box_cpu[nh].nei[box_cpu[nh].nn].x = (k+n);
								box_cpu[nh].nei[box_cpu[nh].nn].y = (j+m);
								box_cpu[nh].nei[box_cpu[nh].nn].z = (i+l);
								box_cpu[nh].nei[box_cpu[nh].nn].number =	(box_cpu[nh].nei[box_cpu[nh].nn].z * dim_cpu.boxes1d_arg * dim_cpu.boxes1d_arg) + 
																			(box_cpu[nh].nei[box_cpu[nh].nn].y * dim_cpu.boxes1d_arg) + 
																			 box_cpu[nh].nei[box_cpu[nh].nn].x;
								box_cpu[nh].nei[box_cpu[nh].nn].offset = box_cpu[nh].nei[box_cpu[nh].nn].number * NUMBER_PAR_PER_BOX;

								// increment neighbor box
								box_cpu[nh].nn = box_cpu[nh].nn + 1;

							}

						} // neighbor boxes in x direction
					} // neighbor boxes in y direction
				} // neighbor boxes in z direction

				// increment home box
				nh = nh + 1;

			} // home boxes in x direction
		} // home boxes in y direction
	} // home boxes in z direction

	// random generator seed set to random value - time in this case
	srand(time(NULL));

	// input (distances)
	rv_cpu = (FOUR_VECTOR*)malloc(dim_cpu.space_mem);
	for(i=0; i<dim_cpu.space_elem; i=i+1){
		rv_cpu[i].v = (rand()%10 + 1) / 10.0;			// get a number in the range 0.1 - 1.0
		rv_cpu[i].x = (rand()%10 + 1) / 10.0;			// get a number in the range 0.1 - 1.0
		rv_cpu[i].y = (rand()%10 + 1) / 10.0;			// get a number in the range 0.1 - 1.0
		rv_cpu[i].z = (rand()%10 + 1) / 10.0;			// get a number in the range 0.1 - 1.0
	}

	// input (charge)
	qv_cpu = (float*)malloc(dim_cpu.space_mem2);
	for(i=0; i<dim_cpu.space_elem; i=i+1){
		qv_cpu[i] = (rand()%10 + 1) / 10.0;			// get a number in the range 0.1 - 1.0
	}

	// output (forces)
	fv_cpu = (FOUR_VECTOR*)malloc(dim_cpu.space_mem);
	for(i=0; i<dim_cpu.space_elem; i=i+1){
		fv_cpu[i].v = 0;								// set to 0, because kernels keeps adding to initial value
		fv_cpu[i].x = 0;								// set to 0, because kernels keeps adding to initial value
		fv_cpu[i].y = 0;								// set to 0, because kernels keeps adding to initial value
		fv_cpu[i].z = 0;								// set to 0, because kernels keeps adding to initial value
	}

	

	kernel_gpu_cuda_wrapper(par_cpu,
							dim_cpu,
							box_cpu,
							rv_cpu,
							qv_cpu,
							fv_cpu);

	// dump results
#ifdef OUTPUT
        FILE *floattr;
	floattr = fopen("result.txt", "w");	
	for(i=0; i<dim_cpu.space_elem; i=i+1){
        	floatrintf(floattr, "%f, %f, %f, %f\n", fv_cpu[i].v, fv_cpu[i].x, fv_cpu[i].y, fv_cpu[i].z);
	}
	fclose(floattr);
#endif       	

	free(rv_cpu);
	free(qv_cpu);
	free(fv_cpu);
	free(box_cpu);

	
	return 0;					
}
