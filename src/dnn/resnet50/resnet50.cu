
#include "Logger.h"
#include "util.h"
#include "TackerConfig.h"
#include "Recorder.h"
#include "./dnn/resnet50/resnet50.h"
#include "./dnn/resnet50/resnet50_kernel_class.h"

extern Logger logger;
extern Recorder recorder;

void Resnet50::initParams() {
    resnet50_cuda_init();
    
    //input argument
    float* Parameter_0_0_host, *Parameter_0_0;
    CUDA_SAFE_CALL(hipHostMalloc((void**)&Parameter_0_0_host, sizeof(float)* 9633792));
    CUDA_SAFE_CALL(hipMalloc((void**)&Parameter_0_0, sizeof(float) * 9633792));
    for (int i = 0; i < 9633792; ++i) Parameter_0_0_host[i] = 1.0f;
    CUDA_SAFE_CALL(hipMemcpy(Parameter_0_0, Parameter_0_0_host, sizeof(float) * 9633792, hipMemcpyHostToDevice));
    this->Input[0] = Parameter_0_0;
    this->InputHost[0] = Parameter_0_0_host;
    this->InputSize[0] = 9633792;

    //output arguments
    float* Result_505_0_host, *Result_505_0;
    CUDA_SAFE_CALL(hipHostMalloc((void**)&Result_505_0_host, sizeof(float) * 64064));

    this->Result = (void**)&Result_505_0;

    //fill input values
    this->gen_vector(Parameter_0_0, (float**)Result);
}