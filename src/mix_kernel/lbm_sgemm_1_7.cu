#include "hip/hip_runtime.h"
#include "header/sgemm_header.h"
#include "header/lbm_header.h"

// lbm
__device__ void lbm_sgemm_lbm0( float* srcGrid, float* dstGrid,
    int grid_dimension_x, int grid_dimension_y, int grid_dimension_z,
    int block_dimension_x, int block_dimension_y, int block_dimension_z,
    int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base) {
	//Using some predefined macros here.  Consider this the declaration 
    //  and initialization of the variables SWEEP_X, SWEEP_Y and SWEEP_Z

    // // ori
    // int thread_id_x = (threadIdx.x - thread_step) % block_dimension_x;

    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;
    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    // int thread_id_y = ((threadIdx.x - thread_step) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = ((threadIdx.x - thread_step) / block_dimension_x) / block_dimension_y;

    SWEEP_VAR
	float temp_swp, tempC, tempN, tempS, tempE, tempW, tempT, tempB;
	float tempNE, tempNW, tempSE, tempSW, tempNT, tempNB, tempST ;
	float tempSB, tempET, tempEB, tempWT, tempWB;

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        // ori
        // int block_id_x = block_pos % grid_dimension_x;
        // int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = (block_pos / grid_dimension_x) / grid_dimension_y;

        int block_id_x = block_pos % grid_dimension_x;
		int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

        // float *d_temp = srcGrid;
        // srcGrid = dstGrid;
        // dstGrid = d_temp;

        SWEEP_X = thread_id_x;
        SWEEP_Y = block_id_x;
        SWEEP_Z = block_id_y;

        //Load all of the input fields
        //This is a gather operation of the SCATTER preprocessor variable
            // is undefined in layout_config.h, or a "local" read otherwise
        tempC = SRC_C(srcGrid);
        tempN = SRC_N(srcGrid);
        tempS = SRC_S(srcGrid);
        tempE = SRC_E(srcGrid);
        tempW = SRC_W(srcGrid);
        tempT = SRC_T(srcGrid);
        tempB = SRC_B(srcGrid);
        tempNE= SRC_NE(srcGrid);
        tempNW= SRC_NW(srcGrid);
        tempSE = SRC_SE(srcGrid);
        tempSW = SRC_SW(srcGrid);
        tempNT = SRC_NT(srcGrid);
        tempNB = SRC_NB(srcGrid);
        tempST = SRC_ST(srcGrid);
        tempSB = SRC_SB(srcGrid);
        tempET = SRC_ET(srcGrid);
        tempEB = SRC_EB(srcGrid);
        tempWT = SRC_WT(srcGrid);
        tempWB = SRC_WB(srcGrid);

        //Test whether the cell is fluid or obstacle
        if( TEST_FLAG_SWEEP( srcGrid, OBSTACLE )) {
            //Swizzle the inputs: reflect any fluid coming into this cell 
            // back to where it came from
            temp_swp = tempN ; tempN = tempS ; tempS = temp_swp ;
            temp_swp = tempE ; tempE = tempW ; tempW = temp_swp;
            temp_swp = tempT ; tempT = tempB ; tempB = temp_swp;
            temp_swp = tempNE; tempNE = tempSW ; tempSW = temp_swp;
            temp_swp = tempNW; tempNW = tempSE ; tempSE = temp_swp;
            temp_swp = tempNT ; tempNT = tempSB ; tempSB = temp_swp; 
            temp_swp = tempNB ; tempNB = tempST ; tempST = temp_swp;
            temp_swp = tempET ; tempET= tempWB ; tempWB = temp_swp;
            temp_swp = tempEB ; tempEB = tempWT ; tempWT = temp_swp;
        }
        else {
            //The math meat of LBM: ignore for optimization
            float ux, uy, uz, rho, u2;
            float temp1, temp2, temp_base;
            rho = tempC + tempN
                + tempS + tempE
                + tempW + tempT
                + tempB + tempNE
                + tempNW + tempSE
                + tempSW + tempNT
                + tempNB + tempST
                + tempSB + tempET
                + tempEB + tempWT
                + tempWB;

            ux = + tempE - tempW
                + tempNE - tempNW
                + tempSE - tempSW
                + tempET + tempEB
                - tempWT - tempWB;
            uy = + tempN - tempS
                + tempNE + tempNW
                - tempSE - tempSW
                + tempNT + tempNB
                - tempST - tempSB;
            uz = + tempT - tempB
                + tempNT - tempNB
                + tempST - tempSB
                + tempET - tempEB
                + tempWT - tempWB;

            ux /= rho;
            uy /= rho;
            uz /= rho;
            if( TEST_FLAG_SWEEP( srcGrid, ACCEL )) {
                ux = 0.005f;
                uy = 0.002f;
                uz = 0.000f;
            }
            u2 = 1.5f * (ux*ux + uy*uy + uz*uz) - 1.0f;
            temp_base = OMEGA*rho;
            temp1 = DFL1*temp_base;


            //Put the output values for this cell in the shared memory
            temp_base = OMEGA*rho;
            temp1 = DFL1*temp_base;
            temp2 = 1.0f-OMEGA;
            tempC = temp2*tempC + temp1*(                                 - u2);
                temp1 = DFL2*temp_base;	
            tempN = temp2*tempN + temp1*(       uy*(4.5f*uy       + 3.0f) - u2);
            tempS = temp2*tempS + temp1*(       uy*(4.5f*uy       - 3.0f) - u2);
            tempT = temp2*tempT + temp1*(       uz*(4.5f*uz       + 3.0f) - u2);
            tempB = temp2*tempB + temp1*(       uz*(4.5f*uz       - 3.0f) - u2);
            tempE = temp2*tempE + temp1*(       ux*(4.5f*ux       + 3.0f) - u2);
            tempW = temp2*tempW + temp1*(       ux*(4.5f*ux       - 3.0f) - u2);
            temp1 = DFL3*temp_base;
            tempNT= temp2*tempNT + temp1 *( (+uy+uz)*(4.5f*(+uy+uz) + 3.0f) - u2);
            tempNB= temp2*tempNB + temp1 *( (+uy-uz)*(4.5f*(+uy-uz) + 3.0f) - u2);
            tempST= temp2*tempST + temp1 *( (-uy+uz)*(4.5f*(-uy+uz) + 3.0f) - u2);
            tempSB= temp2*tempSB + temp1 *( (-uy-uz)*(4.5f*(-uy-uz) + 3.0f) - u2);
            tempNE = temp2*tempNE + temp1 *( (+ux+uy)*(4.5f*(+ux+uy) + 3.0f) - u2);
            tempSE = temp2*tempSE + temp1 *((+ux-uy)*(4.5f*(+ux-uy) + 3.0f) - u2);
            tempET = temp2*tempET + temp1 *( (+ux+uz)*(4.5f*(+ux+uz) + 3.0f) - u2);
            tempEB = temp2*tempEB + temp1 *( (+ux-uz)*(4.5f*(+ux-uz) + 3.0f) - u2);
            tempNW = temp2*tempNW + temp1 *( (-ux+uy)*(4.5f*(-ux+uy) + 3.0f) - u2);
            tempSW = temp2*tempSW + temp1 *( (-ux-uy)*(4.5f*(-ux-uy) + 3.0f) - u2);
            tempWT = temp2*tempWT + temp1 *( (-ux+uz)*(4.5f*(-ux+uz) + 3.0f) - u2);
            tempWB = temp2*tempWB + temp1 *( (-ux-uz)*(4.5f*(-ux-uz) + 3.0f) - u2);
        }

        //Write the results computed above
        //This is a scatter operation of the SCATTER preprocessor variable
            // is defined in layout_config.h, or a "local" write otherwise
        DST_C ( dstGrid ) = tempC;

        DST_N ( dstGrid ) = tempN; 
        DST_S ( dstGrid ) = tempS;
        DST_E ( dstGrid ) = tempE;
        DST_W ( dstGrid ) = tempW;
        DST_T ( dstGrid ) = tempT;
        DST_B ( dstGrid ) = tempB;

        DST_NE( dstGrid ) = tempNE;
        DST_NW( dstGrid ) = tempNW;
        DST_SE( dstGrid ) = tempSE;
        DST_SW( dstGrid ) = tempSW;
        DST_NT( dstGrid ) = tempNT;
        DST_NB( dstGrid ) = tempNB;
        DST_ST( dstGrid ) = tempST;
        DST_SB( dstGrid ) = tempSB;
        DST_ET( dstGrid ) = tempET;
        DST_EB( dstGrid ) = tempEB;
        DST_WT( dstGrid ) = tempWT;
        DST_WB( dstGrid ) = tempWB;
    }
}

// sgemm
__device__ void lbm_sgemm_sgemm0(float *A, float *B, float *C, int NORMAL_M, int NORMAL_N, int NORMAL_K, 
	        int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		        int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base) {
    
    int lda = NORMAL_M;
    int ldb = NORMAL_N;
    int ldc = NORMAL_M;

    float alpha = 2.0f;
    float beta = 2.0f;

    // // ori
    // unsigned int block_pos = blockIdx.x;
    // int thread_id_x = (threadIdx.x - thread_step) % block_dimension_x;
    // int thread_id_y = (threadIdx.x - thread_step) / block_dimension_x;

    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

    __shared__ float b_s[TILE_TB_HEIGHT][TILE_N];

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos % grid_dimension_x;
		int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

        // Partial results
        float c[TILE_N];
        for (int i = 0; i < TILE_N; i++)
            c[i] = 0.0f;
        int mid = (threadIdx.x - thread_base); // TODO: check
        int m = block_id_x * TILE_M + mid;
        int n = block_id_y * TILE_N + thread_id_x;
        

        for (int i = 0; i < NORMAL_K; i += TILE_TB_HEIGHT)
        {
            float a;
            b_s[thread_id_y][thread_id_x] = B[n + (i + thread_id_y) * ldb];
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(1), "r"(128) : "memory");
            for (int j = 0; j < TILE_TB_HEIGHT; j++)
            {
                a = A[m + (i + j) * lda];
                for (int kk = 0; kk < TILE_N; kk++)
                    c[kk] += a * b_s[j][kk];
            }
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(1), "r"(128) : "memory");
        }
        int t = ldc * block_id_y * TILE_N + m;
        for (int i = 0; i < TILE_N; i++)
        {
            C[t + i * ldc] = C[t + i * ldc] * beta + alpha * c[i];
        }
    }
}

__device__ void lbm_sgemm_sgemm1(float *A, float *B, float *C, int NORMAL_M, int NORMAL_N, int NORMAL_K, 
	        int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		        int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base) {
    
    int lda = NORMAL_M;
    int ldb = NORMAL_N;
    int ldc = NORMAL_M;

    float alpha = 2.0f;
    float beta = 2.0f;

    // // ori
    // unsigned int block_pos = blockIdx.x;
    // int thread_id_x = (threadIdx.x - thread_step) % block_dimension_x;
    // int thread_id_y = (threadIdx.x - thread_step) / block_dimension_x;

    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

    __shared__ float b_s[TILE_TB_HEIGHT][TILE_N];

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos % grid_dimension_x;
		int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

        // Partial results
        float c[TILE_N];
        for (int i = 0; i < TILE_N; i++)
            c[i] = 0.0f;
        int mid = (threadIdx.x - thread_base); // TODO: check
        int m = block_id_x * TILE_M + mid;
        int n = block_id_y * TILE_N + thread_id_x;
        

        for (int i = 0; i < NORMAL_K; i += TILE_TB_HEIGHT)
        {
            float a;
            b_s[thread_id_y][thread_id_x] = B[n + (i + thread_id_y) * ldb];
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(2), "r"(128) : "memory");
            for (int j = 0; j < TILE_TB_HEIGHT; j++)
            {
                a = A[m + (i + j) * lda];
                for (int kk = 0; kk < TILE_N; kk++)
                    c[kk] += a * b_s[j][kk];
            }
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(2), "r"(128) : "memory");
        }
        int t = ldc * block_id_y * TILE_N + m;
        for (int i = 0; i < TILE_N; i++)
        {
            C[t + i * ldc] = C[t + i * ldc] * beta + alpha * c[i];
        }
    }
}

__device__ void lbm_sgemm_sgemm2(float *A, float *B, float *C, int NORMAL_M, int NORMAL_N, int NORMAL_K, 
	        int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		        int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base) {
    
    int lda = NORMAL_M;
    int ldb = NORMAL_N;
    int ldc = NORMAL_M;

    float alpha = 2.0f;
    float beta = 2.0f;

    // // ori
    // unsigned int block_pos = blockIdx.x;
    // int thread_id_x = (threadIdx.x - thread_step) % block_dimension_x;
    // int thread_id_y = (threadIdx.x - thread_step) / block_dimension_x;

    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

    __shared__ float b_s[TILE_TB_HEIGHT][TILE_N];

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos % grid_dimension_x;
		int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

        // Partial results
        float c[TILE_N];
        for (int i = 0; i < TILE_N; i++)
            c[i] = 0.0f;
        int mid = (threadIdx.x - thread_base); // TODO: check
        int m = block_id_x * TILE_M + mid;
        int n = block_id_y * TILE_N + thread_id_x;
        

        for (int i = 0; i < NORMAL_K; i += TILE_TB_HEIGHT)
        {
            float a;
            b_s[thread_id_y][thread_id_x] = B[n + (i + thread_id_y) * ldb];
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(3), "r"(128) : "memory");
            for (int j = 0; j < TILE_TB_HEIGHT; j++)
            {
                a = A[m + (i + j) * lda];
                for (int kk = 0; kk < TILE_N; kk++)
                    c[kk] += a * b_s[j][kk];
            }
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(3), "r"(128) : "memory");
        }
        int t = ldc * block_id_y * TILE_N + m;
        for (int i = 0; i < TILE_N; i++)
        {
            C[t + i * ldc] = C[t + i * ldc] * beta + alpha * c[i];
        }
    }
}

__device__ void lbm_sgemm_sgemm3(float *A, float *B, float *C, int NORMAL_M, int NORMAL_N, int NORMAL_K, 
	        int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		        int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base) {
    
    int lda = NORMAL_M;
    int ldb = NORMAL_N;
    int ldc = NORMAL_M;

    float alpha = 2.0f;
    float beta = 2.0f;

    // // ori
    // unsigned int block_pos = blockIdx.x;
    // int thread_id_x = (threadIdx.x - thread_step) % block_dimension_x;
    // int thread_id_y = (threadIdx.x - thread_step) / block_dimension_x;

    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

    __shared__ float b_s[TILE_TB_HEIGHT][TILE_N];

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos % grid_dimension_x;
		int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

        // Partial results
        float c[TILE_N];
        for (int i = 0; i < TILE_N; i++)
            c[i] = 0.0f;
        int mid = (threadIdx.x - thread_base); // TODO: check
        int m = block_id_x * TILE_M + mid;
        int n = block_id_y * TILE_N + thread_id_x;
        

        for (int i = 0; i < NORMAL_K; i += TILE_TB_HEIGHT)
        {
            float a;
            b_s[thread_id_y][thread_id_x] = B[n + (i + thread_id_y) * ldb];
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(4), "r"(128) : "memory");
            for (int j = 0; j < TILE_TB_HEIGHT; j++)
            {
                a = A[m + (i + j) * lda];
                for (int kk = 0; kk < TILE_N; kk++)
                    c[kk] += a * b_s[j][kk];
            }
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(4), "r"(128) : "memory");
        }
        int t = ldc * block_id_y * TILE_N + m;
        for (int i = 0; i < TILE_N; i++)
        {
            C[t + i * ldc] = C[t + i * ldc] * beta + alpha * c[i];
        }
    }
}

__device__ void lbm_sgemm_sgemm4(float *A, float *B, float *C, int NORMAL_M, int NORMAL_N, int NORMAL_K, 
	        int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		        int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base) {
    
    int lda = NORMAL_M;
    int ldb = NORMAL_N;
    int ldc = NORMAL_M;

    float alpha = 2.0f;
    float beta = 2.0f;

    // // ori
    // unsigned int block_pos = blockIdx.x;
    // int thread_id_x = (threadIdx.x - thread_step) % block_dimension_x;
    // int thread_id_y = (threadIdx.x - thread_step) / block_dimension_x;

    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

    __shared__ float b_s[TILE_TB_HEIGHT][TILE_N];

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos % grid_dimension_x;
		int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

        // Partial results
        float c[TILE_N];
        for (int i = 0; i < TILE_N; i++)
            c[i] = 0.0f;
        int mid = (threadIdx.x - thread_base); // TODO: check
        int m = block_id_x * TILE_M + mid;
        int n = block_id_y * TILE_N + thread_id_x;
        

        for (int i = 0; i < NORMAL_K; i += TILE_TB_HEIGHT)
        {
            float a;
            b_s[thread_id_y][thread_id_x] = B[n + (i + thread_id_y) * ldb];
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(5), "r"(128) : "memory");
            for (int j = 0; j < TILE_TB_HEIGHT; j++)
            {
                a = A[m + (i + j) * lda];
                for (int kk = 0; kk < TILE_N; kk++)
                    c[kk] += a * b_s[j][kk];
            }
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(5), "r"(128) : "memory");
        }
        int t = ldc * block_id_y * TILE_N + m;
        for (int i = 0; i < TILE_N; i++)
        {
            C[t + i * ldc] = C[t + i * ldc] * beta + alpha * c[i];
        }
    }
}


__device__ void lbm_sgemm_sgemm5(float *A, float *B, float *C, int NORMAL_M, int NORMAL_N, int NORMAL_K, 
	        int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		        int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base) {
    
    int lda = NORMAL_M;
    int ldb = NORMAL_N;
    int ldc = NORMAL_M;

    float alpha = 2.0f;
    float beta = 2.0f;

    // // ori
    // unsigned int block_pos = blockIdx.x;
    // int thread_id_x = (threadIdx.x - thread_step) % block_dimension_x;
    // int thread_id_y = (threadIdx.x - thread_step) / block_dimension_x;

    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

    __shared__ float b_s[TILE_TB_HEIGHT][TILE_N];

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos % grid_dimension_x;
		int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

        // Partial results
        float c[TILE_N];
        for (int i = 0; i < TILE_N; i++)
            c[i] = 0.0f;
        int mid = (threadIdx.x - thread_base); // TODO: check
        int m = block_id_x * TILE_M + mid;
        int n = block_id_y * TILE_N + thread_id_x;
        

        for (int i = 0; i < NORMAL_K; i += TILE_TB_HEIGHT)
        {
            float a;
            b_s[thread_id_y][thread_id_x] = B[n + (i + thread_id_y) * ldb];
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(6), "r"(128) : "memory");
            for (int j = 0; j < TILE_TB_HEIGHT; j++)
            {
                a = A[m + (i + j) * lda];
                for (int kk = 0; kk < TILE_N; kk++)
                    c[kk] += a * b_s[j][kk];
            }
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(6), "r"(128) : "memory");
        }
        int t = ldc * block_id_y * TILE_N + m;
        for (int i = 0; i < TILE_N; i++)
        {
            C[t + i * ldc] = C[t + i * ldc] * beta + alpha * c[i];
        }
    }
}

__device__ void lbm_sgemm_sgemm6(float *A, float *B, float *C, int NORMAL_M, int NORMAL_N, int NORMAL_K, 
	        int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		        int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base) {
    
    int lda = NORMAL_M;
    int ldb = NORMAL_N;
    int ldc = NORMAL_M;

    float alpha = 2.0f;
    float beta = 2.0f;

    // // ori
    // unsigned int block_pos = blockIdx.x;
    // int thread_id_x = (threadIdx.x - thread_step) % block_dimension_x;
    // int thread_id_y = (threadIdx.x - thread_step) / block_dimension_x;

    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

    __shared__ float b_s[TILE_TB_HEIGHT][TILE_N];

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos % grid_dimension_x;
		int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

        // Partial results
        float c[TILE_N];
        for (int i = 0; i < TILE_N; i++)
            c[i] = 0.0f;
        int mid = (threadIdx.x - thread_base); // TODO: check
        int m = block_id_x * TILE_M + mid;
        int n = block_id_y * TILE_N + thread_id_x;
        

        for (int i = 0; i < NORMAL_K; i += TILE_TB_HEIGHT)
        {
            float a;
            b_s[thread_id_y][thread_id_x] = B[n + (i + thread_id_y) * ldb];
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(7), "r"(128) : "memory");
            for (int j = 0; j < TILE_TB_HEIGHT; j++)
            {
                a = A[m + (i + j) * lda];
                for (int kk = 0; kk < TILE_N; kk++)
                    c[kk] += a * b_s[j][kk];
            }
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(7), "r"(128) : "memory");
        }
        int t = ldc * block_id_y * TILE_N + m;
        for (int i = 0; i < TILE_N; i++)
        {
            C[t + i * ldc] = C[t + i * ldc] * beta + alpha * c[i];
        }
    }
}


// lbm-sgemm-1-7
__global__ void mixed_lbm_sgemm_kernel_1_7(float* lbm0_srcGrid, float* lbm0_dstGrid, int lbm0_grid_dimension_x, int lbm0_grid_dimension_y, int lbm0_grid_dimension_z, int lbm0_block_dimension_x, int lbm0_block_dimension_y, int lbm0_block_dimension_z, int lbm0_ptb_start_block_pos, int lbm0_ptb_iter_block_step, int lbm0_ptb_end_block_pos, float* sgemm1_A, float* sgemm1_B, float* sgemm1_C, int sgemm1_NORMAL_M, int sgemm1_NORMAL_N, int sgemm1_NORMAL_K, int sgemm1_grid_dimension_x, int sgemm1_grid_dimension_y, int sgemm1_grid_dimension_z, int sgemm1_block_dimension_x, int sgemm1_block_dimension_y, int sgemm1_block_dimension_z, int sgemm1_ptb_start_block_pos, int sgemm1_ptb_iter_block_step, int sgemm1_ptb_end_block_pos){
    if (threadIdx.x < 128) {
        lbm_sgemm_lbm0(
            lbm0_srcGrid, lbm0_dstGrid, lbm0_grid_dimension_x, lbm0_grid_dimension_y, lbm0_grid_dimension_z, lbm0_block_dimension_x, lbm0_block_dimension_y, lbm0_block_dimension_z, lbm0_ptb_start_block_pos + 0 * lbm0_ptb_iter_block_step, lbm0_ptb_iter_block_step * 1, lbm0_ptb_end_block_pos, 0
        );
    }
    else if (threadIdx.x < 256) {
        lbm_sgemm_sgemm0(
            sgemm1_A, sgemm1_B, sgemm1_C, sgemm1_NORMAL_M, sgemm1_NORMAL_N, sgemm1_NORMAL_K, sgemm1_grid_dimension_x, sgemm1_grid_dimension_y, sgemm1_grid_dimension_z, sgemm1_block_dimension_x, sgemm1_block_dimension_y, sgemm1_block_dimension_z, sgemm1_ptb_start_block_pos + 0 * sgemm1_ptb_iter_block_step, sgemm1_ptb_iter_block_step * 7, sgemm1_ptb_end_block_pos, 128
        );
    }
    else if (threadIdx.x < 384) {
        lbm_sgemm_sgemm1(
            sgemm1_A, sgemm1_B, sgemm1_C, sgemm1_NORMAL_M, sgemm1_NORMAL_N, sgemm1_NORMAL_K, sgemm1_grid_dimension_x, sgemm1_grid_dimension_y, sgemm1_grid_dimension_z, sgemm1_block_dimension_x, sgemm1_block_dimension_y, sgemm1_block_dimension_z, sgemm1_ptb_start_block_pos + 1 * sgemm1_ptb_iter_block_step, sgemm1_ptb_iter_block_step * 7, sgemm1_ptb_end_block_pos, 256
        );
    }
    else if (threadIdx.x < 512) {
        lbm_sgemm_sgemm2(
            sgemm1_A, sgemm1_B, sgemm1_C, sgemm1_NORMAL_M, sgemm1_NORMAL_N, sgemm1_NORMAL_K, sgemm1_grid_dimension_x, sgemm1_grid_dimension_y, sgemm1_grid_dimension_z, sgemm1_block_dimension_x, sgemm1_block_dimension_y, sgemm1_block_dimension_z, sgemm1_ptb_start_block_pos + 2 * sgemm1_ptb_iter_block_step, sgemm1_ptb_iter_block_step * 7, sgemm1_ptb_end_block_pos, 384
        );
    }
    else if (threadIdx.x < 640) {
        lbm_sgemm_sgemm3(
            sgemm1_A, sgemm1_B, sgemm1_C, sgemm1_NORMAL_M, sgemm1_NORMAL_N, sgemm1_NORMAL_K, sgemm1_grid_dimension_x, sgemm1_grid_dimension_y, sgemm1_grid_dimension_z, sgemm1_block_dimension_x, sgemm1_block_dimension_y, sgemm1_block_dimension_z, sgemm1_ptb_start_block_pos + 3 * sgemm1_ptb_iter_block_step, sgemm1_ptb_iter_block_step * 7, sgemm1_ptb_end_block_pos, 512
        );
    }
    else if (threadIdx.x < 768) {
        lbm_sgemm_sgemm4(
            sgemm1_A, sgemm1_B, sgemm1_C, sgemm1_NORMAL_M, sgemm1_NORMAL_N, sgemm1_NORMAL_K, sgemm1_grid_dimension_x, sgemm1_grid_dimension_y, sgemm1_grid_dimension_z, sgemm1_block_dimension_x, sgemm1_block_dimension_y, sgemm1_block_dimension_z, sgemm1_ptb_start_block_pos + 4 * sgemm1_ptb_iter_block_step, sgemm1_ptb_iter_block_step * 7, sgemm1_ptb_end_block_pos, 640
        );
    }
    else if (threadIdx.x < 896) {
        lbm_sgemm_sgemm5(
            sgemm1_A, sgemm1_B, sgemm1_C, sgemm1_NORMAL_M, sgemm1_NORMAL_N, sgemm1_NORMAL_K, sgemm1_grid_dimension_x, sgemm1_grid_dimension_y, sgemm1_grid_dimension_z, sgemm1_block_dimension_x, sgemm1_block_dimension_y, sgemm1_block_dimension_z, sgemm1_ptb_start_block_pos + 5 * sgemm1_ptb_iter_block_step, sgemm1_ptb_iter_block_step * 7, sgemm1_ptb_end_block_pos, 768
        );
    }
    else if (threadIdx.x < 1024) {
        lbm_sgemm_sgemm6(
            sgemm1_A, sgemm1_B, sgemm1_C, sgemm1_NORMAL_M, sgemm1_NORMAL_N, sgemm1_NORMAL_K, sgemm1_grid_dimension_x, sgemm1_grid_dimension_y, sgemm1_grid_dimension_z, sgemm1_block_dimension_x, sgemm1_block_dimension_y, sgemm1_block_dimension_z, sgemm1_ptb_start_block_pos + 6 * sgemm1_ptb_iter_block_step, sgemm1_ptb_iter_block_step * 7, sgemm1_ptb_end_block_pos, 896
        );
    }

}
