#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hipblas.h>
#include <mma.h>
#include <malloc.h>
#include <sys/time.h>
using namespace nvcuda; 
#include "header/mriq_header.h"
#include "header/fft_header.h"

// fft
__device__ void G_GPU_exchange_fft_mriq_fft0( float2* v, int stride, int idxD, int incD, 
	int idxS, int incS){ 
	__shared__ float work[FFT_T*FFT_R*2];//FFT_T*FFT_R*2
	float* sr = work;
	float* si = work+FFT_T*FFT_R;  
	// __syncthreads(); 
asm volatile("bar.sync %0, %1;" : : "r"(1), "r"(128) : "memory");
	for( int r=0; r<FFT_R; r++ ) { 
		int i = (idxD + r*incD)*stride; 
		sr[i] = v[r].x;
		si[i] = v[r].y;  
	}   
	// __syncthreads(); 
asm volatile("bar.sync %0, %1;" : : "r"(1), "r"(128) : "memory");

	for( int r=0; r<FFT_R; r++ ) { 
		int i = (idxS + r*incS)*stride;     
		v[r] = make_float2(sr[i], si[i]);  
	}        
}  

__device__ void G_GPU_DoFft_fft_mriq_fft0(float2* v, int j, int stride=1) { 
	for( int Ns=1; Ns<FFT_N; Ns*=FFT_R ){ 
		float angle = -2*M_PI*(j%Ns)/(Ns*FFT_R); 
		for( int r=0; r<FFT_R; r++ ){
			v[r] = v[r]*make_float2(cos(r*angle), sin(r*angle));
		}

		GPU_FFT2( v );

		int idxD = GPU_expand(j,Ns,FFT_R); 
		int idxS = GPU_expand(j,FFT_N/FFT_R,FFT_R); 
		G_GPU_exchange_fft_mriq_fft0( v,stride, idxD,Ns, idxS,FFT_N/FFT_R);
	}      
}

__device__ void fft_mriq_fft0(float2* data, 
	int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base){
	
	unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

	// // ori
	// int thread_id_x = threadIdx.x - thread_step;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    // int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

	for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

		// // ori
		// int block_id_x = block_pos;
        int block_id_x = block_pos % grid_dimension_x;
		// int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

		float2 *ori_data = data + block_id_x * FFT_N;
		float2 v[FFT_R];
		// data = ori_data;

		int idxG = thread_id_x; 
		for (int r=0; r<FFT_R; r++) {  
			v[r] = ori_data[idxG + r*FFT_T];
		} 
		G_GPU_DoFft_fft_mriq_fft0( v, thread_id_x, 1);  
		for (int r=0; r<FFT_R; r++) {
			ori_data[idxG + r*FFT_T] = v[r];
		}
	}
}

__device__ void G_GPU_exchange_fft_mriq_fft1( float2* v, int stride, int idxD, int incD, 
	int idxS, int incS){ 
	__shared__ float work[FFT_T*FFT_R*2];//FFT_T*FFT_R*2
	float* sr = work;
	float* si = work+FFT_T*FFT_R;  
	// __syncthreads(); 
asm volatile("bar.sync %0, %1;" : : "r"(2), "r"(128) : "memory");
	for( int r=0; r<FFT_R; r++ ) { 
		int i = (idxD + r*incD)*stride; 
		sr[i] = v[r].x;
		si[i] = v[r].y;  
	}   
	// __syncthreads(); 
asm volatile("bar.sync %0, %1;" : : "r"(2), "r"(128) : "memory");

	for( int r=0; r<FFT_R; r++ ) { 
		int i = (idxS + r*incS)*stride;     
		v[r] = make_float2(sr[i], si[i]);  
	}        
}  

__device__ void G_GPU_DoFft_fft_mriq_fft1(float2* v, int j, int stride=1) { 
	for( int Ns=1; Ns<FFT_N; Ns*=FFT_R ){ 
		float angle = -2*M_PI*(j%Ns)/(Ns*FFT_R); 
		for( int r=0; r<FFT_R; r++ ){
			v[r] = v[r]*make_float2(cos(r*angle), sin(r*angle));
		}

		GPU_FFT2( v );

		int idxD = GPU_expand(j,Ns,FFT_R); 
		int idxS = GPU_expand(j,FFT_N/FFT_R,FFT_R); 
		G_GPU_exchange_fft_mriq_fft1( v,stride, idxD,Ns, idxS,FFT_N/FFT_R);
	}      
}

__device__ void fft_mriq_fft1(float2* data, 
	int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base){
	
	unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

	// // ori
	// int thread_id_x = threadIdx.x - thread_step;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    // int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

	for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

		// // ori
		// int block_id_x = block_pos;
        int block_id_x = block_pos % grid_dimension_x;
		// int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

		float2 *ori_data = data + block_id_x * FFT_N;
		float2 v[FFT_R];
		// data = ori_data;

		int idxG = thread_id_x; 
		for (int r=0; r<FFT_R; r++) {  
			v[r] = ori_data[idxG + r*FFT_T];
		} 
		G_GPU_DoFft_fft_mriq_fft1( v, thread_id_x);  
		for (int r=0; r<FFT_R; r++) {
			ori_data[idxG + r*FFT_T] = v[r];
		}
	}
}
// step_size == launch param == ptb worker num == SM_NUM * ptb_per_sm_number

__device__ void G_GPU_exchange_fft_mriq_fft2( float2* v, int stride, int idxD, int incD, 
	int idxS, int incS){ 
	__shared__ float work[FFT_T*FFT_R*2];//FFT_T*FFT_R*2
	float* sr = work;
	float* si = work+FFT_T*FFT_R;  
	// __syncthreads(); 
asm volatile("bar.sync %0, %1;" : : "r"(3), "r"(128) : "memory");
	for( int r=0; r<FFT_R; r++ ) { 
		int i = (idxD + r*incD)*stride; 
		sr[i] = v[r].x;
		si[i] = v[r].y;  
	}   
	// __syncthreads(); 
asm volatile("bar.sync %0, %1;" : : "r"(3), "r"(128) : "memory");

	for( int r=0; r<FFT_R; r++ ) { 
		int i = (idxS + r*incS)*stride;     
		v[r] = make_float2(sr[i], si[i]);  
	}        
}  

__device__ void G_GPU_DoFft_fft_mriq_fft2(float2* v, int j, int stride=1) { 
	for( int Ns=1; Ns<FFT_N; Ns*=FFT_R ){ 
		float angle = -2*M_PI*(j%Ns)/(Ns*FFT_R); 
		for( int r=0; r<FFT_R; r++ ){
			v[r] = v[r]*make_float2(cos(r*angle), sin(r*angle));
		}

		GPU_FFT2( v );

		int idxD = GPU_expand(j,Ns,FFT_R); 
		int idxS = GPU_expand(j,FFT_N/FFT_R,FFT_R); 
		G_GPU_exchange_fft_mriq_fft2( v,stride, idxD,Ns, idxS,FFT_N/FFT_R);
	}      
}

__device__ void fft_mriq_fft2(float2* data, 
	int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base){
	
	unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

	// // ori
	// int thread_id_x = threadIdx.x - thread_step;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    // int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

	for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

		// // ori
		// int block_id_x = block_pos;
        int block_id_x = block_pos % grid_dimension_x;
		// int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

		float2 *ori_data = data + block_id_x * FFT_N;
		float2 v[FFT_R];
		// data = ori_data;

		int idxG = thread_id_x; 
		for (int r=0; r<FFT_R; r++) {  
			v[r] = ori_data[idxG + r*FFT_T];
		} 
		G_GPU_DoFft_fft_mriq_fft2( v, thread_id_x);  
		for (int r=0; r<FFT_R; r++) {
			ori_data[idxG + r*FFT_T] = v[r];
		}
	}
}

// mriq
__device__ void fft_mriq_mriq0(int numK, int kGlobalIndex, float* x, float* y, float* z, float* Qr , float* Qi,
	    int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		    int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base) {

    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;
	
	int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    // int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }
        
        int block_id_x = block_pos % grid_dimension_x;
		// int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

        for (int QGrid = 0; QGrid < 1; QGrid++) {
            kGlobalIndex = QGrid * KERNEL_Q_K_ELEMS_PER_GRID;

            float sX;
            float sY;
            float sZ;
            float sQr;
            float sQi;

            // Determine the element of the X arrays computed by this thread
            int xIndex = block_id_x * KERNEL_Q_THREADS_PER_BLOCK + thread_id_x;

            // Read block's X values from global mem to shared mem
            sX = x[xIndex];
            sY = y[xIndex];
            sZ = z[xIndex];
            sQr = Qr[xIndex];
            sQi = Qi[xIndex];

            // Loop over all elements of K in constant mem to compute a partial value
            // for X.
            int kIndex = 0;
            // if (numK % 2) {
            //     float expArg = PIx2_MRIQ * (ck[0].Kx * sX + ck[0].Ky * sY + ck[0].Kz * sZ);
            //     sQr += ck[0].PhiMag * cos(expArg);
            //     sQi += ck[0].PhiMag * sin(expArg);
            //     kIndex++;
            //     kGlobalIndex++;
            // }

            for (; (kIndex < KERNEL_Q_K_ELEMS_PER_GRID) && (kGlobalIndex < numK);
                kIndex += 2, kGlobalIndex += 2) {
                float expArg = PIx2_MRIQ * (ck[kIndex].Kx * sX + ck[kIndex].Ky * sY +
                            ck[kIndex].Kz * sZ);
                sQr += ck[kIndex].PhiMag * cos(expArg);
                sQi += ck[kIndex].PhiMag * sin(expArg);

                int kIndex1 = kIndex + 1;
                float expArg1 = PIx2_MRIQ * (ck[kIndex1].Kx * sX + ck[kIndex1].Ky * sY +
                            ck[kIndex1].Kz * sZ);
                sQr += ck[kIndex1].PhiMag * cos(expArg1);
                sQi += ck[kIndex1].PhiMag * sin(expArg1);
            }

            Qr[xIndex] = sQr;
            Qi[xIndex] = sQi;
        }
    }
}

__device__ void fft_mriq_mriq1(int numK, int kGlobalIndex, float* x, float* y, float* z, float* Qr , float* Qi,
	    int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		    int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base) {

    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;
	
	int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    // int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }
        
        int block_id_x = block_pos % grid_dimension_x;
		// int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

        for (int QGrid = 0; QGrid < 1; QGrid++) {
            kGlobalIndex = QGrid * KERNEL_Q_K_ELEMS_PER_GRID;

            float sX;
            float sY;
            float sZ;
            float sQr;
            float sQi;

            // Determine the element of the X arrays computed by this thread
            int xIndex = block_id_x * KERNEL_Q_THREADS_PER_BLOCK + thread_id_x;

            // Read block's X values from global mem to shared mem
            sX = x[xIndex];
            sY = y[xIndex];
            sZ = z[xIndex];
            sQr = Qr[xIndex];
            sQi = Qi[xIndex];

            // Loop over all elements of K in constant mem to compute a partial value
            // for X.
            int kIndex = 0;
            // if (numK % 2) {
            //     float expArg = PIx2_MRIQ * (ck[0].Kx * sX + ck[0].Ky * sY + ck[0].Kz * sZ);
            //     sQr += ck[0].PhiMag * cos(expArg);
            //     sQi += ck[0].PhiMag * sin(expArg);
            //     kIndex++;
            //     kGlobalIndex++;
            // }

            for (; (kIndex < KERNEL_Q_K_ELEMS_PER_GRID) && (kGlobalIndex < numK);
                kIndex += 2, kGlobalIndex += 2) {
                float expArg = PIx2_MRIQ * (ck[kIndex].Kx * sX + ck[kIndex].Ky * sY +
                            ck[kIndex].Kz * sZ);
                sQr += ck[kIndex].PhiMag * cos(expArg);
                sQi += ck[kIndex].PhiMag * sin(expArg);

                int kIndex1 = kIndex + 1;
                float expArg1 = PIx2_MRIQ * (ck[kIndex1].Kx * sX + ck[kIndex1].Ky * sY +
                            ck[kIndex1].Kz * sZ);
                sQr += ck[kIndex1].PhiMag * cos(expArg1);
                sQi += ck[kIndex1].PhiMag * sin(expArg1);
            }

            Qr[xIndex] = sQr;
            Qi[xIndex] = sQi;
        }
    }
}

// fft-mriq-3-2
__global__ void mixed_fft_mriq_kernel_3_2(float2* fft0_data, int fft0_grid_dimension_x, int fft0_grid_dimension_y, int fft0_grid_dimension_z, int fft0_block_dimension_x, int fft0_block_dimension_y, int fft0_block_dimension_z, int fft0_ptb_start_block_pos, int fft0_ptb_iter_block_step, int fft0_ptb_end_block_pos, int mriq1_numK, int mriq1_kGlobalIndex, float* mriq1_x, float* mriq1_y, float* mriq1_z, float* mriq1_Qr, float* mriq1_Qi, int mriq1_grid_dimension_x, int mriq1_grid_dimension_y, int mriq1_grid_dimension_z, int mriq1_block_dimension_x, int mriq1_block_dimension_y, int mriq1_block_dimension_z, int mriq1_ptb_start_block_pos, int mriq1_ptb_iter_block_step, int mriq1_ptb_end_block_pos){
    if (threadIdx.x < 128) {
        fft_mriq_fft0(
            fft0_data, fft0_grid_dimension_x, fft0_grid_dimension_y, fft0_grid_dimension_z, fft0_block_dimension_x, fft0_block_dimension_y, fft0_block_dimension_z, fft0_ptb_start_block_pos + 0 * fft0_ptb_iter_block_step, fft0_ptb_iter_block_step * 3, fft0_ptb_end_block_pos, 0
        );
    }
    else if (threadIdx.x < 256) {
        fft_mriq_fft1(
            fft0_data, fft0_grid_dimension_x, fft0_grid_dimension_y, fft0_grid_dimension_z, fft0_block_dimension_x, fft0_block_dimension_y, fft0_block_dimension_z, fft0_ptb_start_block_pos + 1 * fft0_ptb_iter_block_step, fft0_ptb_iter_block_step * 3, fft0_ptb_end_block_pos, 128
        );
    }
    else if (threadIdx.x < 384) {
        fft_mriq_fft2(
            fft0_data, fft0_grid_dimension_x, fft0_grid_dimension_y, fft0_grid_dimension_z, fft0_block_dimension_x, fft0_block_dimension_y, fft0_block_dimension_z, fft0_ptb_start_block_pos + 2 * fft0_ptb_iter_block_step, fft0_ptb_iter_block_step * 3, fft0_ptb_end_block_pos, 256
        );
    }
    else if (threadIdx.x < 640) {
        fft_mriq_mriq0(
            mriq1_numK, mriq1_kGlobalIndex, mriq1_x, mriq1_y, mriq1_z, mriq1_Qr, mriq1_Qi, mriq1_grid_dimension_x, mriq1_grid_dimension_y, mriq1_grid_dimension_z, mriq1_block_dimension_x, mriq1_block_dimension_y, mriq1_block_dimension_z, mriq1_ptb_start_block_pos + 0 * mriq1_ptb_iter_block_step, mriq1_ptb_iter_block_step * 2, mriq1_ptb_end_block_pos, 384
        );
    }
    else if (threadIdx.x < 896) {
        fft_mriq_mriq1(
            mriq1_numK, mriq1_kGlobalIndex, mriq1_x, mriq1_y, mriq1_z, mriq1_Qr, mriq1_Qi, mriq1_grid_dimension_x, mriq1_grid_dimension_y, mriq1_grid_dimension_z, mriq1_block_dimension_x, mriq1_block_dimension_y, mriq1_block_dimension_z, mriq1_ptb_start_block_pos + 1 * mriq1_ptb_iter_block_step, mriq1_ptb_iter_block_step * 2, mriq1_ptb_end_block_pos, 640
        );
    }

}
