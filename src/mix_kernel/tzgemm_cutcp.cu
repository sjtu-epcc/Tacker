#include "hip/hip_runtime.h"
#include "header/cutcp_header.h"
#include "header/tzgemm_header.h"
#include <mma.h>
using namespace nvcuda; 
__device__ void cutcp_tzgemm_cutcp0(
    int binDim_x,
    int binDim_y,
    float4 *binZeroAddr,    /* address of atom bins starting at origin */
    float h,                /* lattice spacing */
    float cutoff2,          /* square of cutoff distance */
    float inv_cutoff2,
    float *regionZeroAddr, /* address of lattice regions starting at origin */
    int zRegionIndex_t,
    int grid_dimension_x,
    int grid_dimension_y,
    int grid_dimension_z,
    int block_dimension_x,
    int block_dimension_y,
    int block_dimension_z,
    int ptb_start_block_pos,
    int ptb_iter_block_step,
    int ptb_end_block_pos,
    int thread_base
    ) {
    // unsigned int block_pos = blockIdx.x + SM_NUM * 2; // TODO: why SM_NUM * 2?
    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) / (block_dimension_y * block_dimension_z);
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_z) % block_dimension_y;
    int thread_id_z = (threadIdx.x - thread_base) % block_dimension_z;

    // // ori
    // int thread_id_x = (threadIdx.x - thread_step) / (block_dimension_y * block_dimension_z);
    // int thread_id_y = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) / block_dimension_z;
    // int thread_id_z = ((threadIdx.x - thread_step) % (block_dimension_y * block_dimension_z)) % block_dimension_z;

    /* thread id */
	const int tid = (thread_id_z * block_dimension_y + thread_id_y) * block_dimension_x + thread_id_x;

    __shared__ float AtomBinCache[BIN_CACHE_MAXLEN * BIN_DEPTH * 4];
	// __shared__ float *myRegionAddr;
	// __shared__ int3 myBinIndex;

    float *myRegionAddr;
	int3 myBinIndex;

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos / (grid_dimension_y * grid_dimension_z);
        int block_id_y = (block_pos / grid_dimension_z) % grid_dimension_y;
        int block_id_z = block_pos % grid_dimension_z;


        int xRegionIndex = block_id_x;
        int yRegionIndex = block_id_y;
        int zRegionIndex = block_id_z;
    
        /* neighbor index */
        int nbrid;

        /* this is the start of the sub-region indexed by tid */
        myRegionAddr = regionZeroAddr + ((zRegionIndex*grid_dimension_y + yRegionIndex)*grid_dimension_x + xRegionIndex)*REGION_SIZE;

        /* spatial coordinate of this lattice point */
        float x = (8 * xRegionIndex + thread_id_x) * h;
        float y = (8 * yRegionIndex + thread_id_y) * h;
        float z = (8 * zRegionIndex + thread_id_z) * h;

        int totalbins = 0;
        int numbins;

        /* bin number determined by center of region */
        myBinIndex.x = (int) floorf((8 * xRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.y = (int) floorf((8 * yRegionIndex + 4) * h * BIN_INVLEN);
        myBinIndex.z = (int) floorf((8 * zRegionIndex + 4) * h * BIN_INVLEN);

        /* first neighbor in list for me to cache */
        nbrid = (tid >> 4);

        numbins = BIN_CACHE_MAXLEN;

        float energy0 = 0.f;
        float energy1 = 0.f;
        float energy2 = 0.f;
        float energy3 = 0.f;


        for (totalbins = 0;  totalbins < NbrListLen;  totalbins += numbins) {
            int bincnt;

            /* start of where to write in shared memory */
            int startoff = BIN_SIZE * (tid >> 4);

            /* each half-warp to cache up to 4 atom bins */
            for (bincnt = 0;  bincnt < 4 && nbrid < NbrListLen;  bincnt++, nbrid += 8) {
                int i = myBinIndex.x + NbrList[nbrid].x;
                int j = myBinIndex.y + NbrList[nbrid].y;
                int k = myBinIndex.z + NbrList[nbrid].z;

                /* determine global memory location of atom bin */
                float *p_global = ((float *) binZeroAddr) + (((__mul24(k, binDim_y) + j)*binDim_x + i) << BIN_SHIFT);

                /* coalesced read from global memory -
                * retain same ordering in shared memory for now */
                int binIndex = startoff + (bincnt << (3 + BIN_SHIFT));
                int tidmask = tid & 15;

                AtomBinCache[binIndex + tidmask   ] = p_global[tidmask   ];
                AtomBinCache[binIndex + tidmask+16] = p_global[tidmask+16];
            }
        //    __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(1), "r"(128) : "memory");

            /* no warp divergence */
            if (totalbins + BIN_CACHE_MAXLEN > NbrListLen) {
                numbins = NbrListLen - totalbins;
            }

            int stopbin = (numbins << BIN_SHIFT);
            for (bincnt = 0; bincnt < stopbin; bincnt+=BIN_SIZE) {
                for (int i = 0;  i < BIN_DEPTH;  i++) {
                    int off = bincnt + (i<<2);

                    float aq = AtomBinCache[off + 3];
                    if (0.f == aq) 
                        break;  /* no more atoms in bin */

                    float dx = AtomBinCache[off    ] - x;
                    float dz = AtomBinCache[off + 2] - z;
                    float dxdz2 = dx*dx + dz*dz;
                    float dy = AtomBinCache[off + 1] - y;
                    float r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy0 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy1 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy2 += aq * rsqrtf(r2) * s * s;
                    }
                    dy -= 2.0f*h;
                    r2 = dy*dy + dxdz2;

                    if (r2 < cutoff2) {
                        float s = (1.f - r2 * inv_cutoff2);
                        energy3 += aq * rsqrtf(r2) * s * s;
                    }
                } /* end loop over atoms in bin */
            } /* end loop over cached atom bins */
        //    __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(1), "r"(128) : "memory");
        } /* end loop over neighbor list */

        /* store into global memory */
        myRegionAddr[(tid>>4)*64 + (tid&15)     ] = energy0;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 16] = energy1;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 32] = energy2;
        myRegionAddr[(tid>>4)*64 + (tid&15) + 48] = energy3;
    }
}

__device__ void cutcp_tzgemm_tzgemm0(half *A, half *B, float *C, 
		// float alpha, float beta,
		int M_GLOBAL, int N_GLOBAL, int K_GLOBAL,
		int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		        int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base) {

	__shared__ half shmem[BLOCK_COL_TILES * WMMA_M * 2][CHUNK_K * WMMA_K + SKEW_HALF];
	// extern __shared__ half shmem[][CHUNK_K * WMMA_K + SKEW_HALF];

	const unsigned int N_TILES = N_GLOBAL / WMMA_N;
	const unsigned int K_TILES = K_GLOBAL / WMMA_K;
	// const unsigned int M_TILES = M_GLOBAL / WMMA_M;

	float alpha = alpha_g;
	float beta = beta_g;

	unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;

	// Warp and lane identification.
	const unsigned int warpId = thread_id_x / WARP_SIZE;
	const unsigned int laneId = thread_id_x % WARP_SIZE;

	// Offset in shared memory from which the B matrix is stored.
	const size_t shmem_idx_b_off = BLOCK_COL_TILES * WMMA_M;
	// This pointer is used to access the C and D matrix tiles this warp computes.
	float *shmem_warp_tile_ptr = (float *)&shmem[0][0] +
								(warpId / 2) * SHMEM_STRIDE * WMMA_M * 2 +
								(warpId % 2) * SHMEM_OFFSET;

	// This pointer is used to stream the C and D matrices block-wide tile to and
	// from shared memory.
	float *shmem_warp_stream_ptr = (float *)&shmem[0][0] + warpId * SHMEM_STRIDE * WMMA_M;

	// Adjust the beta scaler, as it'll be multiplied by alpha at the end of
	// each tile computation. Technically this is not generally correct (may
	// result in a loss of precision). Zero still needs to be specially handled
	// though.
	beta /= alpha;

	// Each CTA slides along the 128 x 128 tiles from the top left corner of the
	// matrix to the right and down, and selects the next tile to compute. Once
	// there's no such tile, all warps in this CTA exit.
	for (;; block_pos += ptb_iter_block_step) {
		if (block_pos >= ptb_end_block_pos) {
            return;
        }

		const unsigned int block_tile_i =
			((block_pos * BLOCK_ROW_TILES) / N_TILES) * (BLOCK_COL_TILES);
		const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES) % N_TILES;
		// This warp's pointer to the C matrix data to copy memory from to shared
		// memory.
		const size_t gmem_idx =
			(block_tile_i + warpId) * WMMA_M * GLOBAL_MEM_STRIDE + block_tile_j * WMMA_N;


			// These fragments will accumulate the result of A and B matrix fragment
			// multiplications along the K_GLOBAL dimension.
			wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c[WARP_COL_TILES][WARP_ROW_TILES];
			#pragma unroll
			for (int i = 0; i < WARP_COL_TILES; i++) {
				#pragma unroll
				for (int j = 0; j < WARP_ROW_TILES; j++) {
					wmma::fill_fragment(c[i][j], 0.0f);
				}
			}

			// Select what warp copies what matrix to shared memory.
			// Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
			const half *warp_ptr = 
				warpId < (WARPS_PER_BLOCK / 2) 
					? (&A[block_tile_i * WMMA_M * K_GLOBAL] + WMMA_M * K_GLOBAL * (warpId % (WARPS_PER_BLOCK / 2)) * 2)
					: (&B[block_tile_j * WMMA_N * K_GLOBAL] + WMMA_N * K_GLOBAL * (warpId % (WARPS_PER_BLOCK / 2)) * 2);

			// Go through the global K dimension by a fixed step at a time.
			#pragma unroll
			for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
				// Copy slices of the A and B matrices to shared memory.
				// The first half of the warps in the CTA copy the A matrix, 
				// the rest copy the B matrix.
				size_t shmem_idx =
					warpId < (WARPS_PER_BLOCK / 2)
						? (WMMA_M * (warpId % (WARPS_PER_BLOCK / 2)) * 2)
						: (WMMA_N * (warpId % (WARPS_PER_BLOCK / 2)) * 2 + shmem_idx_b_off);

				// First half of the warp copies the first row / column of the matrix,
				// the second half of the warp copies the next.
				int4 *lane_ptr = (int4 *)(warp_ptr + tile_k * WMMA_K + (laneId / CHUNK_COPY_LINE_LANES) * K_GLOBAL) 
					+ (laneId % CHUNK_COPY_LINE_LANES);

				// Shift the second half of the warp to the next row / column in the
				// shared memory.
				shmem_idx += laneId / CHUNK_COPY_LINE_LANES;

				#pragma unroll
				for (int i = 0; i < ((WARP_SIZE / 2) / CHUNK_COPY_LINES_PER_WARP) * 2; i++) {
					// Copy 16 bytes at once in each lane.
					*((int4 *)&shmem[shmem_idx][0] + (laneId % CHUNK_COPY_LINE_LANES)) =
						*lane_ptr;

					// Advance the global memory pointer and the shared memory index.
					lane_ptr =
						(int4 *)((half *)lane_ptr + K_GLOBAL * CHUNK_COPY_LINES_PER_WARP);
					shmem_idx += CHUNK_COPY_LINES_PER_WARP;
				}

				asm volatile("bar.sync %0, %1;" : : "r"(2), "r"(128) : "memory");;

				// Compute a grid of C matrix tiles in each warp.
				#pragma unroll
				for (int k_step = 0; k_step < CHUNK_K; k_step++) {
					wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a[WARP_COL_TILES];
					wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b[WARP_ROW_TILES];

					#pragma unroll
					for (int i = 0; i < WARP_COL_TILES; i++) {
						size_t shmem_idx_a = (warpId / 2) * WMMA_M * 2 + (i * WMMA_M);
						const half *tile_ptr = &shmem[shmem_idx_a][k_step * WMMA_K];
						wmma::load_matrix_sync(a[i], tile_ptr, WMMA_K * CHUNK_K + SKEW_HALF);

						#pragma unroll
						for (int j = 0; j < WARP_ROW_TILES; j++) {
							if (i == 0) {
								// Load the B matrix fragment once, because it is going to be
								// reused against the other A matrix fragments.
								size_t shmem_idx_b = shmem_idx_b_off + (WARP_ROW_TILES * WMMA_N) * (warpId % 2) + (j * WMMA_N);
								const half *tile_ptr = &shmem[shmem_idx_b][k_step * WMMA_K];
								wmma::load_matrix_sync(b[j], tile_ptr, WMMA_K * CHUNK_K + SKEW_HALF);
							}
							wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
						}
					}
				}
				asm volatile("bar.sync %0, %1;" : : "r"(2), "r"(128) : "memory");;
			}

			// Store the D fragments to shared memory.
			#pragma unroll
			for (int i = 0; i < WARP_COL_TILES; i++) {
				#pragma unroll
				for (int j = 0; j < WARP_ROW_TILES; j++) {
					// Uniform, point-wise transformations of ALL fragment elements by ALL
					// threads in the warp are well-defined even though element indices
					// within fragment storage are not defined.
					#pragma unroll
					for (int t = 0; t < c[i][j].num_elements; t++) c[i][j].x[t] *= alpha;

					float *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * WMMA_K + j * WMMA_N;
					wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE, C_LAYOUT);
				}
			}

			asm volatile("bar.sync %0, %1;" : : "r"(2), "r"(128) : "memory");;

			// Now that shared memory contains all the D tiles, stream them to global
			// memory.
			float *dst_gmem_warp_stream_ptr = &C[gmem_idx];

			#pragma unroll
			for (int i = 0; i < 16; i++) {
				*((int2 *)(dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId) =
					*((int2 *)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId);
			}
			asm volatile("bar.sync %0, %1;" : : "r"(2), "r"(128) : "memory");;
		}
}

__global__ void cutcp_tzgemm_mix(
        int cutcp0_binDim_x, int cutcp0_binDim_y, float4* cutcp0_binZeroAddr, float cutcp0_h, float cutcp0_cutoff2, float cutcp0_inv_cutoff2, float* cutcp0_regionZeroAddr, int cutcp0_zRegionIndex_t, 
        int cutcp0_grid_dimension_x, int cutcp0_grid_dimension_y, int cutcp0_grid_dimension_z, int cutcp0_block_dimension_x, int cutcp0_block_dimension_y, int cutcp0_block_dimension_z, int cutcp0_ptb_start_block_pos, int cutcp0_ptb_iter_block_step, int cutcp0_ptb_end_block_pos, 
            half *tzgemm1_A, half *tzgemm1_B, float *tzgemm1_C, int tzgemm1_NORMAL_M, int tzgemm1_NORMAL_N, int tzgemm1_NORMAL_K,
            int tzgemm1_grid_dimension_x, int tzgemm1_grid_dimension_y, int tzgemm1_grid_dimension_z, int tzgemm1_block_dimension_x, int tzgemm1_block_dimension_y, int tzgemm1_block_dimension_z, int tzgemm1_ptb_start_block_pos, int tzgemm1_ptb_iter_block_step, int tzgemm1_ptb_end_block_pos){
    if (threadIdx.x < 128) {
        cutcp_tzgemm_cutcp0(
            cutcp0_binDim_x, cutcp0_binDim_y, cutcp0_binZeroAddr, cutcp0_h, cutcp0_cutoff2, cutcp0_inv_cutoff2, cutcp0_regionZeroAddr, cutcp0_zRegionIndex_t, cutcp0_grid_dimension_x, cutcp0_grid_dimension_y, cutcp0_grid_dimension_z, cutcp0_block_dimension_x, cutcp0_block_dimension_y, cutcp0_block_dimension_z, cutcp0_ptb_start_block_pos + 0 * cutcp0_ptb_iter_block_step, cutcp0_ptb_iter_block_step * 1, cutcp0_ptb_end_block_pos, 0
        );
    }
    else if (threadIdx.x < 256) {
        cutcp_tzgemm_tzgemm0(
            tzgemm1_A, tzgemm1_B, tzgemm1_C, tzgemm1_NORMAL_M, tzgemm1_NORMAL_N, tzgemm1_NORMAL_K, tzgemm1_grid_dimension_x, tzgemm1_grid_dimension_y, tzgemm1_grid_dimension_z, tzgemm1_block_dimension_x, tzgemm1_block_dimension_y, tzgemm1_block_dimension_z, tzgemm1_ptb_start_block_pos + 0 * tzgemm1_ptb_iter_block_step, tzgemm1_ptb_iter_block_step * 1, tzgemm1_ptb_end_block_pos, 128
        );
    }

}
