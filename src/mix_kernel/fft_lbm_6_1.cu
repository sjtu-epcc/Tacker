#include "hip/hip_runtime.h"
#include "header/lbm_header.h"
#include "header/fft_header.h"

// fft
__device__ void G_GPU_exchange_fft_lbm_fft0( float2* v, int stride, int idxD, int incD, 
	int idxS, int incS){ 
	__shared__ float work[FFT_T*FFT_R*2];//FFT_T*FFT_R*2
	float* sr = work;
	float* si = work+FFT_T*FFT_R;  
	// __syncthreads(); 
asm volatile("bar.sync %0, %1;" : : "r"(1), "r"(128) : "memory");
	for( int r=0; r<FFT_R; r++ ) { 
		int i = (idxD + r*incD)*stride; 
		sr[i] = v[r].x;
		si[i] = v[r].y;  
	}   
	// __syncthreads(); 
asm volatile("bar.sync %0, %1;" : : "r"(1), "r"(128) : "memory");

	for( int r=0; r<FFT_R; r++ ) { 
		int i = (idxS + r*incS)*stride;     
		v[r] = make_float2(sr[i], si[i]);  
	}        
}  

__device__ void G_GPU_DoFft_fft_lbm_fft0(float2* v, int j, int stride=1) { 
	for( int Ns=1; Ns<FFT_N; Ns*=FFT_R ){ 
		float angle = -2*M_PI*(j%Ns)/(Ns*FFT_R); 
		for( int r=0; r<FFT_R; r++ ){
			v[r] = v[r]*make_float2(cos(r*angle), sin(r*angle));
		}

		GPU_FFT2( v );

		int idxD = GPU_expand(j,Ns,FFT_R); 
		int idxS = GPU_expand(j,FFT_N/FFT_R,FFT_R); 
		G_GPU_exchange_fft_lbm_fft0( v,stride, idxD,Ns, idxS,FFT_N/FFT_R);
	}      
}

__device__ void fft_lbm_fft0(float2* data, 
	int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base){
	
	unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

	// // ori
	// int thread_id_x = threadIdx.x - thread_step;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    // int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

	for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

		// // ori
		// int block_id_x = block_pos;
        int block_id_x = block_pos % grid_dimension_x;
		// int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

		float2 *ori_data = data + block_id_x * FFT_N;
		float2 v[FFT_R];
		// data = ori_data;

		int idxG = thread_id_x; 
		for (int r=0; r<FFT_R; r++) {  
			v[r] = ori_data[idxG + r*FFT_T];
		} 
		G_GPU_DoFft_fft_lbm_fft0( v, thread_id_x, 1);  
		for (int r=0; r<FFT_R; r++) {
			ori_data[idxG + r*FFT_T] = v[r];
		}
	}
}

__device__ void G_GPU_exchange_fft_lbm_fft1( float2* v, int stride, int idxD, int incD, 
	int idxS, int incS){ 
	__shared__ float work[FFT_T*FFT_R*2];//FFT_T*FFT_R*2
	float* sr = work;
	float* si = work+FFT_T*FFT_R;  
	// __syncthreads(); 
asm volatile("bar.sync %0, %1;" : : "r"(2), "r"(128) : "memory");
	for( int r=0; r<FFT_R; r++ ) { 
		int i = (idxD + r*incD)*stride; 
		sr[i] = v[r].x;
		si[i] = v[r].y;  
	}   
	// __syncthreads(); 
asm volatile("bar.sync %0, %1;" : : "r"(2), "r"(128) : "memory");

	for( int r=0; r<FFT_R; r++ ) { 
		int i = (idxS + r*incS)*stride;     
		v[r] = make_float2(sr[i], si[i]);  
	}        
}  

__device__ void G_GPU_DoFft_fft_lbm_fft1(float2* v, int j, int stride=1) { 
	for( int Ns=1; Ns<FFT_N; Ns*=FFT_R ){ 
		float angle = -2*M_PI*(j%Ns)/(Ns*FFT_R); 
		for( int r=0; r<FFT_R; r++ ){
			v[r] = v[r]*make_float2(cos(r*angle), sin(r*angle));
		}

		GPU_FFT2( v );

		int idxD = GPU_expand(j,Ns,FFT_R); 
		int idxS = GPU_expand(j,FFT_N/FFT_R,FFT_R); 
		G_GPU_exchange_fft_lbm_fft1( v,stride, idxD,Ns, idxS,FFT_N/FFT_R);
	}      
}

__device__ void fft_lbm_fft1(float2* data, 
	int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base){
	
	unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

	// // ori
	// int thread_id_x = threadIdx.x - thread_step;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    // int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

	for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

		// // ori
		// int block_id_x = block_pos;
        int block_id_x = block_pos % grid_dimension_x;
		// int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

		float2 *ori_data = data + block_id_x * FFT_N;
		float2 v[FFT_R];
		// data = ori_data;

		int idxG = thread_id_x; 
		for (int r=0; r<FFT_R; r++) {  
			v[r] = ori_data[idxG + r*FFT_T];
		} 
		G_GPU_DoFft_fft_lbm_fft1( v, thread_id_x);  
		for (int r=0; r<FFT_R; r++) {
			ori_data[idxG + r*FFT_T] = v[r];
		}
	}
}
// step_size == launch param == ptb worker num == SM_NUM * ptb_per_sm_number

__device__ void G_GPU_exchange_fft_lbm_fft2( float2* v, int stride, int idxD, int incD, 
	int idxS, int incS){ 
	__shared__ float work[FFT_T*FFT_R*2];//FFT_T*FFT_R*2
	float* sr = work;
	float* si = work+FFT_T*FFT_R;  
	// __syncthreads(); 
asm volatile("bar.sync %0, %1;" : : "r"(3), "r"(128) : "memory");
	for( int r=0; r<FFT_R; r++ ) { 
		int i = (idxD + r*incD)*stride; 
		sr[i] = v[r].x;
		si[i] = v[r].y;  
	}   
	// __syncthreads(); 
asm volatile("bar.sync %0, %1;" : : "r"(3), "r"(128) : "memory");

	for( int r=0; r<FFT_R; r++ ) { 
		int i = (idxS + r*incS)*stride;     
		v[r] = make_float2(sr[i], si[i]);  
	}        
}  

__device__ void G_GPU_DoFft_fft_lbm_fft2(float2* v, int j, int stride=1) { 
	for( int Ns=1; Ns<FFT_N; Ns*=FFT_R ){ 
		float angle = -2*M_PI*(j%Ns)/(Ns*FFT_R); 
		for( int r=0; r<FFT_R; r++ ){
			v[r] = v[r]*make_float2(cos(r*angle), sin(r*angle));
		}

		GPU_FFT2( v );

		int idxD = GPU_expand(j,Ns,FFT_R); 
		int idxS = GPU_expand(j,FFT_N/FFT_R,FFT_R); 
		G_GPU_exchange_fft_lbm_fft2( v,stride, idxD,Ns, idxS,FFT_N/FFT_R);
	}      
}

__device__ void fft_lbm_fft2(float2* data, 
	int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base){
	
	unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

	// // ori
	// int thread_id_x = threadIdx.x - thread_step;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    // int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

	for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

		// // ori
		// int block_id_x = block_pos;
        int block_id_x = block_pos % grid_dimension_x;
		// int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

		float2 *ori_data = data + block_id_x * FFT_N;
		float2 v[FFT_R];
		// data = ori_data;

		int idxG = thread_id_x; 
		for (int r=0; r<FFT_R; r++) {  
			v[r] = ori_data[idxG + r*FFT_T];
		} 
		G_GPU_DoFft_fft_lbm_fft2( v, thread_id_x);  
		for (int r=0; r<FFT_R; r++) {
			ori_data[idxG + r*FFT_T] = v[r];
		}
	}
}

__device__ void G_GPU_exchange_fft_lbm_fft3( float2* v, int stride, int idxD, int incD, 
	int idxS, int incS){ 
	__shared__ float work[FFT_T*FFT_R*2];//FFT_T*FFT_R*2
	float* sr = work;
	float* si = work+FFT_T*FFT_R;  
	// __syncthreads(); 
asm volatile("bar.sync %0, %1;" : : "r"(4), "r"(128) : "memory");
	for( int r=0; r<FFT_R; r++ ) { 
		int i = (idxD + r*incD)*stride; 
		sr[i] = v[r].x;
		si[i] = v[r].y;  
	}   
	// __syncthreads(); 
asm volatile("bar.sync %0, %1;" : : "r"(4), "r"(128) : "memory");

	for( int r=0; r<FFT_R; r++ ) { 
		int i = (idxS + r*incS)*stride;     
		v[r] = make_float2(sr[i], si[i]);  
	}        
}  

__device__ void G_GPU_DoFft_fft_lbm_fft3(float2* v, int j, int stride=1) { 
	for( int Ns=1; Ns<FFT_N; Ns*=FFT_R ){ 
		float angle = -2*M_PI*(j%Ns)/(Ns*FFT_R); 
		for( int r=0; r<FFT_R; r++ ){
			v[r] = v[r]*make_float2(cos(r*angle), sin(r*angle));
		}

		GPU_FFT2( v );

		int idxD = GPU_expand(j,Ns,FFT_R); 
		int idxS = GPU_expand(j,FFT_N/FFT_R,FFT_R); 
		G_GPU_exchange_fft_lbm_fft3( v,stride, idxD,Ns, idxS,FFT_N/FFT_R);
	}      
}

__device__ void fft_lbm_fft3(float2* data, 
	int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base){
	
	unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

	// // ori
	// int thread_id_x = threadIdx.x - thread_step;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    // int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

	for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

		// // ori
		// int block_id_x = block_pos;
        int block_id_x = block_pos % grid_dimension_x;
		// int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

		float2 *ori_data = data + block_id_x * FFT_N;
		float2 v[FFT_R];
		// data = ori_data;

		int idxG = thread_id_x; 
		for (int r=0; r<FFT_R; r++) {  
			v[r] = ori_data[idxG + r*FFT_T];
		} 
		G_GPU_DoFft_fft_lbm_fft3( v, thread_id_x);  
		for (int r=0; r<FFT_R; r++) {
			ori_data[idxG + r*FFT_T] = v[r];
		}
	}
}

__device__ void G_GPU_exchange_fft_lbm_fft4( float2* v, int stride, int idxD, int incD, 
	int idxS, int incS){ 
	__shared__ float work[FFT_T*FFT_R*2];//FFT_T*FFT_R*2
	float* sr = work;
	float* si = work+FFT_T*FFT_R;  
	// __syncthreads(); 
asm volatile("bar.sync %0, %1;" : : "r"(5), "r"(128) : "memory");
	for( int r=0; r<FFT_R; r++ ) { 
		int i = (idxD + r*incD)*stride; 
		sr[i] = v[r].x;
		si[i] = v[r].y;  
	}   
	// __syncthreads(); 
asm volatile("bar.sync %0, %1;" : : "r"(5), "r"(128) : "memory");

	for( int r=0; r<FFT_R; r++ ) { 
		int i = (idxS + r*incS)*stride;     
		v[r] = make_float2(sr[i], si[i]);  
	}        
}  

__device__ void G_GPU_DoFft_fft_lbm_fft4(float2* v, int j, int stride=1) { 
	for( int Ns=1; Ns<FFT_N; Ns*=FFT_R ){ 
		float angle = -2*M_PI*(j%Ns)/(Ns*FFT_R); 
		for( int r=0; r<FFT_R; r++ ){
			v[r] = v[r]*make_float2(cos(r*angle), sin(r*angle));
		}

		GPU_FFT2( v );

		int idxD = GPU_expand(j,Ns,FFT_R); 
		int idxS = GPU_expand(j,FFT_N/FFT_R,FFT_R); 
		G_GPU_exchange_fft_lbm_fft4( v,stride, idxD,Ns, idxS,FFT_N/FFT_R);
	}      
}

__device__ void fft_lbm_fft4(float2* data, 
	int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base){
	
	unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

	// // ori
	// int thread_id_x = threadIdx.x - thread_step;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    // int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

	for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

		// // ori
		// int block_id_x = block_pos;
        int block_id_x = block_pos % grid_dimension_x;
		// int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

		float2 *ori_data = data + block_id_x * FFT_N;
		float2 v[FFT_R];
		// data = ori_data;

		int idxG = thread_id_x; 
		for (int r=0; r<FFT_R; r++) {  
			v[r] = ori_data[idxG + r*FFT_T];
		} 
		G_GPU_DoFft_fft_lbm_fft4( v, thread_id_x);  
		for (int r=0; r<FFT_R; r++) {
			ori_data[idxG + r*FFT_T] = v[r];
		}
	}
}

__device__ void G_GPU_exchange_fft_lbm_fft5( float2* v, int stride, int idxD, int incD, 
	int idxS, int incS){ 
	__shared__ float work[FFT_T*FFT_R*2];//FFT_T*FFT_R*2
	float* sr = work;
	float* si = work+FFT_T*FFT_R;  
	// __syncthreads(); 
asm volatile("bar.sync %0, %1;" : : "r"(6), "r"(128) : "memory");
	for( int r=0; r<FFT_R; r++ ) { 
		int i = (idxD + r*incD)*stride; 
		sr[i] = v[r].x;
		si[i] = v[r].y;  
	}   
	// __syncthreads(); 
asm volatile("bar.sync %0, %1;" : : "r"(6), "r"(128) : "memory");

	for( int r=0; r<FFT_R; r++ ) { 
		int i = (idxS + r*incS)*stride;     
		v[r] = make_float2(sr[i], si[i]);  
	}        
}  

__device__ void G_GPU_DoFft_fft_lbm_fft5(float2* v, int j, int stride=1) { 
	for( int Ns=1; Ns<FFT_N; Ns*=FFT_R ){ 
		float angle = -2*M_PI*(j%Ns)/(Ns*FFT_R); 
		for( int r=0; r<FFT_R; r++ ){
			v[r] = v[r]*make_float2(cos(r*angle), sin(r*angle));
		}

		GPU_FFT2( v );

		int idxD = GPU_expand(j,Ns,FFT_R); 
		int idxS = GPU_expand(j,FFT_N/FFT_R,FFT_R); 
		G_GPU_exchange_fft_lbm_fft5( v,stride, idxD,Ns, idxS,FFT_N/FFT_R);
	}      
}

__device__ void fft_lbm_fft5(float2* data, 
	int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base){
	
	unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

	// // ori
	// int thread_id_x = threadIdx.x - thread_step;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    // int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

	for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

		// // ori
		// int block_id_x = block_pos;
        int block_id_x = block_pos % grid_dimension_x;
		// int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

		float2 *ori_data = data + block_id_x * FFT_N;
		float2 v[FFT_R];
		// data = ori_data;

		int idxG = thread_id_x; 
		for (int r=0; r<FFT_R; r++) {  
			v[r] = ori_data[idxG + r*FFT_T];
		} 
		G_GPU_DoFft_fft_lbm_fft5( v, thread_id_x );  
		for (int r=0; r<FFT_R; r++) {
			ori_data[idxG + r*FFT_T] = v[r];
		}
	}
}

// lbm
__device__ void fft_lbm_lbm0( float* srcGrid, float* dstGrid,
    int grid_dimension_x, int grid_dimension_y, int grid_dimension_z,
    int block_dimension_x, int block_dimension_y, int block_dimension_z,
    int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base) {
	//Using some predefined macros here.  Consider this the declaration 
    //  and initialization of the variables SWEEP_X, SWEEP_Y and SWEEP_Z

    // // ori
    // int thread_id_x = (threadIdx.x - thread_step) % block_dimension_x;

    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;
    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    // int thread_id_y = ((threadIdx.x - thread_step) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = ((threadIdx.x - thread_step) / block_dimension_x) / block_dimension_y;

    SWEEP_VAR
	float temp_swp, tempC, tempN, tempS, tempE, tempW, tempT, tempB;
	float tempNE, tempNW, tempSE, tempSW, tempNT, tempNB, tempST ;
	float tempSB, tempET, tempEB, tempWT, tempWB;

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        // ori
        // int block_id_x = block_pos % grid_dimension_x;
        // int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = (block_pos / grid_dimension_x) / grid_dimension_y;

        int block_id_x = block_pos % grid_dimension_x;
		int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

        // float *d_temp = srcGrid;
        // srcGrid = dstGrid;
        // dstGrid = d_temp;

        SWEEP_X = thread_id_x;
        SWEEP_Y = block_id_x;
        SWEEP_Z = block_id_y;

        //Load all of the input fields
        //This is a gather operation of the SCATTER preprocessor variable
            // is undefined in layout_config.h, or a "local" read otherwise
        tempC = SRC_C(srcGrid);
        tempN = SRC_N(srcGrid);
        tempS = SRC_S(srcGrid);
        tempE = SRC_E(srcGrid);
        tempW = SRC_W(srcGrid);
        tempT = SRC_T(srcGrid);
        tempB = SRC_B(srcGrid);
        tempNE= SRC_NE(srcGrid);
        tempNW= SRC_NW(srcGrid);
        tempSE = SRC_SE(srcGrid);
        tempSW = SRC_SW(srcGrid);
        tempNT = SRC_NT(srcGrid);
        tempNB = SRC_NB(srcGrid);
        tempST = SRC_ST(srcGrid);
        tempSB = SRC_SB(srcGrid);
        tempET = SRC_ET(srcGrid);
        tempEB = SRC_EB(srcGrid);
        tempWT = SRC_WT(srcGrid);
        tempWB = SRC_WB(srcGrid);

        //Test whether the cell is fluid or obstacle
        if( TEST_FLAG_SWEEP( srcGrid, OBSTACLE )) {
            //Swizzle the inputs: reflect any fluid coming into this cell 
            // back to where it came from
            temp_swp = tempN ; tempN = tempS ; tempS = temp_swp ;
            temp_swp = tempE ; tempE = tempW ; tempW = temp_swp;
            temp_swp = tempT ; tempT = tempB ; tempB = temp_swp;
            temp_swp = tempNE; tempNE = tempSW ; tempSW = temp_swp;
            temp_swp = tempNW; tempNW = tempSE ; tempSE = temp_swp;
            temp_swp = tempNT ; tempNT = tempSB ; tempSB = temp_swp; 
            temp_swp = tempNB ; tempNB = tempST ; tempST = temp_swp;
            temp_swp = tempET ; tempET= tempWB ; tempWB = temp_swp;
            temp_swp = tempEB ; tempEB = tempWT ; tempWT = temp_swp;
        }
        else {
            //The math meat of LBM: ignore for optimization
            float ux, uy, uz, rho, u2;
            float temp1, temp2, temp_base;
            rho = tempC + tempN
                + tempS + tempE
                + tempW + tempT
                + tempB + tempNE
                + tempNW + tempSE
                + tempSW + tempNT
                + tempNB + tempST
                + tempSB + tempET
                + tempEB + tempWT
                + tempWB;

            ux = + tempE - tempW
                + tempNE - tempNW
                + tempSE - tempSW
                + tempET + tempEB
                - tempWT - tempWB;
            uy = + tempN - tempS
                + tempNE + tempNW
                - tempSE - tempSW
                + tempNT + tempNB
                - tempST - tempSB;
            uz = + tempT - tempB
                + tempNT - tempNB
                + tempST - tempSB
                + tempET - tempEB
                + tempWT - tempWB;

            ux /= rho;
            uy /= rho;
            uz /= rho;
            if( TEST_FLAG_SWEEP( srcGrid, ACCEL )) {
                ux = 0.005f;
                uy = 0.002f;
                uz = 0.000f;
            }
            u2 = 1.5f * (ux*ux + uy*uy + uz*uz) - 1.0f;
            temp_base = OMEGA*rho;
            temp1 = DFL1*temp_base;


            //Put the output values for this cell in the shared memory
            temp_base = OMEGA*rho;
            temp1 = DFL1*temp_base;
            temp2 = 1.0f-OMEGA;
            tempC = temp2*tempC + temp1*(                                 - u2);
                temp1 = DFL2*temp_base;	
            tempN = temp2*tempN + temp1*(       uy*(4.5f*uy       + 3.0f) - u2);
            tempS = temp2*tempS + temp1*(       uy*(4.5f*uy       - 3.0f) - u2);
            tempT = temp2*tempT + temp1*(       uz*(4.5f*uz       + 3.0f) - u2);
            tempB = temp2*tempB + temp1*(       uz*(4.5f*uz       - 3.0f) - u2);
            tempE = temp2*tempE + temp1*(       ux*(4.5f*ux       + 3.0f) - u2);
            tempW = temp2*tempW + temp1*(       ux*(4.5f*ux       - 3.0f) - u2);
            temp1 = DFL3*temp_base;
            tempNT= temp2*tempNT + temp1 *( (+uy+uz)*(4.5f*(+uy+uz) + 3.0f) - u2);
            tempNB= temp2*tempNB + temp1 *( (+uy-uz)*(4.5f*(+uy-uz) + 3.0f) - u2);
            tempST= temp2*tempST + temp1 *( (-uy+uz)*(4.5f*(-uy+uz) + 3.0f) - u2);
            tempSB= temp2*tempSB + temp1 *( (-uy-uz)*(4.5f*(-uy-uz) + 3.0f) - u2);
            tempNE = temp2*tempNE + temp1 *( (+ux+uy)*(4.5f*(+ux+uy) + 3.0f) - u2);
            tempSE = temp2*tempSE + temp1 *((+ux-uy)*(4.5f*(+ux-uy) + 3.0f) - u2);
            tempET = temp2*tempET + temp1 *( (+ux+uz)*(4.5f*(+ux+uz) + 3.0f) - u2);
            tempEB = temp2*tempEB + temp1 *( (+ux-uz)*(4.5f*(+ux-uz) + 3.0f) - u2);
            tempNW = temp2*tempNW + temp1 *( (-ux+uy)*(4.5f*(-ux+uy) + 3.0f) - u2);
            tempSW = temp2*tempSW + temp1 *( (-ux-uy)*(4.5f*(-ux-uy) + 3.0f) - u2);
            tempWT = temp2*tempWT + temp1 *( (-ux+uz)*(4.5f*(-ux+uz) + 3.0f) - u2);
            tempWB = temp2*tempWB + temp1 *( (-ux-uz)*(4.5f*(-ux-uz) + 3.0f) - u2);
        }

        //Write the results computed above
        //This is a scatter operation of the SCATTER preprocessor variable
            // is defined in layout_config.h, or a "local" write otherwise
        DST_C ( dstGrid ) = tempC;

        DST_N ( dstGrid ) = tempN; 
        DST_S ( dstGrid ) = tempS;
        DST_E ( dstGrid ) = tempE;
        DST_W ( dstGrid ) = tempW;
        DST_T ( dstGrid ) = tempT;
        DST_B ( dstGrid ) = tempB;

        DST_NE( dstGrid ) = tempNE;
        DST_NW( dstGrid ) = tempNW;
        DST_SE( dstGrid ) = tempSE;
        DST_SW( dstGrid ) = tempSW;
        DST_NT( dstGrid ) = tempNT;
        DST_NB( dstGrid ) = tempNB;
        DST_ST( dstGrid ) = tempST;
        DST_SB( dstGrid ) = tempSB;
        DST_ET( dstGrid ) = tempET;
        DST_EB( dstGrid ) = tempEB;
        DST_WT( dstGrid ) = tempWT;
        DST_WB( dstGrid ) = tempWB;
    }
}

// fft-lbm-6-1
__global__ void mixed_fft_lbm_kernel_6_1(float2* fft0_data, int fft0_grid_dimension_x, int fft0_grid_dimension_y, int fft0_grid_dimension_z, int fft0_block_dimension_x, int fft0_block_dimension_y, int fft0_block_dimension_z, int fft0_ptb_start_block_pos, int fft0_ptb_iter_block_step, int fft0_ptb_end_block_pos, float* lbm1_srcGrid, float* lbm1_dstGrid, int lbm1_grid_dimension_x, int lbm1_grid_dimension_y, int lbm1_grid_dimension_z, int lbm1_block_dimension_x, int lbm1_block_dimension_y, int lbm1_block_dimension_z, int lbm1_ptb_start_block_pos, int lbm1_ptb_iter_block_step, int lbm1_ptb_end_block_pos){
    if (threadIdx.x < 128) {
        fft_lbm_fft0(
            fft0_data, fft0_grid_dimension_x, fft0_grid_dimension_y, fft0_grid_dimension_z, fft0_block_dimension_x, fft0_block_dimension_y, fft0_block_dimension_z, fft0_ptb_start_block_pos + 0 * fft0_ptb_iter_block_step, fft0_ptb_iter_block_step * 6, fft0_ptb_end_block_pos, 0
        );
    }
    else if (threadIdx.x < 256) {
        fft_lbm_fft1(
            fft0_data, fft0_grid_dimension_x, fft0_grid_dimension_y, fft0_grid_dimension_z, fft0_block_dimension_x, fft0_block_dimension_y, fft0_block_dimension_z, fft0_ptb_start_block_pos + 1 * fft0_ptb_iter_block_step, fft0_ptb_iter_block_step * 6, fft0_ptb_end_block_pos, 128
        );
    }
    else if (threadIdx.x < 384) {
        fft_lbm_fft2(
            fft0_data, fft0_grid_dimension_x, fft0_grid_dimension_y, fft0_grid_dimension_z, fft0_block_dimension_x, fft0_block_dimension_y, fft0_block_dimension_z, fft0_ptb_start_block_pos + 2 * fft0_ptb_iter_block_step, fft0_ptb_iter_block_step * 6, fft0_ptb_end_block_pos, 256
        );
    }
    else if (threadIdx.x < 512) {
        fft_lbm_fft3(
            fft0_data, fft0_grid_dimension_x, fft0_grid_dimension_y, fft0_grid_dimension_z, fft0_block_dimension_x, fft0_block_dimension_y, fft0_block_dimension_z, fft0_ptb_start_block_pos + 3 * fft0_ptb_iter_block_step, fft0_ptb_iter_block_step * 6, fft0_ptb_end_block_pos, 384
        );
    }
    else if (threadIdx.x < 640) {
        fft_lbm_fft4(
            fft0_data, fft0_grid_dimension_x, fft0_grid_dimension_y, fft0_grid_dimension_z, fft0_block_dimension_x, fft0_block_dimension_y, fft0_block_dimension_z, fft0_ptb_start_block_pos + 4 * fft0_ptb_iter_block_step, fft0_ptb_iter_block_step * 6, fft0_ptb_end_block_pos, 512
        );
    }
    else if (threadIdx.x < 768) {
        fft_lbm_fft5(
            fft0_data, fft0_grid_dimension_x, fft0_grid_dimension_y, fft0_grid_dimension_z, fft0_block_dimension_x, fft0_block_dimension_y, fft0_block_dimension_z, fft0_ptb_start_block_pos + 5 * fft0_ptb_iter_block_step, fft0_ptb_iter_block_step * 6, fft0_ptb_end_block_pos, 640
        );
    }
    else if (threadIdx.x < 896) {
        fft_lbm_lbm0(
            lbm1_srcGrid, lbm1_dstGrid, lbm1_grid_dimension_x, lbm1_grid_dimension_y, lbm1_grid_dimension_z, lbm1_block_dimension_x, lbm1_block_dimension_y, lbm1_block_dimension_z, lbm1_ptb_start_block_pos + 0 * lbm1_ptb_iter_block_step, lbm1_ptb_iter_block_step * 1, lbm1_ptb_end_block_pos, 768
        );
    }

}
