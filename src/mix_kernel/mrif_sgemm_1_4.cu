#include "hip/hip_runtime.h"
#include "header/sgemm_header.h"
#include "header/mrif_header.h"

// mrif
__device__ void mrif_sgemm_mrif0(int numK, int kGlobalIndex, float* x, float* y, float* z, float* outR, float* outI, 
	    int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		    int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base) {

    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;
	
	int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    // int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

	for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos % grid_dimension_x;
		// int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);
	

        for (int FHGrid = 0; FHGrid < 1; FHGrid++) {

            kGlobalIndex = FHGrid * KERNEL_FH_K_ELEMS_PER_GRID;
            float sX;
            float sY;
            float sZ;
            float sOutR;
            float sOutI;

            // Determine the element of the X arrays computed by this thread
            int xIndex = block_id_x * KERNEL_FH_THREADS_PER_BLOCK + thread_id_x;

            sX = x[xIndex];
            sY = y[xIndex];
            sZ = z[xIndex];
            sOutR = outR[xIndex];
            sOutI = outI[xIndex];

            // Loop over all elements of K in constant mem to compute a partial value
            // for X.
            int kIndex = 0;
            int kCnt = numK - kGlobalIndex;
            if (kCnt < KERNEL_FH_K_ELEMS_PER_GRID) {
                for (kIndex = 0; (kIndex < (kCnt % 4)) && (kGlobalIndex < numK);
                    kIndex++, kGlobalIndex++) {
                    float expArg = PIx2 * (c[kIndex].Kx * sX + c[kIndex].Ky * sY + c[kIndex].Kz * sZ);
                    float cosArg = cos(expArg);
                    float sinArg = sin(expArg);
                    sOutR += c[kIndex].RhoPhiR * cosArg - c[kIndex].RhoPhiI * sinArg;
                    sOutI += c[kIndex].RhoPhiI * cosArg + c[kIndex].RhoPhiR * sinArg;
                }
            }

            for (; (kIndex < KERNEL_FH_K_ELEMS_PER_GRID) && (kGlobalIndex < numK);
                    kIndex += 4, kGlobalIndex += 4) {
                float expArg = PIx2 * (c[kIndex].Kx * sX + c[kIndex].Ky * sY + c[kIndex].Kz * sZ);
                float cosArg = cos(expArg);
                float sinArg = sin(expArg);
                sOutR += c[kIndex].RhoPhiR * cosArg - c[kIndex].RhoPhiI * sinArg;
                sOutI += c[kIndex].RhoPhiI * cosArg + c[kIndex].RhoPhiR * sinArg;

                int kIndex1 = kIndex + 1;
                float expArg1 = PIx2 * (c[kIndex1].Kx * sX + c[kIndex1].Ky * sY + c[kIndex1].Kz * sZ);
                float cosArg1 = cos(expArg1);
                float sinArg1 = sin(expArg1);
                sOutR += c[kIndex1].RhoPhiR * cosArg1 - c[kIndex1].RhoPhiI * sinArg1;
                sOutI += c[kIndex1].RhoPhiI * cosArg1 + c[kIndex1].RhoPhiR * sinArg1;

                int kIndex2 = kIndex + 2;
                float expArg2 = PIx2 * (c[kIndex2].Kx * sX + c[kIndex2].Ky * sY + c[kIndex2].Kz * sZ);
                float cosArg2 = cos(expArg2);
                float sinArg2 = sin(expArg2);
                sOutR += c[kIndex2].RhoPhiR * cosArg2 - c[kIndex2].RhoPhiI * sinArg2;
                sOutI += c[kIndex2].RhoPhiI * cosArg2 + c[kIndex2].RhoPhiR * sinArg2;

                int kIndex3 = kIndex + 3;
                float expArg3 = PIx2 * (c[kIndex3].Kx * sX + c[kIndex3].Ky * sY + c[kIndex3].Kz * sZ);
                float cosArg3 = cos(expArg3);
                float sinArg3 = sin(expArg3);
                sOutR += c[kIndex3].RhoPhiR * cosArg3 - c[kIndex3].RhoPhiI * sinArg3;
                sOutI += c[kIndex3].RhoPhiI * cosArg3 + c[kIndex3].RhoPhiR * sinArg3;    
            }

            outR[xIndex] = sOutR;
            outI[xIndex] = sOutI;
        }
	}
}

// sgemm
__device__ void mrif_sgemm_sgemm0(float *A, float *B, float *C, int NORMAL_M, int NORMAL_N, int NORMAL_K, 
	        int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		        int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base) {
    
    int lda = NORMAL_M;
    int ldb = NORMAL_N;
    int ldc = NORMAL_M;

    float alpha = 2.0f;
    float beta = 2.0f;

    // // ori
    // unsigned int block_pos = blockIdx.x;
    // int thread_id_x = (threadIdx.x - thread_step) % block_dimension_x;
    // int thread_id_y = (threadIdx.x - thread_step) / block_dimension_x;

    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

    __shared__ float b_s[TILE_TB_HEIGHT][TILE_N];

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos % grid_dimension_x;
		int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

        // Partial results
        float c[TILE_N];
        for (int i = 0; i < TILE_N; i++)
            c[i] = 0.0f;
        int mid = (threadIdx.x - thread_base); // TODO: check
        int m = block_id_x * TILE_M + mid;
        int n = block_id_y * TILE_N + thread_id_x;
        

        for (int i = 0; i < NORMAL_K; i += TILE_TB_HEIGHT)
        {
            float a;
            b_s[thread_id_y][thread_id_x] = B[n + (i + thread_id_y) * ldb];
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(1), "r"(128) : "memory");
            for (int j = 0; j < TILE_TB_HEIGHT; j++)
            {
                a = A[m + (i + j) * lda];
                for (int kk = 0; kk < TILE_N; kk++)
                    c[kk] += a * b_s[j][kk];
            }
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(1), "r"(128) : "memory");
        }
        int t = ldc * block_id_y * TILE_N + m;
        for (int i = 0; i < TILE_N; i++)
        {
            C[t + i * ldc] = C[t + i * ldc] * beta + alpha * c[i];
        }
    }
}

__device__ void mrif_sgemm_sgemm1(float *A, float *B, float *C, int NORMAL_M, int NORMAL_N, int NORMAL_K, 
	        int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		        int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base) {
    
    int lda = NORMAL_M;
    int ldb = NORMAL_N;
    int ldc = NORMAL_M;

    float alpha = 2.0f;
    float beta = 2.0f;

    // // ori
    // unsigned int block_pos = blockIdx.x;
    // int thread_id_x = (threadIdx.x - thread_step) % block_dimension_x;
    // int thread_id_y = (threadIdx.x - thread_step) / block_dimension_x;

    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

    __shared__ float b_s[TILE_TB_HEIGHT][TILE_N];

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos % grid_dimension_x;
		int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

        // Partial results
        float c[TILE_N];
        for (int i = 0; i < TILE_N; i++)
            c[i] = 0.0f;
        int mid = (threadIdx.x - thread_base); // TODO: check
        int m = block_id_x * TILE_M + mid;
        int n = block_id_y * TILE_N + thread_id_x;
        

        for (int i = 0; i < NORMAL_K; i += TILE_TB_HEIGHT)
        {
            float a;
            b_s[thread_id_y][thread_id_x] = B[n + (i + thread_id_y) * ldb];
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(2), "r"(128) : "memory");
            for (int j = 0; j < TILE_TB_HEIGHT; j++)
            {
                a = A[m + (i + j) * lda];
                for (int kk = 0; kk < TILE_N; kk++)
                    c[kk] += a * b_s[j][kk];
            }
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(2), "r"(128) : "memory");
        }
        int t = ldc * block_id_y * TILE_N + m;
        for (int i = 0; i < TILE_N; i++)
        {
            C[t + i * ldc] = C[t + i * ldc] * beta + alpha * c[i];
        }
    }
}

__device__ void mrif_sgemm_sgemm2(float *A, float *B, float *C, int NORMAL_M, int NORMAL_N, int NORMAL_K, 
	        int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		        int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base) {
    
    int lda = NORMAL_M;
    int ldb = NORMAL_N;
    int ldc = NORMAL_M;

    float alpha = 2.0f;
    float beta = 2.0f;

    // // ori
    // unsigned int block_pos = blockIdx.x;
    // int thread_id_x = (threadIdx.x - thread_step) % block_dimension_x;
    // int thread_id_y = (threadIdx.x - thread_step) / block_dimension_x;

    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

    __shared__ float b_s[TILE_TB_HEIGHT][TILE_N];

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos % grid_dimension_x;
		int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

        // Partial results
        float c[TILE_N];
        for (int i = 0; i < TILE_N; i++)
            c[i] = 0.0f;
        int mid = (threadIdx.x - thread_base); // TODO: check
        int m = block_id_x * TILE_M + mid;
        int n = block_id_y * TILE_N + thread_id_x;
        

        for (int i = 0; i < NORMAL_K; i += TILE_TB_HEIGHT)
        {
            float a;
            b_s[thread_id_y][thread_id_x] = B[n + (i + thread_id_y) * ldb];
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(3), "r"(128) : "memory");
            for (int j = 0; j < TILE_TB_HEIGHT; j++)
            {
                a = A[m + (i + j) * lda];
                for (int kk = 0; kk < TILE_N; kk++)
                    c[kk] += a * b_s[j][kk];
            }
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(3), "r"(128) : "memory");
        }
        int t = ldc * block_id_y * TILE_N + m;
        for (int i = 0; i < TILE_N; i++)
        {
            C[t + i * ldc] = C[t + i * ldc] * beta + alpha * c[i];
        }
    }
}

__device__ void mrif_sgemm_sgemm3(float *A, float *B, float *C, int NORMAL_M, int NORMAL_N, int NORMAL_K, 
	        int grid_dimension_x, int grid_dimension_y, int grid_dimension_z, int block_dimension_x, int block_dimension_y, int block_dimension_z,  
		        int ptb_start_block_pos, int ptb_iter_block_step, int ptb_end_block_pos, int thread_base) {
    
    int lda = NORMAL_M;
    int ldb = NORMAL_N;
    int ldc = NORMAL_M;

    float alpha = 2.0f;
    float beta = 2.0f;

    // // ori
    // unsigned int block_pos = blockIdx.x;
    // int thread_id_x = (threadIdx.x - thread_step) % block_dimension_x;
    // int thread_id_y = (threadIdx.x - thread_step) / block_dimension_x;

    unsigned int block_pos = blockIdx.x + ptb_start_block_pos;

    int thread_id_x = (threadIdx.x - thread_base) % block_dimension_x;
    int thread_id_y = ((threadIdx.x - thread_base) / block_dimension_x) % block_dimension_y;
    // int thread_id_z = (threadIdx.x - thread_base) / (block_dimension_x * block_dimension_y);

    __shared__ float b_s[TILE_TB_HEIGHT][TILE_N];

    for (;; block_pos += ptb_iter_block_step) {
        if (block_pos >= ptb_end_block_pos) {
            return;
        }

        int block_id_x = block_pos % grid_dimension_x;
		int block_id_y = (block_pos / grid_dimension_x) % grid_dimension_y;
        // int block_id_z = block_pos / (grid_dimension_x * grid_dimension_y);

        // Partial results
        float c[TILE_N];
        for (int i = 0; i < TILE_N; i++)
            c[i] = 0.0f;
        int mid = (threadIdx.x - thread_base); // TODO: check
        int m = block_id_x * TILE_M + mid;
        int n = block_id_y * TILE_N + thread_id_x;
        

        for (int i = 0; i < NORMAL_K; i += TILE_TB_HEIGHT)
        {
            float a;
            b_s[thread_id_y][thread_id_x] = B[n + (i + thread_id_y) * ldb];
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(4), "r"(128) : "memory");
            for (int j = 0; j < TILE_TB_HEIGHT; j++)
            {
                a = A[m + (i + j) * lda];
                for (int kk = 0; kk < TILE_N; kk++)
                    c[kk] += a * b_s[j][kk];
            }
            // __syncthreads();
asm volatile("bar.sync %0, %1;" : : "r"(4), "r"(128) : "memory");
        }
        int t = ldc * block_id_y * TILE_N + m;
        for (int i = 0; i < TILE_N; i++)
        {
            C[t + i * ldc] = C[t + i * ldc] * beta + alpha * c[i];
        }
    }
}

// mrif-sgemm-1-4
__global__ void mixed_mrif_sgemm_kernel_1_4(int mrif0_numK, int mrif0_kGlobalIndex, float* mrif0_x, float* mrif0_y, float* mrif0_z, float* mrif0_outR, float* mrif0_outI, int mrif0_grid_dimension_x, int mrif0_grid_dimension_y, int mrif0_grid_dimension_z, int mrif0_block_dimension_x, int mrif0_block_dimension_y, int mrif0_block_dimension_z, int mrif0_ptb_start_block_pos, int mrif0_ptb_iter_block_step, int mrif0_ptb_end_block_pos, float* sgemm1_A, float* sgemm1_B, float* sgemm1_C, int sgemm1_NORMAL_M, int sgemm1_NORMAL_N, int sgemm1_NORMAL_K, int sgemm1_grid_dimension_x, int sgemm1_grid_dimension_y, int sgemm1_grid_dimension_z, int sgemm1_block_dimension_x, int sgemm1_block_dimension_y, int sgemm1_block_dimension_z, int sgemm1_ptb_start_block_pos, int sgemm1_ptb_iter_block_step, int sgemm1_ptb_end_block_pos){
    if (threadIdx.x < 256) {
        mrif_sgemm_mrif0(
            mrif0_numK, mrif0_kGlobalIndex, mrif0_x, mrif0_y, mrif0_z, mrif0_outR, mrif0_outI, mrif0_grid_dimension_x, mrif0_grid_dimension_y, mrif0_grid_dimension_z, mrif0_block_dimension_x, mrif0_block_dimension_y, mrif0_block_dimension_z, mrif0_ptb_start_block_pos + 0 * mrif0_ptb_iter_block_step, mrif0_ptb_iter_block_step * 1, mrif0_ptb_end_block_pos, 0
        );
    }
    else if (threadIdx.x < 384) {
        mrif_sgemm_sgemm0(
            sgemm1_A, sgemm1_B, sgemm1_C, sgemm1_NORMAL_M, sgemm1_NORMAL_N, sgemm1_NORMAL_K, sgemm1_grid_dimension_x, sgemm1_grid_dimension_y, sgemm1_grid_dimension_z, sgemm1_block_dimension_x, sgemm1_block_dimension_y, sgemm1_block_dimension_z, sgemm1_ptb_start_block_pos + 0 * sgemm1_ptb_iter_block_step, sgemm1_ptb_iter_block_step * 4, sgemm1_ptb_end_block_pos, 256
        );
    }
    else if (threadIdx.x < 512) {
        mrif_sgemm_sgemm1(
            sgemm1_A, sgemm1_B, sgemm1_C, sgemm1_NORMAL_M, sgemm1_NORMAL_N, sgemm1_NORMAL_K, sgemm1_grid_dimension_x, sgemm1_grid_dimension_y, sgemm1_grid_dimension_z, sgemm1_block_dimension_x, sgemm1_block_dimension_y, sgemm1_block_dimension_z, sgemm1_ptb_start_block_pos + 1 * sgemm1_ptb_iter_block_step, sgemm1_ptb_iter_block_step * 4, sgemm1_ptb_end_block_pos, 384
        );
    }
    else if (threadIdx.x < 640) {
        mrif_sgemm_sgemm2(
            sgemm1_A, sgemm1_B, sgemm1_C, sgemm1_NORMAL_M, sgemm1_NORMAL_N, sgemm1_NORMAL_K, sgemm1_grid_dimension_x, sgemm1_grid_dimension_y, sgemm1_grid_dimension_z, sgemm1_block_dimension_x, sgemm1_block_dimension_y, sgemm1_block_dimension_z, sgemm1_ptb_start_block_pos + 2 * sgemm1_ptb_iter_block_step, sgemm1_ptb_iter_block_step * 4, sgemm1_ptb_end_block_pos, 512
        );
    }
    else if (threadIdx.x < 768) {
        mrif_sgemm_sgemm3(
            sgemm1_A, sgemm1_B, sgemm1_C, sgemm1_NORMAL_M, sgemm1_NORMAL_N, sgemm1_NORMAL_K, sgemm1_grid_dimension_x, sgemm1_grid_dimension_y, sgemm1_grid_dimension_z, sgemm1_block_dimension_x, sgemm1_block_dimension_y, sgemm1_block_dimension_z, sgemm1_ptb_start_block_pos + 3 * sgemm1_ptb_iter_block_step, sgemm1_ptb_iter_block_step * 4, sgemm1_ptb_end_block_pos, 640
        );
    }

}
